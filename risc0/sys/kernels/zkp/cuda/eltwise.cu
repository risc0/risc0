#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "fpext.h"

__global__ void eltwise_add_fp(Fp* out, const Fp* x, const Fp* y, const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    out[idx] = x[idx] + y[idx];
  }
}

__global__ void eltwise_mul_factor_fp(Fp* io, Fp factor, const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    io[idx] = io[idx] * factor;
  }
}

__global__ void eltwise_copy_fp(Fp* out, const Fp* in, const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  // If the following check is not included, there is a SIGABRT that causes tests to fail
  // cuda-memcheck also throws lots of out of bounds read errors if this check is omitted
  if (idx < count) {
    out[idx] = in[idx];
  }
}

__global__ void eltwise_copy_fp_region(Fp* into,
                                       const Fp* from,
                                       const uint32_t fromRows,
                                       const uint32_t fromCols,
                                       const uint32_t fromOffset,
                                       const uint32_t fromStride,
                                       const uint32_t intoOffset,
                                       const uint32_t intoStride) {
  uint row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < fromRows) {
    for (uint32_t col = 0; col < fromCols; col++) {
      into[intoOffset + row * intoStride + col] = from[fromOffset + row * fromStride + col];
    }
  }
}

__global__ void
eltwise_sum_fpext(Fp* out, const FpExt* in, const uint32_t to_add, const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    FpExt tot;
    for (size_t i = 0; i < to_add; i++) {
      tot += in[count * i + idx];
    }
    out[idx + 0 * count] = tot.elems[0];
    out[idx + 1 * count] = tot.elems[1];
    out[idx + 2 * count] = tot.elems[2];
    out[idx + 3 * count] = tot.elems[3];
  }
}

__global__ void eltwise_zeroize_fp(Fp* elems, uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
      Fp val = elems[idx];
      elems[idx] = val.zeroize();
  }
}

__global__ void eltwise_zeroize_fpext(FpExt* elems, uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
      FpExt val = elems[idx];
      elems[idx] = val.zeroize();
  }
}
