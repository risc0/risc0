// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hip/hip_runtime.h"
#include "fp.h"
#include "fpext.h"
#include "kernels.h"

#include "vendor/nvtx3/nvtx3.hpp"

#include <cstdint>
#include <exception>

extern "C" {

const char* risc0_zkp_cuda_eltwise_add_fp(Fp* out, const Fp* x, const Fp* y, uint32_t count) {
  return launchKernel(eltwise_add_fp, count, 0, out, x, y, count);
}

const char* risc0_zkp_cuda_eltwise_mul_factor_fp(Fp* io, Fp factor, uint32_t count) {
  return launchKernel(eltwise_mul_factor_fp, count, 0, io, factor, count);
}

const char* risc0_zkp_cuda_eltwise_copy_fp(Fp* out, const Fp* in, const uint32_t count) {
  return launchKernel(eltwise_copy_fp, count, 0, out, in, count);
}

const char* risc0_zkp_cuda_eltwise_copy_fp_region(Fp* into,
                                                  const Fp* from,
                                                  const uint32_t fromRows,
                                                  const uint32_t fromCols,
                                                  const uint32_t fromOffset,
                                                  const uint32_t fromStride,
                                                  const uint32_t intoOffset,
                                                  const uint32_t intoStride) {
  return launchKernel(eltwise_copy_fp_region,
                      fromRows,
                      0,
                      into,
                      from,
                      fromRows,
                      fromCols,
                      fromOffset,
                      fromStride,
                      intoOffset,
                      intoStride);
}

const char* risc0_zkp_cuda_eltwise_sum_fpext(Fp* out,
                                             const FpExt* in,
                                             const uint32_t to_add,
                                             const uint32_t count) {
  return launchKernel(eltwise_sum_fpext, count, 0, out, in, to_add, count);
}

const char* risc0_zkp_cuda_eltwise_zeroize_fp(Fp* elems, const uint32_t count) {
  return launchKernel(eltwise_zeroize_fp, count, 0, elems, count);
}

const char* risc0_zkp_cuda_eltwise_zeroize_fpext(FpExt* elems, const uint32_t count) {
  return launchKernel(eltwise_zeroize_fpext, count, 0, elems, count);
}

const char* risc0_zkp_cuda_fri_fold(Fp* out, const Fp* in, const FpExt* mix, const uint32_t count) {
  return launchKernel(fri_fold, count, 0, out, in, mix, count);
}

const char* risc0_zkp_cuda_mix_poly_coeffs(FpExt* out,
                                           const Fp* in,
                                           const uint32_t* combos,
                                           const FpExt* mixStart,
                                           const FpExt* mix,
                                           const uint32_t inputSize,
                                           const uint32_t count) {
  return launchKernel(mix_poly_coeffs, count, 0, out, in, combos, mixStart, mix, inputSize, count);
}

const char* risc0_zkp_cuda_batch_bit_reverse(Fp* io, const uint32_t nBits, const uint32_t count) {
  return launchKernel(batch_bit_reverse, count, 0, io, nBits, count);
}

const char* risc0_zkp_cuda_batch_evaluate_any(FpExt* out,
                                              const Fp* coeffs,
                                              const uint32_t* which,
                                              const FpExt* xs,
                                              uint32_t shared_size,
                                              const uint32_t count,
                                              const uint32_t deg) {
  return launchKernel(batch_evaluate_any, count, shared_size, out, coeffs, which, xs, deg);
}

const char* risc0_zkp_cuda_gather_sample(
    Fp* dst, const Fp* src, const uint32_t idx, const uint32_t size, const uint32_t stride) {
  return launchKernel(gather_sample, size, 0, dst, src, idx, size, stride);
}

const char* risc0_zkp_cuda_scatter(Fp* into,
                                   const uint32_t* index,
                                   const uint32_t* offsets,
                                   const Fp* values,
                                   const uint32_t count) {
  return launchKernel(scatter, count, 0, into, index, offsets, values, count);
}

const char*
risc0_zkp_cuda_sha_rows(ShaDigest* output, const Fp* matrix, uint32_t rowSize, uint32_t colSize) {
  return launchKernel(sha_rows, rowSize, 0, output, matrix, rowSize, colSize);
}

const char* risc0_zkp_cuda_sha_fold(ShaDigest* output, const ShaDigest* input, uint32_t count) {
  return launchKernel(sha_fold, count, 0, output, input, count);
}

const char* risc0_zkp_cuda_combos_prepare(FpExt* combos,
                                          const FpExt* coeffU,
                                          const uint32_t comboCount,
                                          const uint32_t cycles,
                                          const uint32_t regsCount,
                                          const uint32_t* regSizes,
                                          const uint32_t* regComboIds,
                                          const uint32_t checkSize,
                                          const FpExt* mix) {

  try {
    CudaStream stream;
    combos_prepare<<<1, 1, 0, stream>>>(
        combos, coeffU, regsCount, regSizes, regComboIds, cycles, mix, checkSize, comboCount);
  } catch (const std::exception& err) {
    return strdup(err.what());
  }
  return nullptr;
}

} // extern "C"
