#include <ff/baby_bear.hpp>
#include <util/gpu_t.cuh>
#include <util/rusterror.h>

#include "poseidon2.cuh"

extern "C" RustError::by_value
sppark_poseidon2_fold(poseidon_out_t* d_out, const poseidon_in_t* d_in, size_t num_hashes) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = num_hashes < 256 ? num_hashes : 256;
  size_t num_blocks = num_hashes < 256 ? 1 : num_hashes / 256;

  try {
    CUDA_OK(hipDeviceSynchronize());

    _poseidon2_fold<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, num_hashes);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_poseidon2_rows(poseidon_out_t* d_out, const fr_t* d_in, uint32_t count, uint32_t col_size) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;

  try {
    CUDA_OK(hipDeviceSynchronize());

    _poseidon2_rows<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, count, col_size);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}
