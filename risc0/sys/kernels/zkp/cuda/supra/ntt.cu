#include "ff/baby_bear.hpp"
#include "ntt/ntt.cuh"

extern "C" RustError::by_value sppark_init() {
  uint32_t lg_domain_size = 1;
  uint32_t domain_size = 1U << lg_domain_size;

  std::vector<fr_t> inout{domain_size};
  inout[0] = fr_t(1);
  inout[1] = fr_t(1);

  const gpu_t& gpu = select_gpu();

  try {
    CUDA_OK(hipDeviceSynchronize());

    NTT::Base(gpu,
              &inout[0],
              lg_domain_size,
              NTT::InputOutputOrder::NR,
              NTT::Direction::forward,
              NTT::Type::standard);
    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  } catch (...) {
    return RustError(hipErrorUnknown, "Generic exception");
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value sppark_batch_expand(
    fr_t* d_out, fr_t* d_in, uint32_t lg_domain_size, uint32_t lg_blowup, uint32_t poly_count) {
  if (lg_domain_size == 0)
    return RustError{hipSuccess};

  uint32_t domain_size = 1U << lg_domain_size;
  uint32_t ext_domain_size = domain_size << lg_blowup;

  const gpu_t& gpu = select_gpu();

  try {
    CUDA_OK(hipDeviceSynchronize());

    for (size_t c = 0; c < poly_count; c++) {
      NTT::LDE_expand(
          gpu, &d_out[c * ext_domain_size], &d_in[c * domain_size], lg_domain_size, lg_blowup);
    }

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  } catch (...) {
    return RustError(hipErrorUnknown, "Generic exception");
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_batch_NTT(fr_t* d_inout, uint32_t lg_domain_size, uint32_t poly_count) {
  if (lg_domain_size == 0)
    return RustError{hipSuccess};

  uint32_t domain_size = 1U << lg_domain_size;

  const gpu_t& gpu = select_gpu();

  try {
    CUDA_OK(hipDeviceSynchronize());

    for (size_t c = 0; c < poly_count; c++) {
      NTT::Base_dev_ptr(gpu,
                        &d_inout[c * domain_size],
                        lg_domain_size,
                        NTT::InputOutputOrder::RN,
                        NTT::Direction::forward,
                        NTT::Type::standard);
    }

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  } catch (...) {
    return RustError(hipErrorUnknown, "Generic exception");
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_batch_iNTT(fr_t* d_inout, uint32_t lg_domain_size, uint32_t poly_count) {
  if (lg_domain_size == 0)
    return RustError{hipSuccess};

  uint32_t domain_size = 1U << lg_domain_size;

  const gpu_t& gpu = select_gpu();

  try {
    CUDA_OK(hipDeviceSynchronize());

    for (size_t c = 0; c < poly_count; c++) {
      NTT::Base_dev_ptr(gpu,
                        &d_inout[c * domain_size],
                        lg_domain_size,
                        NTT::InputOutputOrder::NR,
                        NTT::Direction::inverse,
                        NTT::Type::standard);
    }

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  } catch (...) {
    return RustError(hipErrorUnknown, "Generic exception");
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_batch_zk_shift(fr_t* d_inout, uint32_t lg_domain_size, uint32_t poly_count) {
  if (lg_domain_size == 0)
    return RustError{hipSuccess};

  uint32_t domain_size = 1U << lg_domain_size;

  const gpu_t& gpu = select_gpu();

  try {
    CUDA_OK(hipDeviceSynchronize());

    for (size_t c = 0; c < poly_count; c++) {
      NTT::LDE_powers(gpu, &d_inout[c * domain_size], lg_domain_size);
    }

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  } catch (...) {
    return RustError(hipErrorUnknown, "Generic exception");
  }

  return RustError{hipSuccess};
}
