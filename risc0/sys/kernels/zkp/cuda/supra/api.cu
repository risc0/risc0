#include "hip/hip_runtime.h"
#include <ff/alt_bn128.hpp>
#include <ff/baby_bear.hpp>
#include <util/exception.cuh>
#include <util/gpu_t.cuh>
#include <util/rusterror.h>

#include "calc_prefix_operation.cuh"
#include "poly_divide.cuh"
#include "poseidon2.cuh"
#include "poseidon254.cuh"

extern "C" RustError::by_value
sppark_poseidon2_fold(poseidon_out_t* d_out, const poseidon_in_t* d_in, size_t num_hashes) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = num_hashes < 256 ? num_hashes : 256;
  size_t num_blocks = num_hashes < 256 ? 1 : num_hashes / 256;

  try {
    CUDA_OK(hipDeviceSynchronize());

    _poseidon2_fold<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, num_hashes);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_poseidon2_rows(poseidon_out_t* d_out, const fr_t* d_in, uint32_t count, uint32_t col_size) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;

  try {
    CUDA_OK(hipDeviceSynchronize());

    _poseidon2_rows<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, count, col_size);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

static void compute_grid_block_size(size_t total_count, size_t& block_size, size_t& num_blocks) {
  size_t min_block_size = 4 * WARP_SZ;

  if (total_count < (block_size * num_blocks)) {
    size_t count_per_block = total_count / num_blocks;

    if (count_per_block > min_block_size) {
      block_size = ((count_per_block + min_block_size - 1) / min_block_size) * min_block_size;
      num_blocks = (total_count + block_size - 1) / block_size;
    } else {
      block_size = min_block_size;
      num_blocks = (total_count + min_block_size - 1) / min_block_size;
    }
  } else {
    size_t base_iter = (total_count + (num_blocks * block_size) - 1) / (num_blocks * block_size);
    size_t out_block_size = block_size;

    for (size_t cur_block_size = block_size - min_block_size; cur_block_size >= min_block_size;
         cur_block_size -= min_block_size) {
      size_t cur_iter =
          (total_count + (num_blocks * cur_block_size) - 1) / (num_blocks * cur_block_size);

      if (cur_iter != base_iter)
        break;
      out_block_size = cur_block_size;
    }

    block_size = out_block_size;
  }
}

extern "C" RustError::by_value
sppark_poseidon254_fold(alt_bn128::fr_t* d_out, const alt_bn128::fr_t* d_in, size_t num_hashes) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = 512;
  size_t num_blocks = gpu.sm_count();

  compute_grid_block_size(num_hashes, block_size, num_blocks);

  try {
    _poseidon254_fold<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, num_hashes);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_poseidon254_rows(alt_bn128::fr_t* d_out, const fr_t* d_in, size_t count, uint32_t col_size) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = 512;
  size_t num_blocks = gpu.sm_count();

  compute_grid_block_size(count, block_size, num_blocks);

  try {
    _poseidon254_rows<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, count, col_size);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

enum Operation { OpAdd, OpMultiply };

extern "C" RustError::by_value
sppark_calc_prefix_operation(Fp4* in_elems, uint32_t count, Operation op) {
  const gpu_t& gpu = select_gpu();

  auto device_props = gpu_props(gpu.id());
  int major = device_props.major;
  const uint32_t CHUNK_MULT_80 = 44;
  const uint32_t CHUNK_MULT_70 = 45;

  uint32_t block_size = 256;
  const uint32_t CHUNK_ADD = 48;
  uint32_t CHUNK_MULT = major >= 8 ? CHUNK_MULT_80 : CHUNK_MULT_70;
  uint32_t tmp_num_block_add = (count + (block_size * CHUNK_ADD) - 1) / (block_size * CHUNK_ADD);
  uint32_t tmp_num_block_mult = (count + (block_size * CHUNK_MULT) - 1) / (block_size * CHUNK_MULT);
  uint32_t sm_count = gpu.sm_count();
  uint32_t num_blocks_add = std::min(tmp_num_block_add, sm_count);
  uint32_t num_blocks_mult = std::min(tmp_num_block_mult, sm_count);
  size_t sharedMemorySize = block_size * sizeof(Fp4);

  try {
    dev_ptr_t<Fp4> d_mem(count, gpu);
    Fp4* d_elems = &d_mem[0];

    gpu.HtoD(d_elems, in_elems, count);

    if (op == Operation::OpAdd)
      gpu.launch_coop(calc_prefix_op_kernel<Add<Fp4>, Fp4, CHUNK_ADD>,
                      {num_blocks_add, block_size, sharedMemorySize},
                      d_elems,
                      (size_t)count);
    else if (major >= 8)
      gpu.launch_coop(calc_prefix_op_kernel<Multiply<Fp4>, Fp4, CHUNK_MULT_80>,
                      {num_blocks_mult, block_size, sharedMemorySize},
                      d_elems,
                      (size_t)count);
    else
      gpu.launch_coop(calc_prefix_op_kernel<Multiply<Fp4>, Fp4, CHUNK_MULT_70>,
                      {num_blocks_mult, block_size, sharedMemorySize},
                      d_elems,
                      (size_t)count);

    gpu.DtoH(in_elems, d_elems, count);

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
supra_poly_divide(Fp4 d_inout[], size_t len, Fp4* remainder, const Fp4& pow) {
  const gpu_t& gpu = select_gpu();

  try {
    uint32_t gridDim = gpu.sm_count();
    const uint32_t blockDim = DIV_BLOCK_SZ;

    if (gridDim > blockDim) {
      gridDim = blockDim;
    }

    size_t blocks = (len + blockDim - 1) / blockDim;
    if (gridDim > blocks) {
      gridDim = blocks;
    }

    if (gridDim < 3) {
      gridDim = 1;
    }

    size_t sharedSz = sizeof(Fp4) * max(blockDim / WARP_SZ, gridDim);
    sharedSz += sizeof(Fp4) * WARP_SZ;

    gpu.launch_coop(
        d_div_by_x_minus_z<Fp4, true>, {gridDim, blockDim, sharedSz}, d_inout, len, pow);
    gpu.DtoH(remainder, &d_inout[len - 1], 1);

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}
