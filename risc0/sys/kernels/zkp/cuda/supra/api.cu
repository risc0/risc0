#include "hip/hip_runtime.h"
#include <ff/alt_bn128.hpp>
#include <ff/baby_bear.hpp>
#include <util/gpu_t.cuh>
#include <util/rusterror.h>

#include <polynomial/div_by_x_minus_z.cuh>
#include <polynomial/prefix_op.cuh>

#include "poseidon2.cuh"
#include "poseidon254.cuh"

extern "C" RustError::by_value
sppark_poseidon2_fold(poseidon_out_t* d_out, const poseidon_in_t* d_in, size_t num_hashes) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = num_hashes < 256 ? num_hashes : 256;
  size_t num_blocks = num_hashes < 256 ? 1 : num_hashes / 256;

  try {
    CUDA_OK(hipDeviceSynchronize());

    _poseidon2_fold<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, num_hashes);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_poseidon2_rows(poseidon_out_t* d_out, const fr_t* d_in, uint32_t count, uint32_t col_size) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;

  try {
    CUDA_OK(hipDeviceSynchronize());

    _poseidon2_rows<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, count, col_size);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

static void compute_grid_block_size(size_t total_count, size_t& block_size, size_t& num_blocks) {
  size_t min_block_size = 4 * WARP_SZ;

  if (total_count < (block_size * num_blocks)) {
    size_t count_per_block = total_count / num_blocks;

    if (count_per_block > min_block_size) {
      block_size = ((count_per_block + min_block_size - 1) / min_block_size) * min_block_size;
      num_blocks = (total_count + block_size - 1) / block_size;
    } else {
      block_size = min_block_size;
      num_blocks = (total_count + min_block_size - 1) / min_block_size;
    }
  } else {
    size_t base_iter = (total_count + (num_blocks * block_size) - 1) / (num_blocks * block_size);
    size_t out_block_size = block_size;

    for (size_t cur_block_size = block_size - min_block_size; cur_block_size >= min_block_size;
         cur_block_size -= min_block_size) {
      size_t cur_iter =
          (total_count + (num_blocks * cur_block_size) - 1) / (num_blocks * cur_block_size);

      if (cur_iter != base_iter)
        break;
      out_block_size = cur_block_size;
    }

    block_size = out_block_size;
  }
}

extern "C" RustError::by_value
sppark_poseidon254_fold(alt_bn128::fr_t* d_out, const alt_bn128::fr_t* d_in, size_t num_hashes) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = 512;
  size_t num_blocks = gpu.sm_count();

  compute_grid_block_size(num_hashes, block_size, num_blocks);

  try {
    _poseidon254_fold<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, num_hashes);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_poseidon254_rows(alt_bn128::fr_t* d_out, const fr_t* d_in, size_t count, uint32_t col_size) {
  const gpu_t& gpu = select_gpu();

  size_t block_size = 512;
  size_t num_blocks = gpu.sm_count();

  compute_grid_block_size(count, block_size, num_blocks);

  try {
    _poseidon254_rows<<<num_blocks, block_size, 0, gpu>>>(d_out, d_in, count, col_size);

    CUDA_OK(hipGetLastError());

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value sppark_prefix_product(fr4_t d_inout[/*count*/], uint32_t count) {
  const gpu_t& gpu = select_gpu();

  try {
    prefix_op<Multiply<fr4_t>>(d_inout, count, gpu);
    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
supra_poly_divide(fr4_t d_inout[/*len*/], size_t len, fr4_t* remainder, const fr4_t& pow) {
  const gpu_t& gpu = select_gpu();

  try {
    div_by_x_minus_z<true>(d_inout, len, pow, gpu);
    gpu.DtoH(remainder, &d_inout[len - 1], 1);
    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}
