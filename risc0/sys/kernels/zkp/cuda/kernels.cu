#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "fpext.h"

constexpr size_t kFriFold = 16;

/// Compute `ceil(log_2(in))`, i.e. find the smallest value `out` such that `2^out >= in`.
__device__ inline constexpr size_t log2Ceil(size_t in) {
  size_t r = 0;
  while ((1uz << r) < in) {
    r++;
  }
  return r;
}

__global__ void batch_bit_reverse(Fp* io, const uint32_t nBits, const uint32_t count) {
  uint totIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (totIdx < count) {
    uint32_t rowSize = 1 << nBits;
    uint32_t idx = totIdx & (rowSize - 1);
    uint32_t s = totIdx >> nBits;
    uint32_t ridx = __brev(idx) >> (32 - nBits);
    if (idx < ridx) {
      size_t idx1 = s * rowSize + idx;
      size_t idx2 = s * rowSize + ridx;
      Fp tmp = io[idx1];
      io[idx1] = io[idx2];
      io[idx2] = tmp;
    }
  }
}

__global__ void batch_evaluate_any(
    FpExt* out, const Fp* coeffs, const uint32_t* which, const FpExt* xs, const uint32_t deg) {
  const Fp* cur_poly = coeffs + which[blockIdx.x] * deg;
  FpExt x = xs[blockIdx.x];
  FpExt stepx = pow(x, blockDim.x);
  FpExt powx = pow(x, threadIdx.x);
  FpExt tot;
  for (size_t i = threadIdx.x; i < deg; i += blockDim.x) {
    tot += powx * cur_poly[i];
    powx *= stepx;
  }
  extern __shared__ uint32_t totsBuf[];
  FpExt* tots = reinterpret_cast<FpExt*>(totsBuf);
  tots[threadIdx.x] = tot;
  __syncthreads();
  unsigned cur = blockDim.x;
  while (cur) {
    cur /= 2;
    if (threadIdx.x < cur) {
      tots[threadIdx.x] = FpExt(tots[threadIdx.x]) + FpExt(tots[threadIdx.x + cur]);
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    out[blockIdx.x] = tots[0];
  }
}

__global__ void fri_fold(Fp* out, const Fp* in, const FpExt* mix, const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    FpExt tot;
    FpExt curMix(1);
    for (uint32_t i = 0; i < kFriFold; i++) {
      size_t rev_i = __brev(i) >> (32 - log2Ceil(kFriFold));
      size_t rev_idx = rev_i * count + idx;
      FpExt factor(in[0 * count * kFriFold + rev_idx],
                   in[1 * count * kFriFold + rev_idx],
                   in[2 * count * kFriFold + rev_idx],
                   in[3 * count * kFriFold + rev_idx]);
      tot += curMix * factor;
      curMix *= *mix;
    }
    for (size_t i = 0; i < 4; i++) {
      out[count * i + idx] = tot.elems[i];
    }
  }
}

__global__ void gather_sample(
    Fp* dst, const Fp* src, const uint32_t idx, const uint32_t size, const uint32_t stride) {
  uint gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    dst[gid] = src[gid * stride + idx];
  }
}

__global__ void scatter(Fp* into,
                        const uint32_t* index,
                        const uint32_t* offsets,
                        const Fp* values,
                        const uint32_t count) {
  uint gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < count) {
    for (uint32_t idx = index[gid]; idx < index[gid + 1]; idx++) {
      into[offsets[idx]] = values[idx];
    }
  }
}

__global__ void mix_poly_coeffs(FpExt* out,
                                const Fp* in,
                                const uint32_t* combos,
                                const FpExt* mixStart,
                                const FpExt* mix,
                                const uint32_t inputSize,
                                const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    FpExt cur = *mixStart;
    for (size_t i = 0; i < inputSize; i++) {
      size_t id = combos[i];
      out[count * id + idx] += cur * in[count * i + idx];
      cur *= *mix;
    }
  }
}
