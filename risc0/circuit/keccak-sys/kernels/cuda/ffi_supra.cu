#include "hip/hip_runtime.h"
// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0, <LICENSE-APACHE or
// http://apache.org/licenses/LICENSE-2.0> or the MIT license <LICENSE-MIT or
// http://opensource.org/licenses/MIT>, at your option. This file may not be
// copied, modified, or distributed except according to those terms.
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
// SPDX-License-Identifier: Apache-2.0 OR MIT

#include "eval_check.cuh"

#include "hip/hip_runtime.h"
#include "supra/fp.h"

#include <exception>

namespace risc0::circuit::keccak::cuda {

__constant__ FpExt poly_mix[kNumPolyMixPows];

__global__ void eval_check(Fp* check,
                           const Fp* ctrl,
                           const Fp* data,
                           const Fp* accum,
                           const Fp* mix,
                           const Fp* out,
                           const Fp rou,
                           uint32_t po2,
                           uint32_t domain) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle < domain) {
    FpExt tot = poly_fp(cycle, domain, ctrl, out, data, mix, accum);
    Fp x = pow(rou, cycle);
    Fp y = pow(Fp(3) * x, 1 << po2);
    FpExt ret = tot * inv(y - Fp(1));
    check[domain * 0 + cycle] = ret[0];
    check[domain * 1 + cycle] = ret[1];
    check[domain * 2 + cycle] = ret[2];
    check[domain * 3 + cycle] = ret[3];
  }
}

} // namespace risc0::circuit::keccak::cuda

extern "C" {

using namespace risc0::circuit::keccak::cuda;

const char* risc0_circuit_keccak_cuda_eval_check(Fp* check,
                                                 const Fp* ctrl,
                                                 const Fp* data,
                                                 const Fp* accum,
                                                 const Fp* mix,
                                                 const Fp* out,
                                                 const Fp& rou,
                                                 uint32_t po2,
                                                 uint32_t domain,
                                                 const FpExt* poly_mix_pows) {
  try {
    CUDA_OK(hipDeviceSynchronize());
    CudaStream stream;
    auto cfg = getSimpleConfig(domain);
    hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
    eval_check<<<cfg.grid, cfg.block, 0, stream>>>(
        check, ctrl, data, accum, mix, out, rou, po2, domain);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

} // extern "C"
