// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

__device__ void step_Top_13(ExecContext& ctx, MutableBuf arg0) {
  // builtin Sub
  // KeccackNextRound(zirgen/circuit/keccak/top.zir:410)
  // ComputeCurrentStep(zirgen/circuit/keccak/top.zir:461)
  // Top(zirgen/circuit/keccak/top.zir:483)
  Val x1 = (get(ctx, arg0, 15, 1) - Val(23));
  // builtin NondetReg
  // IsZero(zirgen/circuit/keccak/is_zero.zir:8)
  set(ctx, arg0, 935, isz(x1));
  Val x2 = get(ctx, arg0, 935, 0);
  // IsZero(zirgen/circuit/keccak/is_zero.zir:11)
  set(ctx, arg0, 936, inv_0(x1));
  if (to_size_t(x2)) {
    // KeccackNextRound(zirgen/circuit/keccak/top.zir:411)
    step_Top_11(ctx, arg0);
  }
  if (to_size_t((Val(1) - x2))) {
    step_Top_12(ctx, arg0);
  }
  return;
}
__device__ void step_Top_29(ExecContext& ctx, MutableBuf arg0) {
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  // ThetaP1(zirgen/circuit/keccak/keccak.zir:13)
  // KeccakRound0(zirgen/circuit/keccak/top.zir:97)
  // Top(zirgen/circuit/keccak/top.zir:496)
  Val x1 = ((get(ctx, arg0, 16, 2) + get(ctx, arg0, 176, 2)) + get(ctx, arg0, 336, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x2 = bitAnd(((x1 + get(ctx, arg0, 496, 2)) + get(ctx, arg0, 656, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 16, x2);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x3 = ((get(ctx, arg0, 17, 2) + get(ctx, arg0, 177, 2)) + get(ctx, arg0, 337, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x4 = bitAnd(((x3 + get(ctx, arg0, 497, 2)) + get(ctx, arg0, 657, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 17, x4);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x5 = ((get(ctx, arg0, 18, 2) + get(ctx, arg0, 178, 2)) + get(ctx, arg0, 338, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x6 = bitAnd(((x5 + get(ctx, arg0, 498, 2)) + get(ctx, arg0, 658, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 18, x6);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x7 = ((get(ctx, arg0, 19, 2) + get(ctx, arg0, 179, 2)) + get(ctx, arg0, 339, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x8 = bitAnd(((x7 + get(ctx, arg0, 499, 2)) + get(ctx, arg0, 659, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 19, x8);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x9 = ((get(ctx, arg0, 20, 2) + get(ctx, arg0, 180, 2)) + get(ctx, arg0, 340, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x10 = bitAnd(((x9 + get(ctx, arg0, 500, 2)) + get(ctx, arg0, 660, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 20, x10);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x11 = ((get(ctx, arg0, 21, 2) + get(ctx, arg0, 181, 2)) + get(ctx, arg0, 341, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x12 = bitAnd(((x11 + get(ctx, arg0, 501, 2)) + get(ctx, arg0, 661, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 21, x12);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x13 = ((get(ctx, arg0, 22, 2) + get(ctx, arg0, 182, 2)) + get(ctx, arg0, 342, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x14 = bitAnd(((x13 + get(ctx, arg0, 502, 2)) + get(ctx, arg0, 662, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 22, x14);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x15 = ((get(ctx, arg0, 23, 2) + get(ctx, arg0, 183, 2)) + get(ctx, arg0, 343, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x16 = bitAnd(((x15 + get(ctx, arg0, 503, 2)) + get(ctx, arg0, 663, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 23, x16);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x17 = ((get(ctx, arg0, 24, 2) + get(ctx, arg0, 184, 2)) + get(ctx, arg0, 344, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x18 = bitAnd(((x17 + get(ctx, arg0, 504, 2)) + get(ctx, arg0, 664, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 24, x18);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x19 = ((get(ctx, arg0, 25, 2) + get(ctx, arg0, 185, 2)) + get(ctx, arg0, 345, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x20 = bitAnd(((x19 + get(ctx, arg0, 505, 2)) + get(ctx, arg0, 665, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 25, x20);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x21 = ((get(ctx, arg0, 26, 2) + get(ctx, arg0, 186, 2)) + get(ctx, arg0, 346, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x22 = bitAnd(((x21 + get(ctx, arg0, 506, 2)) + get(ctx, arg0, 666, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 26, x22);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x23 = ((get(ctx, arg0, 27, 2) + get(ctx, arg0, 187, 2)) + get(ctx, arg0, 347, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x24 = bitAnd(((x23 + get(ctx, arg0, 507, 2)) + get(ctx, arg0, 667, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 27, x24);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x25 = ((get(ctx, arg0, 28, 2) + get(ctx, arg0, 188, 2)) + get(ctx, arg0, 348, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x26 = bitAnd(((x25 + get(ctx, arg0, 508, 2)) + get(ctx, arg0, 668, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 28, x26);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x27 = ((get(ctx, arg0, 29, 2) + get(ctx, arg0, 189, 2)) + get(ctx, arg0, 349, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x28 = bitAnd(((x27 + get(ctx, arg0, 509, 2)) + get(ctx, arg0, 669, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 29, x28);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x29 = ((get(ctx, arg0, 30, 2) + get(ctx, arg0, 190, 2)) + get(ctx, arg0, 350, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x30 = bitAnd(((x29 + get(ctx, arg0, 510, 2)) + get(ctx, arg0, 670, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 30, x30);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x31 = ((get(ctx, arg0, 31, 2) + get(ctx, arg0, 191, 2)) + get(ctx, arg0, 351, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x32 = bitAnd(((x31 + get(ctx, arg0, 511, 2)) + get(ctx, arg0, 671, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 31, x32);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x33 = ((get(ctx, arg0, 32, 2) + get(ctx, arg0, 192, 2)) + get(ctx, arg0, 352, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x34 = bitAnd(((x33 + get(ctx, arg0, 512, 2)) + get(ctx, arg0, 672, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 32, x34);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x35 = ((get(ctx, arg0, 33, 2) + get(ctx, arg0, 193, 2)) + get(ctx, arg0, 353, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x36 = bitAnd(((x35 + get(ctx, arg0, 513, 2)) + get(ctx, arg0, 673, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 33, x36);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x37 = ((get(ctx, arg0, 34, 2) + get(ctx, arg0, 194, 2)) + get(ctx, arg0, 354, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x38 = bitAnd(((x37 + get(ctx, arg0, 514, 2)) + get(ctx, arg0, 674, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 34, x38);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x39 = ((get(ctx, arg0, 35, 2) + get(ctx, arg0, 195, 2)) + get(ctx, arg0, 355, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x40 = bitAnd(((x39 + get(ctx, arg0, 515, 2)) + get(ctx, arg0, 675, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 35, x40);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x41 = ((get(ctx, arg0, 36, 2) + get(ctx, arg0, 196, 2)) + get(ctx, arg0, 356, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x42 = bitAnd(((x41 + get(ctx, arg0, 516, 2)) + get(ctx, arg0, 676, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 36, x42);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x43 = ((get(ctx, arg0, 37, 2) + get(ctx, arg0, 197, 2)) + get(ctx, arg0, 357, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x44 = bitAnd(((x43 + get(ctx, arg0, 517, 2)) + get(ctx, arg0, 677, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 37, x44);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x45 = ((get(ctx, arg0, 38, 2) + get(ctx, arg0, 198, 2)) + get(ctx, arg0, 358, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x46 = bitAnd(((x45 + get(ctx, arg0, 518, 2)) + get(ctx, arg0, 678, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 38, x46);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x47 = ((get(ctx, arg0, 39, 2) + get(ctx, arg0, 199, 2)) + get(ctx, arg0, 359, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x48 = bitAnd(((x47 + get(ctx, arg0, 519, 2)) + get(ctx, arg0, 679, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 39, x48);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x49 = ((get(ctx, arg0, 40, 2) + get(ctx, arg0, 200, 2)) + get(ctx, arg0, 360, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x50 = bitAnd(((x49 + get(ctx, arg0, 520, 2)) + get(ctx, arg0, 680, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 40, x50);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x51 = ((get(ctx, arg0, 41, 2) + get(ctx, arg0, 201, 2)) + get(ctx, arg0, 361, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x52 = bitAnd(((x51 + get(ctx, arg0, 521, 2)) + get(ctx, arg0, 681, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 41, x52);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x53 = ((get(ctx, arg0, 42, 2) + get(ctx, arg0, 202, 2)) + get(ctx, arg0, 362, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x54 = bitAnd(((x53 + get(ctx, arg0, 522, 2)) + get(ctx, arg0, 682, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 42, x54);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x55 = ((get(ctx, arg0, 43, 2) + get(ctx, arg0, 203, 2)) + get(ctx, arg0, 363, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x56 = bitAnd(((x55 + get(ctx, arg0, 523, 2)) + get(ctx, arg0, 683, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 43, x56);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x57 = ((get(ctx, arg0, 44, 2) + get(ctx, arg0, 204, 2)) + get(ctx, arg0, 364, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x58 = bitAnd(((x57 + get(ctx, arg0, 524, 2)) + get(ctx, arg0, 684, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 44, x58);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x59 = ((get(ctx, arg0, 45, 2) + get(ctx, arg0, 205, 2)) + get(ctx, arg0, 365, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x60 = bitAnd(((x59 + get(ctx, arg0, 525, 2)) + get(ctx, arg0, 685, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 45, x60);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x61 = ((get(ctx, arg0, 46, 2) + get(ctx, arg0, 206, 2)) + get(ctx, arg0, 366, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x62 = bitAnd(((x61 + get(ctx, arg0, 526, 2)) + get(ctx, arg0, 686, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 46, x62);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x63 = ((get(ctx, arg0, 47, 2) + get(ctx, arg0, 207, 2)) + get(ctx, arg0, 367, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x64 = bitAnd(((x63 + get(ctx, arg0, 527, 2)) + get(ctx, arg0, 687, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 47, x64);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x65 = ((get(ctx, arg0, 16, 1) + get(ctx, arg0, 176, 1)) + get(ctx, arg0, 336, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x66 = bitAnd(((x65 + get(ctx, arg0, 496, 1)) + get(ctx, arg0, 656, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 48, x66);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x67 = ((get(ctx, arg0, 17, 1) + get(ctx, arg0, 177, 1)) + get(ctx, arg0, 337, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x68 = bitAnd(((x67 + get(ctx, arg0, 497, 1)) + get(ctx, arg0, 657, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 49, x68);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x69 = ((get(ctx, arg0, 18, 1) + get(ctx, arg0, 178, 1)) + get(ctx, arg0, 338, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x70 = bitAnd(((x69 + get(ctx, arg0, 498, 1)) + get(ctx, arg0, 658, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 50, x70);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x71 = ((get(ctx, arg0, 19, 1) + get(ctx, arg0, 179, 1)) + get(ctx, arg0, 339, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x72 = bitAnd(((x71 + get(ctx, arg0, 499, 1)) + get(ctx, arg0, 659, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 51, x72);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x73 = ((get(ctx, arg0, 20, 1) + get(ctx, arg0, 180, 1)) + get(ctx, arg0, 340, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x74 = bitAnd(((x73 + get(ctx, arg0, 500, 1)) + get(ctx, arg0, 660, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 52, x74);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x75 = ((get(ctx, arg0, 21, 1) + get(ctx, arg0, 181, 1)) + get(ctx, arg0, 341, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x76 = bitAnd(((x75 + get(ctx, arg0, 501, 1)) + get(ctx, arg0, 661, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 53, x76);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x77 = ((get(ctx, arg0, 22, 1) + get(ctx, arg0, 182, 1)) + get(ctx, arg0, 342, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x78 = bitAnd(((x77 + get(ctx, arg0, 502, 1)) + get(ctx, arg0, 662, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 54, x78);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x79 = ((get(ctx, arg0, 23, 1) + get(ctx, arg0, 183, 1)) + get(ctx, arg0, 343, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x80 = bitAnd(((x79 + get(ctx, arg0, 503, 1)) + get(ctx, arg0, 663, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 55, x80);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x81 = ((get(ctx, arg0, 24, 1) + get(ctx, arg0, 184, 1)) + get(ctx, arg0, 344, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x82 = bitAnd(((x81 + get(ctx, arg0, 504, 1)) + get(ctx, arg0, 664, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 56, x82);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x83 = ((get(ctx, arg0, 25, 1) + get(ctx, arg0, 185, 1)) + get(ctx, arg0, 345, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x84 = bitAnd(((x83 + get(ctx, arg0, 505, 1)) + get(ctx, arg0, 665, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 57, x84);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x85 = ((get(ctx, arg0, 26, 1) + get(ctx, arg0, 186, 1)) + get(ctx, arg0, 346, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x86 = bitAnd(((x85 + get(ctx, arg0, 506, 1)) + get(ctx, arg0, 666, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 58, x86);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x87 = ((get(ctx, arg0, 27, 1) + get(ctx, arg0, 187, 1)) + get(ctx, arg0, 347, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x88 = bitAnd(((x87 + get(ctx, arg0, 507, 1)) + get(ctx, arg0, 667, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 59, x88);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x89 = ((get(ctx, arg0, 28, 1) + get(ctx, arg0, 188, 1)) + get(ctx, arg0, 348, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x90 = bitAnd(((x89 + get(ctx, arg0, 508, 1)) + get(ctx, arg0, 668, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 60, x90);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x91 = ((get(ctx, arg0, 29, 1) + get(ctx, arg0, 189, 1)) + get(ctx, arg0, 349, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x92 = bitAnd(((x91 + get(ctx, arg0, 509, 1)) + get(ctx, arg0, 669, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 61, x92);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x93 = ((get(ctx, arg0, 30, 1) + get(ctx, arg0, 190, 1)) + get(ctx, arg0, 350, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x94 = bitAnd(((x93 + get(ctx, arg0, 510, 1)) + get(ctx, arg0, 670, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 62, x94);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x95 = ((get(ctx, arg0, 31, 1) + get(ctx, arg0, 191, 1)) + get(ctx, arg0, 351, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x96 = bitAnd(((x95 + get(ctx, arg0, 511, 1)) + get(ctx, arg0, 671, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 63, x96);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x97 = ((get(ctx, arg0, 32, 1) + get(ctx, arg0, 192, 1)) + get(ctx, arg0, 352, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x98 = bitAnd(((x97 + get(ctx, arg0, 512, 1)) + get(ctx, arg0, 672, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 64, x98);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x99 = ((get(ctx, arg0, 33, 1) + get(ctx, arg0, 193, 1)) + get(ctx, arg0, 353, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x100 = bitAnd(((x99 + get(ctx, arg0, 513, 1)) + get(ctx, arg0, 673, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 65, x100);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x101 = ((get(ctx, arg0, 34, 1) + get(ctx, arg0, 194, 1)) + get(ctx, arg0, 354, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x102 = bitAnd(((x101 + get(ctx, arg0, 514, 1)) + get(ctx, arg0, 674, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 66, x102);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x103 = ((get(ctx, arg0, 35, 1) + get(ctx, arg0, 195, 1)) + get(ctx, arg0, 355, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x104 = bitAnd(((x103 + get(ctx, arg0, 515, 1)) + get(ctx, arg0, 675, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 67, x104);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x105 = ((get(ctx, arg0, 36, 1) + get(ctx, arg0, 196, 1)) + get(ctx, arg0, 356, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x106 = bitAnd(((x105 + get(ctx, arg0, 516, 1)) + get(ctx, arg0, 676, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 68, x106);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x107 = ((get(ctx, arg0, 37, 1) + get(ctx, arg0, 197, 1)) + get(ctx, arg0, 357, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x108 = bitAnd(((x107 + get(ctx, arg0, 517, 1)) + get(ctx, arg0, 677, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 69, x108);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x109 = ((get(ctx, arg0, 38, 1) + get(ctx, arg0, 198, 1)) + get(ctx, arg0, 358, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x110 = bitAnd(((x109 + get(ctx, arg0, 518, 1)) + get(ctx, arg0, 678, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 70, x110);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x111 = ((get(ctx, arg0, 39, 1) + get(ctx, arg0, 199, 1)) + get(ctx, arg0, 359, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x112 = bitAnd(((x111 + get(ctx, arg0, 519, 1)) + get(ctx, arg0, 679, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 71, x112);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x113 = ((get(ctx, arg0, 40, 1) + get(ctx, arg0, 200, 1)) + get(ctx, arg0, 360, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x114 = bitAnd(((x113 + get(ctx, arg0, 520, 1)) + get(ctx, arg0, 680, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 72, x114);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x115 = ((get(ctx, arg0, 41, 1) + get(ctx, arg0, 201, 1)) + get(ctx, arg0, 361, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x116 = bitAnd(((x115 + get(ctx, arg0, 521, 1)) + get(ctx, arg0, 681, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 73, x116);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x117 = ((get(ctx, arg0, 42, 1) + get(ctx, arg0, 202, 1)) + get(ctx, arg0, 362, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x118 = bitAnd(((x117 + get(ctx, arg0, 522, 1)) + get(ctx, arg0, 682, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 74, x118);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x119 = ((get(ctx, arg0, 43, 1) + get(ctx, arg0, 203, 1)) + get(ctx, arg0, 363, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x120 = bitAnd(((x119 + get(ctx, arg0, 523, 1)) + get(ctx, arg0, 683, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 75, x120);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x121 = ((get(ctx, arg0, 44, 1) + get(ctx, arg0, 204, 1)) + get(ctx, arg0, 364, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x122 = bitAnd(((x121 + get(ctx, arg0, 524, 1)) + get(ctx, arg0, 684, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 76, x122);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x123 = ((get(ctx, arg0, 45, 1) + get(ctx, arg0, 205, 1)) + get(ctx, arg0, 365, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x124 = bitAnd(((x123 + get(ctx, arg0, 525, 1)) + get(ctx, arg0, 685, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 77, x124);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x125 = ((get(ctx, arg0, 46, 1) + get(ctx, arg0, 206, 1)) + get(ctx, arg0, 366, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x126 = bitAnd(((x125 + get(ctx, arg0, 526, 1)) + get(ctx, arg0, 686, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 78, x126);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x127 = ((get(ctx, arg0, 47, 1) + get(ctx, arg0, 207, 1)) + get(ctx, arg0, 367, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x128 = bitAnd(((x127 + get(ctx, arg0, 527, 1)) + get(ctx, arg0, 687, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 79, x128);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x129 = ((get(ctx, arg0, 48, 2) + get(ctx, arg0, 208, 2)) + get(ctx, arg0, 368, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x130 = bitAnd(((x129 + get(ctx, arg0, 528, 2)) + get(ctx, arg0, 688, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 80, x130);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x131 = ((get(ctx, arg0, 49, 2) + get(ctx, arg0, 209, 2)) + get(ctx, arg0, 369, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x132 = bitAnd(((x131 + get(ctx, arg0, 529, 2)) + get(ctx, arg0, 689, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 81, x132);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x133 = ((get(ctx, arg0, 50, 2) + get(ctx, arg0, 210, 2)) + get(ctx, arg0, 370, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x134 = bitAnd(((x133 + get(ctx, arg0, 530, 2)) + get(ctx, arg0, 690, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 82, x134);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x135 = ((get(ctx, arg0, 51, 2) + get(ctx, arg0, 211, 2)) + get(ctx, arg0, 371, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x136 = bitAnd(((x135 + get(ctx, arg0, 531, 2)) + get(ctx, arg0, 691, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 83, x136);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x137 = ((get(ctx, arg0, 52, 2) + get(ctx, arg0, 212, 2)) + get(ctx, arg0, 372, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x138 = bitAnd(((x137 + get(ctx, arg0, 532, 2)) + get(ctx, arg0, 692, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 84, x138);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x139 = ((get(ctx, arg0, 53, 2) + get(ctx, arg0, 213, 2)) + get(ctx, arg0, 373, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x140 = bitAnd(((x139 + get(ctx, arg0, 533, 2)) + get(ctx, arg0, 693, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 85, x140);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x141 = ((get(ctx, arg0, 54, 2) + get(ctx, arg0, 214, 2)) + get(ctx, arg0, 374, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x142 = bitAnd(((x141 + get(ctx, arg0, 534, 2)) + get(ctx, arg0, 694, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 86, x142);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x143 = ((get(ctx, arg0, 55, 2) + get(ctx, arg0, 215, 2)) + get(ctx, arg0, 375, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x144 = bitAnd(((x143 + get(ctx, arg0, 535, 2)) + get(ctx, arg0, 695, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 87, x144);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x145 = ((get(ctx, arg0, 56, 2) + get(ctx, arg0, 216, 2)) + get(ctx, arg0, 376, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x146 = bitAnd(((x145 + get(ctx, arg0, 536, 2)) + get(ctx, arg0, 696, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 88, x146);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x147 = ((get(ctx, arg0, 57, 2) + get(ctx, arg0, 217, 2)) + get(ctx, arg0, 377, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x148 = bitAnd(((x147 + get(ctx, arg0, 537, 2)) + get(ctx, arg0, 697, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 89, x148);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x149 = ((get(ctx, arg0, 58, 2) + get(ctx, arg0, 218, 2)) + get(ctx, arg0, 378, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x150 = bitAnd(((x149 + get(ctx, arg0, 538, 2)) + get(ctx, arg0, 698, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 90, x150);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x151 = ((get(ctx, arg0, 59, 2) + get(ctx, arg0, 219, 2)) + get(ctx, arg0, 379, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x152 = bitAnd(((x151 + get(ctx, arg0, 539, 2)) + get(ctx, arg0, 699, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 91, x152);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x153 = ((get(ctx, arg0, 60, 2) + get(ctx, arg0, 220, 2)) + get(ctx, arg0, 380, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x154 = bitAnd(((x153 + get(ctx, arg0, 540, 2)) + get(ctx, arg0, 700, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 92, x154);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x155 = ((get(ctx, arg0, 61, 2) + get(ctx, arg0, 221, 2)) + get(ctx, arg0, 381, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x156 = bitAnd(((x155 + get(ctx, arg0, 541, 2)) + get(ctx, arg0, 701, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 93, x156);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x157 = ((get(ctx, arg0, 62, 2) + get(ctx, arg0, 222, 2)) + get(ctx, arg0, 382, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x158 = bitAnd(((x157 + get(ctx, arg0, 542, 2)) + get(ctx, arg0, 702, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 94, x158);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x159 = ((get(ctx, arg0, 63, 2) + get(ctx, arg0, 223, 2)) + get(ctx, arg0, 383, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x160 = bitAnd(((x159 + get(ctx, arg0, 543, 2)) + get(ctx, arg0, 703, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 95, x160);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x161 = ((get(ctx, arg0, 64, 2) + get(ctx, arg0, 224, 2)) + get(ctx, arg0, 384, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x162 = bitAnd(((x161 + get(ctx, arg0, 544, 2)) + get(ctx, arg0, 704, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 96, x162);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x163 = ((get(ctx, arg0, 65, 2) + get(ctx, arg0, 225, 2)) + get(ctx, arg0, 385, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x164 = bitAnd(((x163 + get(ctx, arg0, 545, 2)) + get(ctx, arg0, 705, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 97, x164);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x165 = ((get(ctx, arg0, 66, 2) + get(ctx, arg0, 226, 2)) + get(ctx, arg0, 386, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x166 = bitAnd(((x165 + get(ctx, arg0, 546, 2)) + get(ctx, arg0, 706, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 98, x166);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x167 = ((get(ctx, arg0, 67, 2) + get(ctx, arg0, 227, 2)) + get(ctx, arg0, 387, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x168 = bitAnd(((x167 + get(ctx, arg0, 547, 2)) + get(ctx, arg0, 707, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 99, x168);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x169 = ((get(ctx, arg0, 68, 2) + get(ctx, arg0, 228, 2)) + get(ctx, arg0, 388, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x170 = bitAnd(((x169 + get(ctx, arg0, 548, 2)) + get(ctx, arg0, 708, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 100, x170);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x171 = ((get(ctx, arg0, 69, 2) + get(ctx, arg0, 229, 2)) + get(ctx, arg0, 389, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x172 = bitAnd(((x171 + get(ctx, arg0, 549, 2)) + get(ctx, arg0, 709, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 101, x172);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x173 = ((get(ctx, arg0, 70, 2) + get(ctx, arg0, 230, 2)) + get(ctx, arg0, 390, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x174 = bitAnd(((x173 + get(ctx, arg0, 550, 2)) + get(ctx, arg0, 710, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 102, x174);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x175 = ((get(ctx, arg0, 71, 2) + get(ctx, arg0, 231, 2)) + get(ctx, arg0, 391, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x176 = bitAnd(((x175 + get(ctx, arg0, 551, 2)) + get(ctx, arg0, 711, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 103, x176);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x177 = ((get(ctx, arg0, 72, 2) + get(ctx, arg0, 232, 2)) + get(ctx, arg0, 392, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x178 = bitAnd(((x177 + get(ctx, arg0, 552, 2)) + get(ctx, arg0, 712, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 104, x178);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x179 = ((get(ctx, arg0, 73, 2) + get(ctx, arg0, 233, 2)) + get(ctx, arg0, 393, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x180 = bitAnd(((x179 + get(ctx, arg0, 553, 2)) + get(ctx, arg0, 713, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 105, x180);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x181 = ((get(ctx, arg0, 74, 2) + get(ctx, arg0, 234, 2)) + get(ctx, arg0, 394, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x182 = bitAnd(((x181 + get(ctx, arg0, 554, 2)) + get(ctx, arg0, 714, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 106, x182);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x183 = ((get(ctx, arg0, 75, 2) + get(ctx, arg0, 235, 2)) + get(ctx, arg0, 395, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x184 = bitAnd(((x183 + get(ctx, arg0, 555, 2)) + get(ctx, arg0, 715, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 107, x184);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x185 = ((get(ctx, arg0, 76, 2) + get(ctx, arg0, 236, 2)) + get(ctx, arg0, 396, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x186 = bitAnd(((x185 + get(ctx, arg0, 556, 2)) + get(ctx, arg0, 716, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 108, x186);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x187 = ((get(ctx, arg0, 77, 2) + get(ctx, arg0, 237, 2)) + get(ctx, arg0, 397, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x188 = bitAnd(((x187 + get(ctx, arg0, 557, 2)) + get(ctx, arg0, 717, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 109, x188);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x189 = ((get(ctx, arg0, 78, 2) + get(ctx, arg0, 238, 2)) + get(ctx, arg0, 398, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x190 = bitAnd(((x189 + get(ctx, arg0, 558, 2)) + get(ctx, arg0, 718, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 110, x190);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x191 = ((get(ctx, arg0, 79, 2) + get(ctx, arg0, 239, 2)) + get(ctx, arg0, 399, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x192 = bitAnd(((x191 + get(ctx, arg0, 559, 2)) + get(ctx, arg0, 719, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 111, x192);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x193 = ((get(ctx, arg0, 48, 1) + get(ctx, arg0, 208, 1)) + get(ctx, arg0, 368, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x194 = bitAnd(((x193 + get(ctx, arg0, 528, 1)) + get(ctx, arg0, 688, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 112, x194);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x195 = ((get(ctx, arg0, 49, 1) + get(ctx, arg0, 209, 1)) + get(ctx, arg0, 369, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x196 = bitAnd(((x195 + get(ctx, arg0, 529, 1)) + get(ctx, arg0, 689, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 113, x196);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x197 = ((get(ctx, arg0, 50, 1) + get(ctx, arg0, 210, 1)) + get(ctx, arg0, 370, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x198 = bitAnd(((x197 + get(ctx, arg0, 530, 1)) + get(ctx, arg0, 690, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 114, x198);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x199 = ((get(ctx, arg0, 51, 1) + get(ctx, arg0, 211, 1)) + get(ctx, arg0, 371, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x200 = bitAnd(((x199 + get(ctx, arg0, 531, 1)) + get(ctx, arg0, 691, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 115, x200);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x201 = ((get(ctx, arg0, 52, 1) + get(ctx, arg0, 212, 1)) + get(ctx, arg0, 372, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x202 = bitAnd(((x201 + get(ctx, arg0, 532, 1)) + get(ctx, arg0, 692, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 116, x202);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x203 = ((get(ctx, arg0, 53, 1) + get(ctx, arg0, 213, 1)) + get(ctx, arg0, 373, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x204 = bitAnd(((x203 + get(ctx, arg0, 533, 1)) + get(ctx, arg0, 693, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 117, x204);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x205 = ((get(ctx, arg0, 54, 1) + get(ctx, arg0, 214, 1)) + get(ctx, arg0, 374, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x206 = bitAnd(((x205 + get(ctx, arg0, 534, 1)) + get(ctx, arg0, 694, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 118, x206);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x207 = ((get(ctx, arg0, 55, 1) + get(ctx, arg0, 215, 1)) + get(ctx, arg0, 375, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x208 = bitAnd(((x207 + get(ctx, arg0, 535, 1)) + get(ctx, arg0, 695, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 119, x208);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x209 = ((get(ctx, arg0, 56, 1) + get(ctx, arg0, 216, 1)) + get(ctx, arg0, 376, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x210 = bitAnd(((x209 + get(ctx, arg0, 536, 1)) + get(ctx, arg0, 696, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 120, x210);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x211 = ((get(ctx, arg0, 57, 1) + get(ctx, arg0, 217, 1)) + get(ctx, arg0, 377, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x212 = bitAnd(((x211 + get(ctx, arg0, 537, 1)) + get(ctx, arg0, 697, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 121, x212);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x213 = ((get(ctx, arg0, 58, 1) + get(ctx, arg0, 218, 1)) + get(ctx, arg0, 378, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x214 = bitAnd(((x213 + get(ctx, arg0, 538, 1)) + get(ctx, arg0, 698, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 122, x214);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x215 = ((get(ctx, arg0, 59, 1) + get(ctx, arg0, 219, 1)) + get(ctx, arg0, 379, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x216 = bitAnd(((x215 + get(ctx, arg0, 539, 1)) + get(ctx, arg0, 699, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 123, x216);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x217 = ((get(ctx, arg0, 60, 1) + get(ctx, arg0, 220, 1)) + get(ctx, arg0, 380, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x218 = bitAnd(((x217 + get(ctx, arg0, 540, 1)) + get(ctx, arg0, 700, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 124, x218);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x219 = ((get(ctx, arg0, 61, 1) + get(ctx, arg0, 221, 1)) + get(ctx, arg0, 381, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x220 = bitAnd(((x219 + get(ctx, arg0, 541, 1)) + get(ctx, arg0, 701, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 125, x220);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x221 = ((get(ctx, arg0, 62, 1) + get(ctx, arg0, 222, 1)) + get(ctx, arg0, 382, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x222 = bitAnd(((x221 + get(ctx, arg0, 542, 1)) + get(ctx, arg0, 702, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 126, x222);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x223 = ((get(ctx, arg0, 63, 1) + get(ctx, arg0, 223, 1)) + get(ctx, arg0, 383, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x224 = bitAnd(((x223 + get(ctx, arg0, 543, 1)) + get(ctx, arg0, 703, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 127, x224);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x225 = ((get(ctx, arg0, 64, 1) + get(ctx, arg0, 224, 1)) + get(ctx, arg0, 384, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x226 = bitAnd(((x225 + get(ctx, arg0, 544, 1)) + get(ctx, arg0, 704, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 128, x226);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x227 = ((get(ctx, arg0, 65, 1) + get(ctx, arg0, 225, 1)) + get(ctx, arg0, 385, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x228 = bitAnd(((x227 + get(ctx, arg0, 545, 1)) + get(ctx, arg0, 705, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 129, x228);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x229 = ((get(ctx, arg0, 66, 1) + get(ctx, arg0, 226, 1)) + get(ctx, arg0, 386, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x230 = bitAnd(((x229 + get(ctx, arg0, 546, 1)) + get(ctx, arg0, 706, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 130, x230);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x231 = ((get(ctx, arg0, 67, 1) + get(ctx, arg0, 227, 1)) + get(ctx, arg0, 387, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x232 = bitAnd(((x231 + get(ctx, arg0, 547, 1)) + get(ctx, arg0, 707, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 131, x232);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x233 = ((get(ctx, arg0, 68, 1) + get(ctx, arg0, 228, 1)) + get(ctx, arg0, 388, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x234 = bitAnd(((x233 + get(ctx, arg0, 548, 1)) + get(ctx, arg0, 708, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 132, x234);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x235 = ((get(ctx, arg0, 69, 1) + get(ctx, arg0, 229, 1)) + get(ctx, arg0, 389, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x236 = bitAnd(((x235 + get(ctx, arg0, 549, 1)) + get(ctx, arg0, 709, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 133, x236);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x237 = ((get(ctx, arg0, 70, 1) + get(ctx, arg0, 230, 1)) + get(ctx, arg0, 390, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x238 = bitAnd(((x237 + get(ctx, arg0, 550, 1)) + get(ctx, arg0, 710, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 134, x238);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x239 = ((get(ctx, arg0, 71, 1) + get(ctx, arg0, 231, 1)) + get(ctx, arg0, 391, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x240 = bitAnd(((x239 + get(ctx, arg0, 551, 1)) + get(ctx, arg0, 711, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 135, x240);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x241 = ((get(ctx, arg0, 72, 1) + get(ctx, arg0, 232, 1)) + get(ctx, arg0, 392, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x242 = bitAnd(((x241 + get(ctx, arg0, 552, 1)) + get(ctx, arg0, 712, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 136, x242);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x243 = ((get(ctx, arg0, 73, 1) + get(ctx, arg0, 233, 1)) + get(ctx, arg0, 393, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x244 = bitAnd(((x243 + get(ctx, arg0, 553, 1)) + get(ctx, arg0, 713, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 137, x244);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x245 = ((get(ctx, arg0, 74, 1) + get(ctx, arg0, 234, 1)) + get(ctx, arg0, 394, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x246 = bitAnd(((x245 + get(ctx, arg0, 554, 1)) + get(ctx, arg0, 714, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 138, x246);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x247 = ((get(ctx, arg0, 75, 1) + get(ctx, arg0, 235, 1)) + get(ctx, arg0, 395, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x248 = bitAnd(((x247 + get(ctx, arg0, 555, 1)) + get(ctx, arg0, 715, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 139, x248);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x249 = ((get(ctx, arg0, 76, 1) + get(ctx, arg0, 236, 1)) + get(ctx, arg0, 396, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x250 = bitAnd(((x249 + get(ctx, arg0, 556, 1)) + get(ctx, arg0, 716, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 140, x250);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x251 = ((get(ctx, arg0, 77, 1) + get(ctx, arg0, 237, 1)) + get(ctx, arg0, 397, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x252 = bitAnd(((x251 + get(ctx, arg0, 557, 1)) + get(ctx, arg0, 717, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 141, x252);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x253 = ((get(ctx, arg0, 78, 1) + get(ctx, arg0, 238, 1)) + get(ctx, arg0, 398, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x254 = bitAnd(((x253 + get(ctx, arg0, 558, 1)) + get(ctx, arg0, 718, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 142, x254);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x255 = ((get(ctx, arg0, 79, 1) + get(ctx, arg0, 239, 1)) + get(ctx, arg0, 399, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x256 = bitAnd(((x255 + get(ctx, arg0, 559, 1)) + get(ctx, arg0, 719, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 143, x256);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x257 = ((get(ctx, arg0, 80, 2) + get(ctx, arg0, 240, 2)) + get(ctx, arg0, 400, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x258 = bitAnd(((x257 + get(ctx, arg0, 560, 2)) + get(ctx, arg0, 720, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 144, x258);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x259 = ((get(ctx, arg0, 81, 2) + get(ctx, arg0, 241, 2)) + get(ctx, arg0, 401, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x260 = bitAnd(((x259 + get(ctx, arg0, 561, 2)) + get(ctx, arg0, 721, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 145, x260);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x261 = ((get(ctx, arg0, 82, 2) + get(ctx, arg0, 242, 2)) + get(ctx, arg0, 402, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x262 = bitAnd(((x261 + get(ctx, arg0, 562, 2)) + get(ctx, arg0, 722, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 146, x262);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x263 = ((get(ctx, arg0, 83, 2) + get(ctx, arg0, 243, 2)) + get(ctx, arg0, 403, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x264 = bitAnd(((x263 + get(ctx, arg0, 563, 2)) + get(ctx, arg0, 723, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 147, x264);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x265 = ((get(ctx, arg0, 84, 2) + get(ctx, arg0, 244, 2)) + get(ctx, arg0, 404, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x266 = bitAnd(((x265 + get(ctx, arg0, 564, 2)) + get(ctx, arg0, 724, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 148, x266);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x267 = ((get(ctx, arg0, 85, 2) + get(ctx, arg0, 245, 2)) + get(ctx, arg0, 405, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x268 = bitAnd(((x267 + get(ctx, arg0, 565, 2)) + get(ctx, arg0, 725, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 149, x268);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x269 = ((get(ctx, arg0, 86, 2) + get(ctx, arg0, 246, 2)) + get(ctx, arg0, 406, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x270 = bitAnd(((x269 + get(ctx, arg0, 566, 2)) + get(ctx, arg0, 726, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 150, x270);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x271 = ((get(ctx, arg0, 87, 2) + get(ctx, arg0, 247, 2)) + get(ctx, arg0, 407, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x272 = bitAnd(((x271 + get(ctx, arg0, 567, 2)) + get(ctx, arg0, 727, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 151, x272);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x273 = ((get(ctx, arg0, 88, 2) + get(ctx, arg0, 248, 2)) + get(ctx, arg0, 408, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x274 = bitAnd(((x273 + get(ctx, arg0, 568, 2)) + get(ctx, arg0, 728, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 152, x274);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x275 = ((get(ctx, arg0, 89, 2) + get(ctx, arg0, 249, 2)) + get(ctx, arg0, 409, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x276 = bitAnd(((x275 + get(ctx, arg0, 569, 2)) + get(ctx, arg0, 729, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 153, x276);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x277 = ((get(ctx, arg0, 90, 2) + get(ctx, arg0, 250, 2)) + get(ctx, arg0, 410, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x278 = bitAnd(((x277 + get(ctx, arg0, 570, 2)) + get(ctx, arg0, 730, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 154, x278);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x279 = ((get(ctx, arg0, 91, 2) + get(ctx, arg0, 251, 2)) + get(ctx, arg0, 411, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x280 = bitAnd(((x279 + get(ctx, arg0, 571, 2)) + get(ctx, arg0, 731, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 155, x280);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x281 = ((get(ctx, arg0, 92, 2) + get(ctx, arg0, 252, 2)) + get(ctx, arg0, 412, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x282 = bitAnd(((x281 + get(ctx, arg0, 572, 2)) + get(ctx, arg0, 732, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 156, x282);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x283 = ((get(ctx, arg0, 93, 2) + get(ctx, arg0, 253, 2)) + get(ctx, arg0, 413, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x284 = bitAnd(((x283 + get(ctx, arg0, 573, 2)) + get(ctx, arg0, 733, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 157, x284);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x285 = ((get(ctx, arg0, 94, 2) + get(ctx, arg0, 254, 2)) + get(ctx, arg0, 414, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x286 = bitAnd(((x285 + get(ctx, arg0, 574, 2)) + get(ctx, arg0, 734, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 158, x286);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x287 = ((get(ctx, arg0, 95, 2) + get(ctx, arg0, 255, 2)) + get(ctx, arg0, 415, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x288 = bitAnd(((x287 + get(ctx, arg0, 575, 2)) + get(ctx, arg0, 735, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 159, x288);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x289 = ((get(ctx, arg0, 96, 2) + get(ctx, arg0, 256, 2)) + get(ctx, arg0, 416, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x290 = bitAnd(((x289 + get(ctx, arg0, 576, 2)) + get(ctx, arg0, 736, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 160, x290);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x291 = ((get(ctx, arg0, 97, 2) + get(ctx, arg0, 257, 2)) + get(ctx, arg0, 417, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x292 = bitAnd(((x291 + get(ctx, arg0, 577, 2)) + get(ctx, arg0, 737, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 161, x292);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x293 = ((get(ctx, arg0, 98, 2) + get(ctx, arg0, 258, 2)) + get(ctx, arg0, 418, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x294 = bitAnd(((x293 + get(ctx, arg0, 578, 2)) + get(ctx, arg0, 738, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 162, x294);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x295 = ((get(ctx, arg0, 99, 2) + get(ctx, arg0, 259, 2)) + get(ctx, arg0, 419, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x296 = bitAnd(((x295 + get(ctx, arg0, 579, 2)) + get(ctx, arg0, 739, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 163, x296);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x297 = ((get(ctx, arg0, 100, 2) + get(ctx, arg0, 260, 2)) + get(ctx, arg0, 420, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x298 = bitAnd(((x297 + get(ctx, arg0, 580, 2)) + get(ctx, arg0, 740, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 164, x298);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x299 = ((get(ctx, arg0, 101, 2) + get(ctx, arg0, 261, 2)) + get(ctx, arg0, 421, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x300 = bitAnd(((x299 + get(ctx, arg0, 581, 2)) + get(ctx, arg0, 741, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 165, x300);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x301 = ((get(ctx, arg0, 102, 2) + get(ctx, arg0, 262, 2)) + get(ctx, arg0, 422, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x302 = bitAnd(((x301 + get(ctx, arg0, 582, 2)) + get(ctx, arg0, 742, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 166, x302);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x303 = ((get(ctx, arg0, 103, 2) + get(ctx, arg0, 263, 2)) + get(ctx, arg0, 423, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x304 = bitAnd(((x303 + get(ctx, arg0, 583, 2)) + get(ctx, arg0, 743, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 167, x304);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x305 = ((get(ctx, arg0, 104, 2) + get(ctx, arg0, 264, 2)) + get(ctx, arg0, 424, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x306 = bitAnd(((x305 + get(ctx, arg0, 584, 2)) + get(ctx, arg0, 744, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 168, x306);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x307 = ((get(ctx, arg0, 105, 2) + get(ctx, arg0, 265, 2)) + get(ctx, arg0, 425, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x308 = bitAnd(((x307 + get(ctx, arg0, 585, 2)) + get(ctx, arg0, 745, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 169, x308);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x309 = ((get(ctx, arg0, 106, 2) + get(ctx, arg0, 266, 2)) + get(ctx, arg0, 426, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x310 = bitAnd(((x309 + get(ctx, arg0, 586, 2)) + get(ctx, arg0, 746, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 170, x310);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x311 = ((get(ctx, arg0, 107, 2) + get(ctx, arg0, 267, 2)) + get(ctx, arg0, 427, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x312 = bitAnd(((x311 + get(ctx, arg0, 587, 2)) + get(ctx, arg0, 747, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 171, x312);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x313 = ((get(ctx, arg0, 108, 2) + get(ctx, arg0, 268, 2)) + get(ctx, arg0, 428, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x314 = bitAnd(((x313 + get(ctx, arg0, 588, 2)) + get(ctx, arg0, 748, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 172, x314);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x315 = ((get(ctx, arg0, 109, 2) + get(ctx, arg0, 269, 2)) + get(ctx, arg0, 429, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x316 = bitAnd(((x315 + get(ctx, arg0, 589, 2)) + get(ctx, arg0, 749, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 173, x316);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x317 = ((get(ctx, arg0, 110, 2) + get(ctx, arg0, 270, 2)) + get(ctx, arg0, 430, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x318 = bitAnd(((x317 + get(ctx, arg0, 590, 2)) + get(ctx, arg0, 750, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 174, x318);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x319 = ((get(ctx, arg0, 111, 2) + get(ctx, arg0, 271, 2)) + get(ctx, arg0, 431, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x320 = bitAnd(((x319 + get(ctx, arg0, 591, 2)) + get(ctx, arg0, 751, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 175, x320);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x321 = ((get(ctx, arg0, 80, 1) + get(ctx, arg0, 240, 1)) + get(ctx, arg0, 400, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x322 = bitAnd(((x321 + get(ctx, arg0, 560, 1)) + get(ctx, arg0, 720, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 176, x322);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x323 = ((get(ctx, arg0, 81, 1) + get(ctx, arg0, 241, 1)) + get(ctx, arg0, 401, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x324 = bitAnd(((x323 + get(ctx, arg0, 561, 1)) + get(ctx, arg0, 721, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 177, x324);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x325 = ((get(ctx, arg0, 82, 1) + get(ctx, arg0, 242, 1)) + get(ctx, arg0, 402, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x326 = bitAnd(((x325 + get(ctx, arg0, 562, 1)) + get(ctx, arg0, 722, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 178, x326);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x327 = ((get(ctx, arg0, 83, 1) + get(ctx, arg0, 243, 1)) + get(ctx, arg0, 403, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x328 = bitAnd(((x327 + get(ctx, arg0, 563, 1)) + get(ctx, arg0, 723, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 179, x328);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x329 = ((get(ctx, arg0, 84, 1) + get(ctx, arg0, 244, 1)) + get(ctx, arg0, 404, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x330 = bitAnd(((x329 + get(ctx, arg0, 564, 1)) + get(ctx, arg0, 724, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 180, x330);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x331 = ((get(ctx, arg0, 85, 1) + get(ctx, arg0, 245, 1)) + get(ctx, arg0, 405, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x332 = bitAnd(((x331 + get(ctx, arg0, 565, 1)) + get(ctx, arg0, 725, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 181, x332);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x333 = ((get(ctx, arg0, 86, 1) + get(ctx, arg0, 246, 1)) + get(ctx, arg0, 406, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x334 = bitAnd(((x333 + get(ctx, arg0, 566, 1)) + get(ctx, arg0, 726, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 182, x334);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x335 = ((get(ctx, arg0, 87, 1) + get(ctx, arg0, 247, 1)) + get(ctx, arg0, 407, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x336 = bitAnd(((x335 + get(ctx, arg0, 567, 1)) + get(ctx, arg0, 727, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 183, x336);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x337 = ((get(ctx, arg0, 88, 1) + get(ctx, arg0, 248, 1)) + get(ctx, arg0, 408, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x338 = bitAnd(((x337 + get(ctx, arg0, 568, 1)) + get(ctx, arg0, 728, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 184, x338);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x339 = ((get(ctx, arg0, 89, 1) + get(ctx, arg0, 249, 1)) + get(ctx, arg0, 409, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x340 = bitAnd(((x339 + get(ctx, arg0, 569, 1)) + get(ctx, arg0, 729, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 185, x340);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x341 = ((get(ctx, arg0, 90, 1) + get(ctx, arg0, 250, 1)) + get(ctx, arg0, 410, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x342 = bitAnd(((x341 + get(ctx, arg0, 570, 1)) + get(ctx, arg0, 730, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 186, x342);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x343 = ((get(ctx, arg0, 91, 1) + get(ctx, arg0, 251, 1)) + get(ctx, arg0, 411, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x344 = bitAnd(((x343 + get(ctx, arg0, 571, 1)) + get(ctx, arg0, 731, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 187, x344);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x345 = ((get(ctx, arg0, 92, 1) + get(ctx, arg0, 252, 1)) + get(ctx, arg0, 412, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x346 = bitAnd(((x345 + get(ctx, arg0, 572, 1)) + get(ctx, arg0, 732, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 188, x346);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x347 = ((get(ctx, arg0, 93, 1) + get(ctx, arg0, 253, 1)) + get(ctx, arg0, 413, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x348 = bitAnd(((x347 + get(ctx, arg0, 573, 1)) + get(ctx, arg0, 733, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 189, x348);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x349 = ((get(ctx, arg0, 94, 1) + get(ctx, arg0, 254, 1)) + get(ctx, arg0, 414, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x350 = bitAnd(((x349 + get(ctx, arg0, 574, 1)) + get(ctx, arg0, 734, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 190, x350);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x351 = ((get(ctx, arg0, 95, 1) + get(ctx, arg0, 255, 1)) + get(ctx, arg0, 415, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x352 = bitAnd(((x351 + get(ctx, arg0, 575, 1)) + get(ctx, arg0, 735, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 191, x352);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x353 = ((get(ctx, arg0, 96, 1) + get(ctx, arg0, 256, 1)) + get(ctx, arg0, 416, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x354 = bitAnd(((x353 + get(ctx, arg0, 576, 1)) + get(ctx, arg0, 736, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 192, x354);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x355 = ((get(ctx, arg0, 97, 1) + get(ctx, arg0, 257, 1)) + get(ctx, arg0, 417, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x356 = bitAnd(((x355 + get(ctx, arg0, 577, 1)) + get(ctx, arg0, 737, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 193, x356);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x357 = ((get(ctx, arg0, 98, 1) + get(ctx, arg0, 258, 1)) + get(ctx, arg0, 418, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x358 = bitAnd(((x357 + get(ctx, arg0, 578, 1)) + get(ctx, arg0, 738, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 194, x358);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x359 = ((get(ctx, arg0, 99, 1) + get(ctx, arg0, 259, 1)) + get(ctx, arg0, 419, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x360 = bitAnd(((x359 + get(ctx, arg0, 579, 1)) + get(ctx, arg0, 739, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 195, x360);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x361 = ((get(ctx, arg0, 100, 1) + get(ctx, arg0, 260, 1)) + get(ctx, arg0, 420, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x362 = bitAnd(((x361 + get(ctx, arg0, 580, 1)) + get(ctx, arg0, 740, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 196, x362);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x363 = ((get(ctx, arg0, 101, 1) + get(ctx, arg0, 261, 1)) + get(ctx, arg0, 421, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x364 = bitAnd(((x363 + get(ctx, arg0, 581, 1)) + get(ctx, arg0, 741, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 197, x364);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x365 = ((get(ctx, arg0, 102, 1) + get(ctx, arg0, 262, 1)) + get(ctx, arg0, 422, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x366 = bitAnd(((x365 + get(ctx, arg0, 582, 1)) + get(ctx, arg0, 742, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 198, x366);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x367 = ((get(ctx, arg0, 103, 1) + get(ctx, arg0, 263, 1)) + get(ctx, arg0, 423, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x368 = bitAnd(((x367 + get(ctx, arg0, 583, 1)) + get(ctx, arg0, 743, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 199, x368);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x369 = ((get(ctx, arg0, 104, 1) + get(ctx, arg0, 264, 1)) + get(ctx, arg0, 424, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x370 = bitAnd(((x369 + get(ctx, arg0, 584, 1)) + get(ctx, arg0, 744, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 200, x370);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x371 = ((get(ctx, arg0, 105, 1) + get(ctx, arg0, 265, 1)) + get(ctx, arg0, 425, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x372 = bitAnd(((x371 + get(ctx, arg0, 585, 1)) + get(ctx, arg0, 745, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 201, x372);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x373 = ((get(ctx, arg0, 106, 1) + get(ctx, arg0, 266, 1)) + get(ctx, arg0, 426, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x374 = bitAnd(((x373 + get(ctx, arg0, 586, 1)) + get(ctx, arg0, 746, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 202, x374);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x375 = ((get(ctx, arg0, 107, 1) + get(ctx, arg0, 267, 1)) + get(ctx, arg0, 427, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x376 = bitAnd(((x375 + get(ctx, arg0, 587, 1)) + get(ctx, arg0, 747, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 203, x376);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x377 = ((get(ctx, arg0, 108, 1) + get(ctx, arg0, 268, 1)) + get(ctx, arg0, 428, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x378 = bitAnd(((x377 + get(ctx, arg0, 588, 1)) + get(ctx, arg0, 748, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 204, x378);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x379 = ((get(ctx, arg0, 109, 1) + get(ctx, arg0, 269, 1)) + get(ctx, arg0, 429, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x380 = bitAnd(((x379 + get(ctx, arg0, 589, 1)) + get(ctx, arg0, 749, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 205, x380);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x381 = ((get(ctx, arg0, 110, 1) + get(ctx, arg0, 270, 1)) + get(ctx, arg0, 430, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x382 = bitAnd(((x381 + get(ctx, arg0, 590, 1)) + get(ctx, arg0, 750, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 206, x382);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x383 = ((get(ctx, arg0, 111, 1) + get(ctx, arg0, 271, 1)) + get(ctx, arg0, 431, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x384 = bitAnd(((x383 + get(ctx, arg0, 591, 1)) + get(ctx, arg0, 751, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 207, x384);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x385 = ((get(ctx, arg0, 112, 2) + get(ctx, arg0, 272, 2)) + get(ctx, arg0, 432, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x386 = bitAnd(((x385 + get(ctx, arg0, 592, 2)) + get(ctx, arg0, 752, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 208, x386);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x387 = ((get(ctx, arg0, 113, 2) + get(ctx, arg0, 273, 2)) + get(ctx, arg0, 433, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x388 = bitAnd(((x387 + get(ctx, arg0, 593, 2)) + get(ctx, arg0, 753, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 209, x388);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x389 = ((get(ctx, arg0, 114, 2) + get(ctx, arg0, 274, 2)) + get(ctx, arg0, 434, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x390 = bitAnd(((x389 + get(ctx, arg0, 594, 2)) + get(ctx, arg0, 754, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 210, x390);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x391 = ((get(ctx, arg0, 115, 2) + get(ctx, arg0, 275, 2)) + get(ctx, arg0, 435, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x392 = bitAnd(((x391 + get(ctx, arg0, 595, 2)) + get(ctx, arg0, 755, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 211, x392);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x393 = ((get(ctx, arg0, 116, 2) + get(ctx, arg0, 276, 2)) + get(ctx, arg0, 436, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x394 = bitAnd(((x393 + get(ctx, arg0, 596, 2)) + get(ctx, arg0, 756, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 212, x394);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x395 = ((get(ctx, arg0, 117, 2) + get(ctx, arg0, 277, 2)) + get(ctx, arg0, 437, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x396 = bitAnd(((x395 + get(ctx, arg0, 597, 2)) + get(ctx, arg0, 757, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 213, x396);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x397 = ((get(ctx, arg0, 118, 2) + get(ctx, arg0, 278, 2)) + get(ctx, arg0, 438, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x398 = bitAnd(((x397 + get(ctx, arg0, 598, 2)) + get(ctx, arg0, 758, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 214, x398);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x399 = ((get(ctx, arg0, 119, 2) + get(ctx, arg0, 279, 2)) + get(ctx, arg0, 439, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x400 = bitAnd(((x399 + get(ctx, arg0, 599, 2)) + get(ctx, arg0, 759, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 215, x400);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x401 = ((get(ctx, arg0, 120, 2) + get(ctx, arg0, 280, 2)) + get(ctx, arg0, 440, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x402 = bitAnd(((x401 + get(ctx, arg0, 600, 2)) + get(ctx, arg0, 760, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 216, x402);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x403 = ((get(ctx, arg0, 121, 2) + get(ctx, arg0, 281, 2)) + get(ctx, arg0, 441, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x404 = bitAnd(((x403 + get(ctx, arg0, 601, 2)) + get(ctx, arg0, 761, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 217, x404);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x405 = ((get(ctx, arg0, 122, 2) + get(ctx, arg0, 282, 2)) + get(ctx, arg0, 442, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x406 = bitAnd(((x405 + get(ctx, arg0, 602, 2)) + get(ctx, arg0, 762, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 218, x406);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x407 = ((get(ctx, arg0, 123, 2) + get(ctx, arg0, 283, 2)) + get(ctx, arg0, 443, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x408 = bitAnd(((x407 + get(ctx, arg0, 603, 2)) + get(ctx, arg0, 763, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 219, x408);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x409 = ((get(ctx, arg0, 124, 2) + get(ctx, arg0, 284, 2)) + get(ctx, arg0, 444, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x410 = bitAnd(((x409 + get(ctx, arg0, 604, 2)) + get(ctx, arg0, 764, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 220, x410);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x411 = ((get(ctx, arg0, 125, 2) + get(ctx, arg0, 285, 2)) + get(ctx, arg0, 445, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x412 = bitAnd(((x411 + get(ctx, arg0, 605, 2)) + get(ctx, arg0, 765, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 221, x412);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x413 = ((get(ctx, arg0, 126, 2) + get(ctx, arg0, 286, 2)) + get(ctx, arg0, 446, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x414 = bitAnd(((x413 + get(ctx, arg0, 606, 2)) + get(ctx, arg0, 766, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 222, x414);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x415 = ((get(ctx, arg0, 127, 2) + get(ctx, arg0, 287, 2)) + get(ctx, arg0, 447, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x416 = bitAnd(((x415 + get(ctx, arg0, 607, 2)) + get(ctx, arg0, 767, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 223, x416);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x417 = ((get(ctx, arg0, 128, 2) + get(ctx, arg0, 288, 2)) + get(ctx, arg0, 448, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x418 = bitAnd(((x417 + get(ctx, arg0, 608, 2)) + get(ctx, arg0, 768, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 224, x418);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x419 = ((get(ctx, arg0, 129, 2) + get(ctx, arg0, 289, 2)) + get(ctx, arg0, 449, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x420 = bitAnd(((x419 + get(ctx, arg0, 609, 2)) + get(ctx, arg0, 769, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 225, x420);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x421 = ((get(ctx, arg0, 130, 2) + get(ctx, arg0, 290, 2)) + get(ctx, arg0, 450, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x422 = bitAnd(((x421 + get(ctx, arg0, 610, 2)) + get(ctx, arg0, 770, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 226, x422);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x423 = ((get(ctx, arg0, 131, 2) + get(ctx, arg0, 291, 2)) + get(ctx, arg0, 451, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x424 = bitAnd(((x423 + get(ctx, arg0, 611, 2)) + get(ctx, arg0, 771, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 227, x424);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x425 = ((get(ctx, arg0, 132, 2) + get(ctx, arg0, 292, 2)) + get(ctx, arg0, 452, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x426 = bitAnd(((x425 + get(ctx, arg0, 612, 2)) + get(ctx, arg0, 772, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 228, x426);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x427 = ((get(ctx, arg0, 133, 2) + get(ctx, arg0, 293, 2)) + get(ctx, arg0, 453, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x428 = bitAnd(((x427 + get(ctx, arg0, 613, 2)) + get(ctx, arg0, 773, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 229, x428);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x429 = ((get(ctx, arg0, 134, 2) + get(ctx, arg0, 294, 2)) + get(ctx, arg0, 454, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x430 = bitAnd(((x429 + get(ctx, arg0, 614, 2)) + get(ctx, arg0, 774, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 230, x430);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x431 = ((get(ctx, arg0, 135, 2) + get(ctx, arg0, 295, 2)) + get(ctx, arg0, 455, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x432 = bitAnd(((x431 + get(ctx, arg0, 615, 2)) + get(ctx, arg0, 775, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 231, x432);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x433 = ((get(ctx, arg0, 136, 2) + get(ctx, arg0, 296, 2)) + get(ctx, arg0, 456, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x434 = bitAnd(((x433 + get(ctx, arg0, 616, 2)) + get(ctx, arg0, 776, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 232, x434);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x435 = ((get(ctx, arg0, 137, 2) + get(ctx, arg0, 297, 2)) + get(ctx, arg0, 457, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x436 = bitAnd(((x435 + get(ctx, arg0, 617, 2)) + get(ctx, arg0, 777, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 233, x436);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x437 = ((get(ctx, arg0, 138, 2) + get(ctx, arg0, 298, 2)) + get(ctx, arg0, 458, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x438 = bitAnd(((x437 + get(ctx, arg0, 618, 2)) + get(ctx, arg0, 778, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 234, x438);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x439 = ((get(ctx, arg0, 139, 2) + get(ctx, arg0, 299, 2)) + get(ctx, arg0, 459, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x440 = bitAnd(((x439 + get(ctx, arg0, 619, 2)) + get(ctx, arg0, 779, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 235, x440);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x441 = ((get(ctx, arg0, 140, 2) + get(ctx, arg0, 300, 2)) + get(ctx, arg0, 460, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x442 = bitAnd(((x441 + get(ctx, arg0, 620, 2)) + get(ctx, arg0, 780, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 236, x442);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x443 = ((get(ctx, arg0, 141, 2) + get(ctx, arg0, 301, 2)) + get(ctx, arg0, 461, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x444 = bitAnd(((x443 + get(ctx, arg0, 621, 2)) + get(ctx, arg0, 781, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 237, x444);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x445 = ((get(ctx, arg0, 142, 2) + get(ctx, arg0, 302, 2)) + get(ctx, arg0, 462, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x446 = bitAnd(((x445 + get(ctx, arg0, 622, 2)) + get(ctx, arg0, 782, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 238, x446);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x447 = ((get(ctx, arg0, 143, 2) + get(ctx, arg0, 303, 2)) + get(ctx, arg0, 463, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x448 = bitAnd(((x447 + get(ctx, arg0, 623, 2)) + get(ctx, arg0, 783, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 239, x448);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x449 = ((get(ctx, arg0, 112, 1) + get(ctx, arg0, 272, 1)) + get(ctx, arg0, 432, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x450 = bitAnd(((x449 + get(ctx, arg0, 592, 1)) + get(ctx, arg0, 752, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 240, x450);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x451 = ((get(ctx, arg0, 113, 1) + get(ctx, arg0, 273, 1)) + get(ctx, arg0, 433, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x452 = bitAnd(((x451 + get(ctx, arg0, 593, 1)) + get(ctx, arg0, 753, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 241, x452);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x453 = ((get(ctx, arg0, 114, 1) + get(ctx, arg0, 274, 1)) + get(ctx, arg0, 434, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x454 = bitAnd(((x453 + get(ctx, arg0, 594, 1)) + get(ctx, arg0, 754, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 242, x454);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x455 = ((get(ctx, arg0, 115, 1) + get(ctx, arg0, 275, 1)) + get(ctx, arg0, 435, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x456 = bitAnd(((x455 + get(ctx, arg0, 595, 1)) + get(ctx, arg0, 755, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 243, x456);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x457 = ((get(ctx, arg0, 116, 1) + get(ctx, arg0, 276, 1)) + get(ctx, arg0, 436, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x458 = bitAnd(((x457 + get(ctx, arg0, 596, 1)) + get(ctx, arg0, 756, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 244, x458);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x459 = ((get(ctx, arg0, 117, 1) + get(ctx, arg0, 277, 1)) + get(ctx, arg0, 437, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x460 = bitAnd(((x459 + get(ctx, arg0, 597, 1)) + get(ctx, arg0, 757, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 245, x460);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x461 = ((get(ctx, arg0, 118, 1) + get(ctx, arg0, 278, 1)) + get(ctx, arg0, 438, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x462 = bitAnd(((x461 + get(ctx, arg0, 598, 1)) + get(ctx, arg0, 758, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 246, x462);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x463 = ((get(ctx, arg0, 119, 1) + get(ctx, arg0, 279, 1)) + get(ctx, arg0, 439, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x464 = bitAnd(((x463 + get(ctx, arg0, 599, 1)) + get(ctx, arg0, 759, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 247, x464);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x465 = ((get(ctx, arg0, 120, 1) + get(ctx, arg0, 280, 1)) + get(ctx, arg0, 440, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x466 = bitAnd(((x465 + get(ctx, arg0, 600, 1)) + get(ctx, arg0, 760, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 248, x466);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x467 = ((get(ctx, arg0, 121, 1) + get(ctx, arg0, 281, 1)) + get(ctx, arg0, 441, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x468 = bitAnd(((x467 + get(ctx, arg0, 601, 1)) + get(ctx, arg0, 761, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 249, x468);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x469 = ((get(ctx, arg0, 122, 1) + get(ctx, arg0, 282, 1)) + get(ctx, arg0, 442, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x470 = bitAnd(((x469 + get(ctx, arg0, 602, 1)) + get(ctx, arg0, 762, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 250, x470);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x471 = ((get(ctx, arg0, 123, 1) + get(ctx, arg0, 283, 1)) + get(ctx, arg0, 443, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x472 = bitAnd(((x471 + get(ctx, arg0, 603, 1)) + get(ctx, arg0, 763, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 251, x472);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x473 = ((get(ctx, arg0, 124, 1) + get(ctx, arg0, 284, 1)) + get(ctx, arg0, 444, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x474 = bitAnd(((x473 + get(ctx, arg0, 604, 1)) + get(ctx, arg0, 764, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 252, x474);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x475 = ((get(ctx, arg0, 125, 1) + get(ctx, arg0, 285, 1)) + get(ctx, arg0, 445, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x476 = bitAnd(((x475 + get(ctx, arg0, 605, 1)) + get(ctx, arg0, 765, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 253, x476);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x477 = ((get(ctx, arg0, 126, 1) + get(ctx, arg0, 286, 1)) + get(ctx, arg0, 446, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x478 = bitAnd(((x477 + get(ctx, arg0, 606, 1)) + get(ctx, arg0, 766, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 254, x478);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x479 = ((get(ctx, arg0, 127, 1) + get(ctx, arg0, 287, 1)) + get(ctx, arg0, 447, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x480 = bitAnd(((x479 + get(ctx, arg0, 607, 1)) + get(ctx, arg0, 767, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 255, x480);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x481 = ((get(ctx, arg0, 128, 1) + get(ctx, arg0, 288, 1)) + get(ctx, arg0, 448, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x482 = bitAnd(((x481 + get(ctx, arg0, 608, 1)) + get(ctx, arg0, 768, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 256, x482);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x483 = ((get(ctx, arg0, 129, 1) + get(ctx, arg0, 289, 1)) + get(ctx, arg0, 449, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x484 = bitAnd(((x483 + get(ctx, arg0, 609, 1)) + get(ctx, arg0, 769, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 257, x484);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x485 = ((get(ctx, arg0, 130, 1) + get(ctx, arg0, 290, 1)) + get(ctx, arg0, 450, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x486 = bitAnd(((x485 + get(ctx, arg0, 610, 1)) + get(ctx, arg0, 770, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 258, x486);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x487 = ((get(ctx, arg0, 131, 1) + get(ctx, arg0, 291, 1)) + get(ctx, arg0, 451, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x488 = bitAnd(((x487 + get(ctx, arg0, 611, 1)) + get(ctx, arg0, 771, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 259, x488);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x489 = ((get(ctx, arg0, 132, 1) + get(ctx, arg0, 292, 1)) + get(ctx, arg0, 452, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x490 = bitAnd(((x489 + get(ctx, arg0, 612, 1)) + get(ctx, arg0, 772, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 260, x490);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x491 = ((get(ctx, arg0, 133, 1) + get(ctx, arg0, 293, 1)) + get(ctx, arg0, 453, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x492 = bitAnd(((x491 + get(ctx, arg0, 613, 1)) + get(ctx, arg0, 773, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 261, x492);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x493 = ((get(ctx, arg0, 134, 1) + get(ctx, arg0, 294, 1)) + get(ctx, arg0, 454, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x494 = bitAnd(((x493 + get(ctx, arg0, 614, 1)) + get(ctx, arg0, 774, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 262, x494);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x495 = ((get(ctx, arg0, 135, 1) + get(ctx, arg0, 295, 1)) + get(ctx, arg0, 455, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x496 = bitAnd(((x495 + get(ctx, arg0, 615, 1)) + get(ctx, arg0, 775, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 263, x496);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x497 = ((get(ctx, arg0, 136, 1) + get(ctx, arg0, 296, 1)) + get(ctx, arg0, 456, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x498 = bitAnd(((x497 + get(ctx, arg0, 616, 1)) + get(ctx, arg0, 776, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 264, x498);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x499 = ((get(ctx, arg0, 137, 1) + get(ctx, arg0, 297, 1)) + get(ctx, arg0, 457, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x500 = bitAnd(((x499 + get(ctx, arg0, 617, 1)) + get(ctx, arg0, 777, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 265, x500);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x501 = ((get(ctx, arg0, 138, 1) + get(ctx, arg0, 298, 1)) + get(ctx, arg0, 458, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x502 = bitAnd(((x501 + get(ctx, arg0, 618, 1)) + get(ctx, arg0, 778, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 266, x502);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x503 = ((get(ctx, arg0, 139, 1) + get(ctx, arg0, 299, 1)) + get(ctx, arg0, 459, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x504 = bitAnd(((x503 + get(ctx, arg0, 619, 1)) + get(ctx, arg0, 779, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 267, x504);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x505 = ((get(ctx, arg0, 140, 1) + get(ctx, arg0, 300, 1)) + get(ctx, arg0, 460, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x506 = bitAnd(((x505 + get(ctx, arg0, 620, 1)) + get(ctx, arg0, 780, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 268, x506);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x507 = ((get(ctx, arg0, 141, 1) + get(ctx, arg0, 301, 1)) + get(ctx, arg0, 461, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x508 = bitAnd(((x507 + get(ctx, arg0, 621, 1)) + get(ctx, arg0, 781, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 269, x508);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x509 = ((get(ctx, arg0, 142, 1) + get(ctx, arg0, 302, 1)) + get(ctx, arg0, 462, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x510 = bitAnd(((x509 + get(ctx, arg0, 622, 1)) + get(ctx, arg0, 782, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 270, x510);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x511 = ((get(ctx, arg0, 143, 1) + get(ctx, arg0, 303, 1)) + get(ctx, arg0, 463, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x512 = bitAnd(((x511 + get(ctx, arg0, 623, 1)) + get(ctx, arg0, 783, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 271, x512);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x513 = ((get(ctx, arg0, 144, 2) + get(ctx, arg0, 304, 2)) + get(ctx, arg0, 464, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x514 = bitAnd(((x513 + get(ctx, arg0, 624, 2)) + get(ctx, arg0, 784, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 272, x514);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x515 = ((get(ctx, arg0, 145, 2) + get(ctx, arg0, 305, 2)) + get(ctx, arg0, 465, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x516 = bitAnd(((x515 + get(ctx, arg0, 625, 2)) + get(ctx, arg0, 785, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 273, x516);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x517 = ((get(ctx, arg0, 146, 2) + get(ctx, arg0, 306, 2)) + get(ctx, arg0, 466, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x518 = bitAnd(((x517 + get(ctx, arg0, 626, 2)) + get(ctx, arg0, 786, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 274, x518);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x519 = ((get(ctx, arg0, 147, 2) + get(ctx, arg0, 307, 2)) + get(ctx, arg0, 467, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x520 = bitAnd(((x519 + get(ctx, arg0, 627, 2)) + get(ctx, arg0, 787, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 275, x520);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x521 = ((get(ctx, arg0, 148, 2) + get(ctx, arg0, 308, 2)) + get(ctx, arg0, 468, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x522 = bitAnd(((x521 + get(ctx, arg0, 628, 2)) + get(ctx, arg0, 788, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 276, x522);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x523 = ((get(ctx, arg0, 149, 2) + get(ctx, arg0, 309, 2)) + get(ctx, arg0, 469, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x524 = bitAnd(((x523 + get(ctx, arg0, 629, 2)) + get(ctx, arg0, 789, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 277, x524);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x525 = ((get(ctx, arg0, 150, 2) + get(ctx, arg0, 310, 2)) + get(ctx, arg0, 470, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x526 = bitAnd(((x525 + get(ctx, arg0, 630, 2)) + get(ctx, arg0, 790, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 278, x526);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x527 = ((get(ctx, arg0, 151, 2) + get(ctx, arg0, 311, 2)) + get(ctx, arg0, 471, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x528 = bitAnd(((x527 + get(ctx, arg0, 631, 2)) + get(ctx, arg0, 791, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 279, x528);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x529 = ((get(ctx, arg0, 152, 2) + get(ctx, arg0, 312, 2)) + get(ctx, arg0, 472, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x530 = bitAnd(((x529 + get(ctx, arg0, 632, 2)) + get(ctx, arg0, 792, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 280, x530);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x531 = ((get(ctx, arg0, 153, 2) + get(ctx, arg0, 313, 2)) + get(ctx, arg0, 473, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x532 = bitAnd(((x531 + get(ctx, arg0, 633, 2)) + get(ctx, arg0, 793, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 281, x532);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x533 = ((get(ctx, arg0, 154, 2) + get(ctx, arg0, 314, 2)) + get(ctx, arg0, 474, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x534 = bitAnd(((x533 + get(ctx, arg0, 634, 2)) + get(ctx, arg0, 794, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 282, x534);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x535 = ((get(ctx, arg0, 155, 2) + get(ctx, arg0, 315, 2)) + get(ctx, arg0, 475, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x536 = bitAnd(((x535 + get(ctx, arg0, 635, 2)) + get(ctx, arg0, 795, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 283, x536);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x537 = ((get(ctx, arg0, 156, 2) + get(ctx, arg0, 316, 2)) + get(ctx, arg0, 476, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x538 = bitAnd(((x537 + get(ctx, arg0, 636, 2)) + get(ctx, arg0, 796, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 284, x538);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x539 = ((get(ctx, arg0, 157, 2) + get(ctx, arg0, 317, 2)) + get(ctx, arg0, 477, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x540 = bitAnd(((x539 + get(ctx, arg0, 637, 2)) + get(ctx, arg0, 797, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 285, x540);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x541 = ((get(ctx, arg0, 158, 2) + get(ctx, arg0, 318, 2)) + get(ctx, arg0, 478, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x542 = bitAnd(((x541 + get(ctx, arg0, 638, 2)) + get(ctx, arg0, 798, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 286, x542);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x543 = ((get(ctx, arg0, 159, 2) + get(ctx, arg0, 319, 2)) + get(ctx, arg0, 479, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x544 = bitAnd(((x543 + get(ctx, arg0, 639, 2)) + get(ctx, arg0, 799, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 287, x544);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x545 = ((get(ctx, arg0, 160, 2) + get(ctx, arg0, 320, 2)) + get(ctx, arg0, 480, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x546 = bitAnd(((x545 + get(ctx, arg0, 640, 2)) + get(ctx, arg0, 800, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 288, x546);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x547 = ((get(ctx, arg0, 161, 2) + get(ctx, arg0, 321, 2)) + get(ctx, arg0, 481, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x548 = bitAnd(((x547 + get(ctx, arg0, 641, 2)) + get(ctx, arg0, 801, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 289, x548);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x549 = ((get(ctx, arg0, 162, 2) + get(ctx, arg0, 322, 2)) + get(ctx, arg0, 482, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x550 = bitAnd(((x549 + get(ctx, arg0, 642, 2)) + get(ctx, arg0, 802, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 290, x550);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x551 = ((get(ctx, arg0, 163, 2) + get(ctx, arg0, 323, 2)) + get(ctx, arg0, 483, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x552 = bitAnd(((x551 + get(ctx, arg0, 643, 2)) + get(ctx, arg0, 803, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 291, x552);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x553 = ((get(ctx, arg0, 164, 2) + get(ctx, arg0, 324, 2)) + get(ctx, arg0, 484, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x554 = bitAnd(((x553 + get(ctx, arg0, 644, 2)) + get(ctx, arg0, 804, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 292, x554);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x555 = ((get(ctx, arg0, 165, 2) + get(ctx, arg0, 325, 2)) + get(ctx, arg0, 485, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x556 = bitAnd(((x555 + get(ctx, arg0, 645, 2)) + get(ctx, arg0, 805, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 293, x556);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x557 = ((get(ctx, arg0, 166, 2) + get(ctx, arg0, 326, 2)) + get(ctx, arg0, 486, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x558 = bitAnd(((x557 + get(ctx, arg0, 646, 2)) + get(ctx, arg0, 806, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 294, x558);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x559 = ((get(ctx, arg0, 167, 2) + get(ctx, arg0, 327, 2)) + get(ctx, arg0, 487, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x560 = bitAnd(((x559 + get(ctx, arg0, 647, 2)) + get(ctx, arg0, 807, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 295, x560);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x561 = ((get(ctx, arg0, 168, 2) + get(ctx, arg0, 328, 2)) + get(ctx, arg0, 488, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x562 = bitAnd(((x561 + get(ctx, arg0, 648, 2)) + get(ctx, arg0, 808, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 296, x562);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x563 = ((get(ctx, arg0, 169, 2) + get(ctx, arg0, 329, 2)) + get(ctx, arg0, 489, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x564 = bitAnd(((x563 + get(ctx, arg0, 649, 2)) + get(ctx, arg0, 809, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 297, x564);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x565 = ((get(ctx, arg0, 170, 2) + get(ctx, arg0, 330, 2)) + get(ctx, arg0, 490, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x566 = bitAnd(((x565 + get(ctx, arg0, 650, 2)) + get(ctx, arg0, 810, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 298, x566);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x567 = ((get(ctx, arg0, 171, 2) + get(ctx, arg0, 331, 2)) + get(ctx, arg0, 491, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x568 = bitAnd(((x567 + get(ctx, arg0, 651, 2)) + get(ctx, arg0, 811, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 299, x568);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x569 = ((get(ctx, arg0, 172, 2) + get(ctx, arg0, 332, 2)) + get(ctx, arg0, 492, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x570 = bitAnd(((x569 + get(ctx, arg0, 652, 2)) + get(ctx, arg0, 812, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 300, x570);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x571 = ((get(ctx, arg0, 173, 2) + get(ctx, arg0, 333, 2)) + get(ctx, arg0, 493, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x572 = bitAnd(((x571 + get(ctx, arg0, 653, 2)) + get(ctx, arg0, 813, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 301, x572);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x573 = ((get(ctx, arg0, 174, 2) + get(ctx, arg0, 334, 2)) + get(ctx, arg0, 494, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x574 = bitAnd(((x573 + get(ctx, arg0, 654, 2)) + get(ctx, arg0, 814, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 302, x574);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x575 = ((get(ctx, arg0, 175, 2) + get(ctx, arg0, 335, 2)) + get(ctx, arg0, 495, 2));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x576 = bitAnd(((x575 + get(ctx, arg0, 655, 2)) + get(ctx, arg0, 815, 2)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 303, x576);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x577 = ((get(ctx, arg0, 144, 1) + get(ctx, arg0, 304, 1)) + get(ctx, arg0, 464, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x578 = bitAnd(((x577 + get(ctx, arg0, 624, 1)) + get(ctx, arg0, 784, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 304, x578);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x579 = ((get(ctx, arg0, 145, 1) + get(ctx, arg0, 305, 1)) + get(ctx, arg0, 465, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x580 = bitAnd(((x579 + get(ctx, arg0, 625, 1)) + get(ctx, arg0, 785, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 305, x580);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x581 = ((get(ctx, arg0, 146, 1) + get(ctx, arg0, 306, 1)) + get(ctx, arg0, 466, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x582 = bitAnd(((x581 + get(ctx, arg0, 626, 1)) + get(ctx, arg0, 786, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 306, x582);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x583 = ((get(ctx, arg0, 147, 1) + get(ctx, arg0, 307, 1)) + get(ctx, arg0, 467, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x584 = bitAnd(((x583 + get(ctx, arg0, 627, 1)) + get(ctx, arg0, 787, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 307, x584);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x585 = ((get(ctx, arg0, 148, 1) + get(ctx, arg0, 308, 1)) + get(ctx, arg0, 468, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x586 = bitAnd(((x585 + get(ctx, arg0, 628, 1)) + get(ctx, arg0, 788, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 308, x586);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x587 = ((get(ctx, arg0, 149, 1) + get(ctx, arg0, 309, 1)) + get(ctx, arg0, 469, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x588 = bitAnd(((x587 + get(ctx, arg0, 629, 1)) + get(ctx, arg0, 789, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 309, x588);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x589 = ((get(ctx, arg0, 150, 1) + get(ctx, arg0, 310, 1)) + get(ctx, arg0, 470, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x590 = bitAnd(((x589 + get(ctx, arg0, 630, 1)) + get(ctx, arg0, 790, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 310, x590);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x591 = ((get(ctx, arg0, 151, 1) + get(ctx, arg0, 311, 1)) + get(ctx, arg0, 471, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x592 = bitAnd(((x591 + get(ctx, arg0, 631, 1)) + get(ctx, arg0, 791, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 311, x592);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x593 = ((get(ctx, arg0, 152, 1) + get(ctx, arg0, 312, 1)) + get(ctx, arg0, 472, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x594 = bitAnd(((x593 + get(ctx, arg0, 632, 1)) + get(ctx, arg0, 792, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 312, x594);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x595 = ((get(ctx, arg0, 153, 1) + get(ctx, arg0, 313, 1)) + get(ctx, arg0, 473, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x596 = bitAnd(((x595 + get(ctx, arg0, 633, 1)) + get(ctx, arg0, 793, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 313, x596);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x597 = ((get(ctx, arg0, 154, 1) + get(ctx, arg0, 314, 1)) + get(ctx, arg0, 474, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x598 = bitAnd(((x597 + get(ctx, arg0, 634, 1)) + get(ctx, arg0, 794, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 314, x598);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x599 = ((get(ctx, arg0, 155, 1) + get(ctx, arg0, 315, 1)) + get(ctx, arg0, 475, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x600 = bitAnd(((x599 + get(ctx, arg0, 635, 1)) + get(ctx, arg0, 795, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 315, x600);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x601 = ((get(ctx, arg0, 156, 1) + get(ctx, arg0, 316, 1)) + get(ctx, arg0, 476, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x602 = bitAnd(((x601 + get(ctx, arg0, 636, 1)) + get(ctx, arg0, 796, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 316, x602);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x603 = ((get(ctx, arg0, 157, 1) + get(ctx, arg0, 317, 1)) + get(ctx, arg0, 477, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x604 = bitAnd(((x603 + get(ctx, arg0, 637, 1)) + get(ctx, arg0, 797, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 317, x604);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x605 = ((get(ctx, arg0, 158, 1) + get(ctx, arg0, 318, 1)) + get(ctx, arg0, 478, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x606 = bitAnd(((x605 + get(ctx, arg0, 638, 1)) + get(ctx, arg0, 798, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 318, x606);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x607 = ((get(ctx, arg0, 159, 1) + get(ctx, arg0, 319, 1)) + get(ctx, arg0, 479, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x608 = bitAnd(((x607 + get(ctx, arg0, 639, 1)) + get(ctx, arg0, 799, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 319, x608);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x609 = ((get(ctx, arg0, 160, 1) + get(ctx, arg0, 320, 1)) + get(ctx, arg0, 480, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x610 = bitAnd(((x609 + get(ctx, arg0, 640, 1)) + get(ctx, arg0, 800, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 320, x610);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x611 = ((get(ctx, arg0, 161, 1) + get(ctx, arg0, 321, 1)) + get(ctx, arg0, 481, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x612 = bitAnd(((x611 + get(ctx, arg0, 641, 1)) + get(ctx, arg0, 801, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 321, x612);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x613 = ((get(ctx, arg0, 162, 1) + get(ctx, arg0, 322, 1)) + get(ctx, arg0, 482, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x614 = bitAnd(((x613 + get(ctx, arg0, 642, 1)) + get(ctx, arg0, 802, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 322, x614);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x615 = ((get(ctx, arg0, 163, 1) + get(ctx, arg0, 323, 1)) + get(ctx, arg0, 483, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x616 = bitAnd(((x615 + get(ctx, arg0, 643, 1)) + get(ctx, arg0, 803, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 323, x616);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x617 = ((get(ctx, arg0, 164, 1) + get(ctx, arg0, 324, 1)) + get(ctx, arg0, 484, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x618 = bitAnd(((x617 + get(ctx, arg0, 644, 1)) + get(ctx, arg0, 804, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 324, x618);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x619 = ((get(ctx, arg0, 165, 1) + get(ctx, arg0, 325, 1)) + get(ctx, arg0, 485, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x620 = bitAnd(((x619 + get(ctx, arg0, 645, 1)) + get(ctx, arg0, 805, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 325, x620);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x621 = ((get(ctx, arg0, 166, 1) + get(ctx, arg0, 326, 1)) + get(ctx, arg0, 486, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x622 = bitAnd(((x621 + get(ctx, arg0, 646, 1)) + get(ctx, arg0, 806, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 326, x622);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x623 = ((get(ctx, arg0, 167, 1) + get(ctx, arg0, 327, 1)) + get(ctx, arg0, 487, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x624 = bitAnd(((x623 + get(ctx, arg0, 647, 1)) + get(ctx, arg0, 807, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 327, x624);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x625 = ((get(ctx, arg0, 168, 1) + get(ctx, arg0, 328, 1)) + get(ctx, arg0, 488, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x626 = bitAnd(((x625 + get(ctx, arg0, 648, 1)) + get(ctx, arg0, 808, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 328, x626);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x627 = ((get(ctx, arg0, 169, 1) + get(ctx, arg0, 329, 1)) + get(ctx, arg0, 489, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x628 = bitAnd(((x627 + get(ctx, arg0, 649, 1)) + get(ctx, arg0, 809, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 329, x628);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x629 = ((get(ctx, arg0, 170, 1) + get(ctx, arg0, 330, 1)) + get(ctx, arg0, 490, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x630 = bitAnd(((x629 + get(ctx, arg0, 650, 1)) + get(ctx, arg0, 810, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 330, x630);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x631 = ((get(ctx, arg0, 171, 1) + get(ctx, arg0, 331, 1)) + get(ctx, arg0, 491, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x632 = bitAnd(((x631 + get(ctx, arg0, 651, 1)) + get(ctx, arg0, 811, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 331, x632);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x633 = ((get(ctx, arg0, 172, 1) + get(ctx, arg0, 332, 1)) + get(ctx, arg0, 492, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x634 = bitAnd(((x633 + get(ctx, arg0, 652, 1)) + get(ctx, arg0, 812, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 332, x634);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x635 = ((get(ctx, arg0, 173, 1) + get(ctx, arg0, 333, 1)) + get(ctx, arg0, 493, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x636 = bitAnd(((x635 + get(ctx, arg0, 653, 1)) + get(ctx, arg0, 813, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 333, x636);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x637 = ((get(ctx, arg0, 174, 1) + get(ctx, arg0, 334, 1)) + get(ctx, arg0, 494, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x638 = bitAnd(((x637 + get(ctx, arg0, 654, 1)) + get(ctx, arg0, 814, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 334, x638);
  // builtin Add
  // Xor5(zirgen/circuit/keccak/xor5.zir:7)
  Val x639 = ((get(ctx, arg0, 175, 1) + get(ctx, arg0, 335, 1)) + get(ctx, arg0, 495, 1));
  // builtin BitAnd
  // Xor5(zirgen/circuit/keccak/xor5.zir:8)
  Val x640 = bitAnd(((x639 + get(ctx, arg0, 655, 1)) + get(ctx, arg0, 815, 1)), Val(1));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 335, x640);
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  // KeccakRound0(zirgen/circuit/keccak/top.zir:98)
  set(ctx, arg0, 16, get(ctx, arg0, 16, 0));
  set(ctx, arg0, 17, get(ctx, arg0, 17, 0));
  set(ctx, arg0, 18, get(ctx, arg0, 18, 0));
  set(ctx, arg0, 19, get(ctx, arg0, 19, 0));
  set(ctx, arg0, 20, get(ctx, arg0, 20, 0));
  set(ctx, arg0, 21, get(ctx, arg0, 21, 0));
  set(ctx, arg0, 22, get(ctx, arg0, 22, 0));
  set(ctx, arg0, 23, get(ctx, arg0, 23, 0));
  set(ctx, arg0, 24, get(ctx, arg0, 24, 0));
  set(ctx, arg0, 25, get(ctx, arg0, 25, 0));
  set(ctx, arg0, 26, get(ctx, arg0, 26, 0));
  set(ctx, arg0, 27, get(ctx, arg0, 27, 0));
  set(ctx, arg0, 28, get(ctx, arg0, 28, 0));
  set(ctx, arg0, 29, get(ctx, arg0, 29, 0));
  set(ctx, arg0, 30, get(ctx, arg0, 30, 0));
  set(ctx, arg0, 31, get(ctx, arg0, 31, 0));
  set(ctx, arg0, 32, get(ctx, arg0, 32, 0));
  set(ctx, arg0, 33, get(ctx, arg0, 33, 0));
  set(ctx, arg0, 34, get(ctx, arg0, 34, 0));
  set(ctx, arg0, 35, get(ctx, arg0, 35, 0));
  set(ctx, arg0, 36, get(ctx, arg0, 36, 0));
  set(ctx, arg0, 37, get(ctx, arg0, 37, 0));
  set(ctx, arg0, 38, get(ctx, arg0, 38, 0));
  set(ctx, arg0, 39, get(ctx, arg0, 39, 0));
  set(ctx, arg0, 40, get(ctx, arg0, 40, 0));
  set(ctx, arg0, 41, get(ctx, arg0, 41, 0));
  set(ctx, arg0, 42, get(ctx, arg0, 42, 0));
  set(ctx, arg0, 43, get(ctx, arg0, 43, 0));
  set(ctx, arg0, 44, get(ctx, arg0, 44, 0));
  set(ctx, arg0, 45, get(ctx, arg0, 45, 0));
  set(ctx, arg0, 46, get(ctx, arg0, 46, 0));
  set(ctx, arg0, 47, get(ctx, arg0, 47, 0));
  set(ctx, arg0, 48, get(ctx, arg0, 48, 0));
  set(ctx, arg0, 49, get(ctx, arg0, 49, 0));
  set(ctx, arg0, 50, get(ctx, arg0, 50, 0));
  set(ctx, arg0, 51, get(ctx, arg0, 51, 0));
  set(ctx, arg0, 52, get(ctx, arg0, 52, 0));
  set(ctx, arg0, 53, get(ctx, arg0, 53, 0));
  set(ctx, arg0, 54, get(ctx, arg0, 54, 0));
  set(ctx, arg0, 55, get(ctx, arg0, 55, 0));
  set(ctx, arg0, 56, get(ctx, arg0, 56, 0));
  set(ctx, arg0, 57, get(ctx, arg0, 57, 0));
  set(ctx, arg0, 58, get(ctx, arg0, 58, 0));
  set(ctx, arg0, 59, get(ctx, arg0, 59, 0));
  set(ctx, arg0, 60, get(ctx, arg0, 60, 0));
  set(ctx, arg0, 61, get(ctx, arg0, 61, 0));
  set(ctx, arg0, 62, get(ctx, arg0, 62, 0));
  set(ctx, arg0, 63, get(ctx, arg0, 63, 0));
  set(ctx, arg0, 64, get(ctx, arg0, 64, 0));
  set(ctx, arg0, 65, get(ctx, arg0, 65, 0));
  set(ctx, arg0, 66, get(ctx, arg0, 66, 0));
  set(ctx, arg0, 67, get(ctx, arg0, 67, 0));
  set(ctx, arg0, 68, get(ctx, arg0, 68, 0));
  set(ctx, arg0, 69, get(ctx, arg0, 69, 0));
  set(ctx, arg0, 70, get(ctx, arg0, 70, 0));
  set(ctx, arg0, 71, get(ctx, arg0, 71, 0));
  set(ctx, arg0, 72, get(ctx, arg0, 72, 0));
  set(ctx, arg0, 73, get(ctx, arg0, 73, 0));
  set(ctx, arg0, 74, get(ctx, arg0, 74, 0));
  set(ctx, arg0, 75, get(ctx, arg0, 75, 0));
  set(ctx, arg0, 76, get(ctx, arg0, 76, 0));
  set(ctx, arg0, 77, get(ctx, arg0, 77, 0));
  set(ctx, arg0, 78, get(ctx, arg0, 78, 0));
  set(ctx, arg0, 79, get(ctx, arg0, 79, 0));
  set(ctx, arg0, 80, get(ctx, arg0, 80, 0));
  set(ctx, arg0, 81, get(ctx, arg0, 81, 0));
  set(ctx, arg0, 82, get(ctx, arg0, 82, 0));
  set(ctx, arg0, 83, get(ctx, arg0, 83, 0));
  set(ctx, arg0, 84, get(ctx, arg0, 84, 0));
  set(ctx, arg0, 85, get(ctx, arg0, 85, 0));
  set(ctx, arg0, 86, get(ctx, arg0, 86, 0));
  set(ctx, arg0, 87, get(ctx, arg0, 87, 0));
  set(ctx, arg0, 88, get(ctx, arg0, 88, 0));
  set(ctx, arg0, 89, get(ctx, arg0, 89, 0));
  set(ctx, arg0, 90, get(ctx, arg0, 90, 0));
  set(ctx, arg0, 91, get(ctx, arg0, 91, 0));
  set(ctx, arg0, 92, get(ctx, arg0, 92, 0));
  set(ctx, arg0, 93, get(ctx, arg0, 93, 0));
  set(ctx, arg0, 94, get(ctx, arg0, 94, 0));
  set(ctx, arg0, 95, get(ctx, arg0, 95, 0));
  set(ctx, arg0, 96, get(ctx, arg0, 96, 0));
  set(ctx, arg0, 97, get(ctx, arg0, 97, 0));
  set(ctx, arg0, 98, get(ctx, arg0, 98, 0));
  set(ctx, arg0, 99, get(ctx, arg0, 99, 0));
  set(ctx, arg0, 100, get(ctx, arg0, 100, 0));
  set(ctx, arg0, 101, get(ctx, arg0, 101, 0));
  set(ctx, arg0, 102, get(ctx, arg0, 102, 0));
  set(ctx, arg0, 103, get(ctx, arg0, 103, 0));
  set(ctx, arg0, 104, get(ctx, arg0, 104, 0));
  set(ctx, arg0, 105, get(ctx, arg0, 105, 0));
  set(ctx, arg0, 106, get(ctx, arg0, 106, 0));
  set(ctx, arg0, 107, get(ctx, arg0, 107, 0));
  set(ctx, arg0, 108, get(ctx, arg0, 108, 0));
  set(ctx, arg0, 109, get(ctx, arg0, 109, 0));
  set(ctx, arg0, 110, get(ctx, arg0, 110, 0));
  set(ctx, arg0, 111, get(ctx, arg0, 111, 0));
  set(ctx, arg0, 112, get(ctx, arg0, 112, 0));
  set(ctx, arg0, 113, get(ctx, arg0, 113, 0));
  set(ctx, arg0, 114, get(ctx, arg0, 114, 0));
  set(ctx, arg0, 115, get(ctx, arg0, 115, 0));
  set(ctx, arg0, 116, get(ctx, arg0, 116, 0));
  set(ctx, arg0, 117, get(ctx, arg0, 117, 0));
  set(ctx, arg0, 118, get(ctx, arg0, 118, 0));
  set(ctx, arg0, 119, get(ctx, arg0, 119, 0));
  set(ctx, arg0, 120, get(ctx, arg0, 120, 0));
  set(ctx, arg0, 121, get(ctx, arg0, 121, 0));
  set(ctx, arg0, 122, get(ctx, arg0, 122, 0));
  set(ctx, arg0, 123, get(ctx, arg0, 123, 0));
  set(ctx, arg0, 124, get(ctx, arg0, 124, 0));
  set(ctx, arg0, 125, get(ctx, arg0, 125, 0));
  set(ctx, arg0, 126, get(ctx, arg0, 126, 0));
  set(ctx, arg0, 127, get(ctx, arg0, 127, 0));
  set(ctx, arg0, 128, get(ctx, arg0, 128, 0));
  set(ctx, arg0, 129, get(ctx, arg0, 129, 0));
  set(ctx, arg0, 130, get(ctx, arg0, 130, 0));
  set(ctx, arg0, 131, get(ctx, arg0, 131, 0));
  set(ctx, arg0, 132, get(ctx, arg0, 132, 0));
  set(ctx, arg0, 133, get(ctx, arg0, 133, 0));
  set(ctx, arg0, 134, get(ctx, arg0, 134, 0));
  set(ctx, arg0, 135, get(ctx, arg0, 135, 0));
  set(ctx, arg0, 136, get(ctx, arg0, 136, 0));
  set(ctx, arg0, 137, get(ctx, arg0, 137, 0));
  set(ctx, arg0, 138, get(ctx, arg0, 138, 0));
  set(ctx, arg0, 139, get(ctx, arg0, 139, 0));
  set(ctx, arg0, 140, get(ctx, arg0, 140, 0));
  set(ctx, arg0, 141, get(ctx, arg0, 141, 0));
  set(ctx, arg0, 142, get(ctx, arg0, 142, 0));
  set(ctx, arg0, 143, get(ctx, arg0, 143, 0));
  set(ctx, arg0, 144, get(ctx, arg0, 144, 0));
  set(ctx, arg0, 145, get(ctx, arg0, 145, 0));
  set(ctx, arg0, 146, get(ctx, arg0, 146, 0));
  set(ctx, arg0, 147, get(ctx, arg0, 147, 0));
  set(ctx, arg0, 148, get(ctx, arg0, 148, 0));
  set(ctx, arg0, 149, get(ctx, arg0, 149, 0));
  set(ctx, arg0, 150, get(ctx, arg0, 150, 0));
  set(ctx, arg0, 151, get(ctx, arg0, 151, 0));
  set(ctx, arg0, 152, get(ctx, arg0, 152, 0));
  set(ctx, arg0, 153, get(ctx, arg0, 153, 0));
  set(ctx, arg0, 154, get(ctx, arg0, 154, 0));
  set(ctx, arg0, 155, get(ctx, arg0, 155, 0));
  set(ctx, arg0, 156, get(ctx, arg0, 156, 0));
  set(ctx, arg0, 157, get(ctx, arg0, 157, 0));
  set(ctx, arg0, 158, get(ctx, arg0, 158, 0));
  set(ctx, arg0, 159, get(ctx, arg0, 159, 0));
  set(ctx, arg0, 160, get(ctx, arg0, 160, 0));
  set(ctx, arg0, 161, get(ctx, arg0, 161, 0));
  set(ctx, arg0, 162, get(ctx, arg0, 162, 0));
  set(ctx, arg0, 163, get(ctx, arg0, 163, 0));
  set(ctx, arg0, 164, get(ctx, arg0, 164, 0));
  set(ctx, arg0, 165, get(ctx, arg0, 165, 0));
  set(ctx, arg0, 166, get(ctx, arg0, 166, 0));
  set(ctx, arg0, 167, get(ctx, arg0, 167, 0));
  set(ctx, arg0, 168, get(ctx, arg0, 168, 0));
  set(ctx, arg0, 169, get(ctx, arg0, 169, 0));
  set(ctx, arg0, 170, get(ctx, arg0, 170, 0));
  set(ctx, arg0, 171, get(ctx, arg0, 171, 0));
  set(ctx, arg0, 172, get(ctx, arg0, 172, 0));
  set(ctx, arg0, 173, get(ctx, arg0, 173, 0));
  set(ctx, arg0, 174, get(ctx, arg0, 174, 0));
  set(ctx, arg0, 175, get(ctx, arg0, 175, 0));
  set(ctx, arg0, 176, get(ctx, arg0, 176, 0));
  set(ctx, arg0, 177, get(ctx, arg0, 177, 0));
  set(ctx, arg0, 178, get(ctx, arg0, 178, 0));
  set(ctx, arg0, 179, get(ctx, arg0, 179, 0));
  set(ctx, arg0, 180, get(ctx, arg0, 180, 0));
  set(ctx, arg0, 181, get(ctx, arg0, 181, 0));
  set(ctx, arg0, 182, get(ctx, arg0, 182, 0));
  set(ctx, arg0, 183, get(ctx, arg0, 183, 0));
  set(ctx, arg0, 184, get(ctx, arg0, 184, 0));
  set(ctx, arg0, 185, get(ctx, arg0, 185, 0));
  set(ctx, arg0, 186, get(ctx, arg0, 186, 0));
  set(ctx, arg0, 187, get(ctx, arg0, 187, 0));
  set(ctx, arg0, 188, get(ctx, arg0, 188, 0));
  set(ctx, arg0, 189, get(ctx, arg0, 189, 0));
  set(ctx, arg0, 190, get(ctx, arg0, 190, 0));
  set(ctx, arg0, 191, get(ctx, arg0, 191, 0));
  set(ctx, arg0, 192, get(ctx, arg0, 192, 0));
  set(ctx, arg0, 193, get(ctx, arg0, 193, 0));
  set(ctx, arg0, 194, get(ctx, arg0, 194, 0));
  set(ctx, arg0, 195, get(ctx, arg0, 195, 0));
  set(ctx, arg0, 196, get(ctx, arg0, 196, 0));
  set(ctx, arg0, 197, get(ctx, arg0, 197, 0));
  set(ctx, arg0, 198, get(ctx, arg0, 198, 0));
  set(ctx, arg0, 199, get(ctx, arg0, 199, 0));
  set(ctx, arg0, 200, get(ctx, arg0, 200, 0));
  set(ctx, arg0, 201, get(ctx, arg0, 201, 0));
  set(ctx, arg0, 202, get(ctx, arg0, 202, 0));
  set(ctx, arg0, 203, get(ctx, arg0, 203, 0));
  set(ctx, arg0, 204, get(ctx, arg0, 204, 0));
  set(ctx, arg0, 205, get(ctx, arg0, 205, 0));
  set(ctx, arg0, 206, get(ctx, arg0, 206, 0));
  set(ctx, arg0, 207, get(ctx, arg0, 207, 0));
  set(ctx, arg0, 208, get(ctx, arg0, 208, 0));
  set(ctx, arg0, 209, get(ctx, arg0, 209, 0));
  set(ctx, arg0, 210, get(ctx, arg0, 210, 0));
  set(ctx, arg0, 211, get(ctx, arg0, 211, 0));
  set(ctx, arg0, 212, get(ctx, arg0, 212, 0));
  set(ctx, arg0, 213, get(ctx, arg0, 213, 0));
  set(ctx, arg0, 214, get(ctx, arg0, 214, 0));
  set(ctx, arg0, 215, get(ctx, arg0, 215, 0));
  set(ctx, arg0, 216, get(ctx, arg0, 216, 0));
  set(ctx, arg0, 217, get(ctx, arg0, 217, 0));
  set(ctx, arg0, 218, get(ctx, arg0, 218, 0));
  set(ctx, arg0, 219, get(ctx, arg0, 219, 0));
  set(ctx, arg0, 220, get(ctx, arg0, 220, 0));
  set(ctx, arg0, 221, get(ctx, arg0, 221, 0));
  set(ctx, arg0, 222, get(ctx, arg0, 222, 0));
  set(ctx, arg0, 223, get(ctx, arg0, 223, 0));
  set(ctx, arg0, 224, get(ctx, arg0, 224, 0));
  set(ctx, arg0, 225, get(ctx, arg0, 225, 0));
  set(ctx, arg0, 226, get(ctx, arg0, 226, 0));
  set(ctx, arg0, 227, get(ctx, arg0, 227, 0));
  set(ctx, arg0, 228, get(ctx, arg0, 228, 0));
  set(ctx, arg0, 229, get(ctx, arg0, 229, 0));
  set(ctx, arg0, 230, get(ctx, arg0, 230, 0));
  set(ctx, arg0, 231, get(ctx, arg0, 231, 0));
  set(ctx, arg0, 232, get(ctx, arg0, 232, 0));
  set(ctx, arg0, 233, get(ctx, arg0, 233, 0));
  set(ctx, arg0, 234, get(ctx, arg0, 234, 0));
  set(ctx, arg0, 235, get(ctx, arg0, 235, 0));
  set(ctx, arg0, 236, get(ctx, arg0, 236, 0));
  set(ctx, arg0, 237, get(ctx, arg0, 237, 0));
  set(ctx, arg0, 238, get(ctx, arg0, 238, 0));
  set(ctx, arg0, 239, get(ctx, arg0, 239, 0));
  set(ctx, arg0, 240, get(ctx, arg0, 240, 0));
  set(ctx, arg0, 241, get(ctx, arg0, 241, 0));
  set(ctx, arg0, 242, get(ctx, arg0, 242, 0));
  set(ctx, arg0, 243, get(ctx, arg0, 243, 0));
  set(ctx, arg0, 244, get(ctx, arg0, 244, 0));
  set(ctx, arg0, 245, get(ctx, arg0, 245, 0));
  set(ctx, arg0, 246, get(ctx, arg0, 246, 0));
  set(ctx, arg0, 247, get(ctx, arg0, 247, 0));
  set(ctx, arg0, 248, get(ctx, arg0, 248, 0));
  set(ctx, arg0, 249, get(ctx, arg0, 249, 0));
  set(ctx, arg0, 250, get(ctx, arg0, 250, 0));
  set(ctx, arg0, 251, get(ctx, arg0, 251, 0));
  set(ctx, arg0, 252, get(ctx, arg0, 252, 0));
  set(ctx, arg0, 253, get(ctx, arg0, 253, 0));
  set(ctx, arg0, 254, get(ctx, arg0, 254, 0));
  set(ctx, arg0, 255, get(ctx, arg0, 255, 0));
  set(ctx, arg0, 256, get(ctx, arg0, 256, 0));
  set(ctx, arg0, 257, get(ctx, arg0, 257, 0));
  set(ctx, arg0, 258, get(ctx, arg0, 258, 0));
  set(ctx, arg0, 259, get(ctx, arg0, 259, 0));
  set(ctx, arg0, 260, get(ctx, arg0, 260, 0));
  set(ctx, arg0, 261, get(ctx, arg0, 261, 0));
  set(ctx, arg0, 262, get(ctx, arg0, 262, 0));
  set(ctx, arg0, 263, get(ctx, arg0, 263, 0));
  set(ctx, arg0, 264, get(ctx, arg0, 264, 0));
  set(ctx, arg0, 265, get(ctx, arg0, 265, 0));
  set(ctx, arg0, 266, get(ctx, arg0, 266, 0));
  set(ctx, arg0, 267, get(ctx, arg0, 267, 0));
  set(ctx, arg0, 268, get(ctx, arg0, 268, 0));
  set(ctx, arg0, 269, get(ctx, arg0, 269, 0));
  set(ctx, arg0, 270, get(ctx, arg0, 270, 0));
  set(ctx, arg0, 271, get(ctx, arg0, 271, 0));
  set(ctx, arg0, 272, get(ctx, arg0, 272, 0));
  set(ctx, arg0, 273, get(ctx, arg0, 273, 0));
  set(ctx, arg0, 274, get(ctx, arg0, 274, 0));
  set(ctx, arg0, 275, get(ctx, arg0, 275, 0));
  set(ctx, arg0, 276, get(ctx, arg0, 276, 0));
  set(ctx, arg0, 277, get(ctx, arg0, 277, 0));
  set(ctx, arg0, 278, get(ctx, arg0, 278, 0));
  set(ctx, arg0, 279, get(ctx, arg0, 279, 0));
  set(ctx, arg0, 280, get(ctx, arg0, 280, 0));
  set(ctx, arg0, 281, get(ctx, arg0, 281, 0));
  set(ctx, arg0, 282, get(ctx, arg0, 282, 0));
  set(ctx, arg0, 283, get(ctx, arg0, 283, 0));
  set(ctx, arg0, 284, get(ctx, arg0, 284, 0));
  set(ctx, arg0, 285, get(ctx, arg0, 285, 0));
  set(ctx, arg0, 286, get(ctx, arg0, 286, 0));
  set(ctx, arg0, 287, get(ctx, arg0, 287, 0));
  set(ctx, arg0, 288, get(ctx, arg0, 288, 0));
  set(ctx, arg0, 289, get(ctx, arg0, 289, 0));
  set(ctx, arg0, 290, get(ctx, arg0, 290, 0));
  set(ctx, arg0, 291, get(ctx, arg0, 291, 0));
  set(ctx, arg0, 292, get(ctx, arg0, 292, 0));
  set(ctx, arg0, 293, get(ctx, arg0, 293, 0));
  set(ctx, arg0, 294, get(ctx, arg0, 294, 0));
  set(ctx, arg0, 295, get(ctx, arg0, 295, 0));
  set(ctx, arg0, 296, get(ctx, arg0, 296, 0));
  set(ctx, arg0, 297, get(ctx, arg0, 297, 0));
  set(ctx, arg0, 298, get(ctx, arg0, 298, 0));
  set(ctx, arg0, 299, get(ctx, arg0, 299, 0));
  set(ctx, arg0, 300, get(ctx, arg0, 300, 0));
  set(ctx, arg0, 301, get(ctx, arg0, 301, 0));
  set(ctx, arg0, 302, get(ctx, arg0, 302, 0));
  set(ctx, arg0, 303, get(ctx, arg0, 303, 0));
  set(ctx, arg0, 304, get(ctx, arg0, 304, 0));
  set(ctx, arg0, 305, get(ctx, arg0, 305, 0));
  set(ctx, arg0, 306, get(ctx, arg0, 306, 0));
  set(ctx, arg0, 307, get(ctx, arg0, 307, 0));
  set(ctx, arg0, 308, get(ctx, arg0, 308, 0));
  set(ctx, arg0, 309, get(ctx, arg0, 309, 0));
  set(ctx, arg0, 310, get(ctx, arg0, 310, 0));
  set(ctx, arg0, 311, get(ctx, arg0, 311, 0));
  set(ctx, arg0, 312, get(ctx, arg0, 312, 0));
  set(ctx, arg0, 313, get(ctx, arg0, 313, 0));
  set(ctx, arg0, 314, get(ctx, arg0, 314, 0));
  set(ctx, arg0, 315, get(ctx, arg0, 315, 0));
  set(ctx, arg0, 316, get(ctx, arg0, 316, 0));
  set(ctx, arg0, 317, get(ctx, arg0, 317, 0));
  set(ctx, arg0, 318, get(ctx, arg0, 318, 0));
  set(ctx, arg0, 319, get(ctx, arg0, 319, 0));
  set(ctx, arg0, 320, get(ctx, arg0, 320, 0));
  set(ctx, arg0, 321, get(ctx, arg0, 321, 0));
  set(ctx, arg0, 322, get(ctx, arg0, 322, 0));
  set(ctx, arg0, 323, get(ctx, arg0, 323, 0));
  set(ctx, arg0, 324, get(ctx, arg0, 324, 0));
  set(ctx, arg0, 325, get(ctx, arg0, 325, 0));
  set(ctx, arg0, 326, get(ctx, arg0, 326, 0));
  set(ctx, arg0, 327, get(ctx, arg0, 327, 0));
  set(ctx, arg0, 328, get(ctx, arg0, 328, 0));
  set(ctx, arg0, 329, get(ctx, arg0, 329, 0));
  set(ctx, arg0, 330, get(ctx, arg0, 330, 0));
  set(ctx, arg0, 331, get(ctx, arg0, 331, 0));
  set(ctx, arg0, 332, get(ctx, arg0, 332, 0));
  set(ctx, arg0, 333, get(ctx, arg0, 333, 0));
  set(ctx, arg0, 334, get(ctx, arg0, 334, 0));
  set(ctx, arg0, 335, get(ctx, arg0, 335, 0));
  set(ctx, arg0, 336, Val(0));
  set(ctx, arg0, 337, Val(0));
  set(ctx, arg0, 338, Val(0));
  set(ctx, arg0, 339, Val(0));
  set(ctx, arg0, 340, Val(0));
  set(ctx, arg0, 341, Val(0));
  set(ctx, arg0, 342, Val(0));
  set(ctx, arg0, 343, Val(0));
  set(ctx, arg0, 344, Val(0));
  set(ctx, arg0, 345, Val(0));
  set(ctx, arg0, 346, Val(0));
  set(ctx, arg0, 347, Val(0));
  set(ctx, arg0, 348, Val(0));
  set(ctx, arg0, 349, Val(0));
  set(ctx, arg0, 350, Val(0));
  set(ctx, arg0, 351, Val(0));
  set(ctx, arg0, 352, Val(0));
  set(ctx, arg0, 353, Val(0));
  set(ctx, arg0, 354, Val(0));
  set(ctx, arg0, 355, Val(0));
  set(ctx, arg0, 356, Val(0));
  set(ctx, arg0, 357, Val(0));
  set(ctx, arg0, 358, Val(0));
  set(ctx, arg0, 359, Val(0));
  set(ctx, arg0, 360, Val(0));
  set(ctx, arg0, 361, Val(0));
  set(ctx, arg0, 362, Val(0));
  set(ctx, arg0, 363, Val(0));
  set(ctx, arg0, 364, Val(0));
  set(ctx, arg0, 365, Val(0));
  set(ctx, arg0, 366, Val(0));
  set(ctx, arg0, 367, Val(0));
  set(ctx, arg0, 368, Val(0));
  set(ctx, arg0, 369, Val(0));
  set(ctx, arg0, 370, Val(0));
  set(ctx, arg0, 371, Val(0));
  set(ctx, arg0, 372, Val(0));
  set(ctx, arg0, 373, Val(0));
  set(ctx, arg0, 374, Val(0));
  set(ctx, arg0, 375, Val(0));
  set(ctx, arg0, 376, Val(0));
  set(ctx, arg0, 377, Val(0));
  set(ctx, arg0, 378, Val(0));
  set(ctx, arg0, 379, Val(0));
  set(ctx, arg0, 380, Val(0));
  set(ctx, arg0, 381, Val(0));
  set(ctx, arg0, 382, Val(0));
  set(ctx, arg0, 383, Val(0));
  set(ctx, arg0, 384, Val(0));
  set(ctx, arg0, 385, Val(0));
  set(ctx, arg0, 386, Val(0));
  set(ctx, arg0, 387, Val(0));
  set(ctx, arg0, 388, Val(0));
  set(ctx, arg0, 389, Val(0));
  set(ctx, arg0, 390, Val(0));
  set(ctx, arg0, 391, Val(0));
  set(ctx, arg0, 392, Val(0));
  set(ctx, arg0, 393, Val(0));
  set(ctx, arg0, 394, Val(0));
  set(ctx, arg0, 395, Val(0));
  set(ctx, arg0, 396, Val(0));
  set(ctx, arg0, 397, Val(0));
  set(ctx, arg0, 398, Val(0));
  set(ctx, arg0, 399, Val(0));
  set(ctx, arg0, 400, Val(0));
  set(ctx, arg0, 401, Val(0));
  set(ctx, arg0, 402, Val(0));
  set(ctx, arg0, 403, Val(0));
  set(ctx, arg0, 404, Val(0));
  set(ctx, arg0, 405, Val(0));
  set(ctx, arg0, 406, Val(0));
  set(ctx, arg0, 407, Val(0));
  set(ctx, arg0, 408, Val(0));
  set(ctx, arg0, 409, Val(0));
  set(ctx, arg0, 410, Val(0));
  set(ctx, arg0, 411, Val(0));
  set(ctx, arg0, 412, Val(0));
  set(ctx, arg0, 413, Val(0));
  set(ctx, arg0, 414, Val(0));
  set(ctx, arg0, 415, Val(0));
  set(ctx, arg0, 416, Val(0));
  set(ctx, arg0, 417, Val(0));
  set(ctx, arg0, 418, Val(0));
  set(ctx, arg0, 419, Val(0));
  set(ctx, arg0, 420, Val(0));
  set(ctx, arg0, 421, Val(0));
  set(ctx, arg0, 422, Val(0));
  set(ctx, arg0, 423, Val(0));
  set(ctx, arg0, 424, Val(0));
  set(ctx, arg0, 425, Val(0));
  set(ctx, arg0, 426, Val(0));
  set(ctx, arg0, 427, Val(0));
  set(ctx, arg0, 428, Val(0));
  set(ctx, arg0, 429, Val(0));
  set(ctx, arg0, 430, Val(0));
  set(ctx, arg0, 431, Val(0));
  set(ctx, arg0, 432, Val(0));
  set(ctx, arg0, 433, Val(0));
  set(ctx, arg0, 434, Val(0));
  set(ctx, arg0, 435, Val(0));
  set(ctx, arg0, 436, Val(0));
  set(ctx, arg0, 437, Val(0));
  set(ctx, arg0, 438, Val(0));
  set(ctx, arg0, 439, Val(0));
  set(ctx, arg0, 440, Val(0));
  set(ctx, arg0, 441, Val(0));
  set(ctx, arg0, 442, Val(0));
  set(ctx, arg0, 443, Val(0));
  set(ctx, arg0, 444, Val(0));
  set(ctx, arg0, 445, Val(0));
  set(ctx, arg0, 446, Val(0));
  set(ctx, arg0, 447, Val(0));
  set(ctx, arg0, 448, Val(0));
  set(ctx, arg0, 449, Val(0));
  set(ctx, arg0, 450, Val(0));
  set(ctx, arg0, 451, Val(0));
  set(ctx, arg0, 452, Val(0));
  set(ctx, arg0, 453, Val(0));
  set(ctx, arg0, 454, Val(0));
  set(ctx, arg0, 455, Val(0));
  set(ctx, arg0, 456, Val(0));
  set(ctx, arg0, 457, Val(0));
  set(ctx, arg0, 458, Val(0));
  set(ctx, arg0, 459, Val(0));
  set(ctx, arg0, 460, Val(0));
  set(ctx, arg0, 461, Val(0));
  set(ctx, arg0, 462, Val(0));
  set(ctx, arg0, 463, Val(0));
  set(ctx, arg0, 464, Val(0));
  set(ctx, arg0, 465, Val(0));
  set(ctx, arg0, 466, Val(0));
  set(ctx, arg0, 467, Val(0));
  set(ctx, arg0, 468, Val(0));
  set(ctx, arg0, 469, Val(0));
  set(ctx, arg0, 470, Val(0));
  set(ctx, arg0, 471, Val(0));
  set(ctx, arg0, 472, Val(0));
  set(ctx, arg0, 473, Val(0));
  set(ctx, arg0, 474, Val(0));
  set(ctx, arg0, 475, Val(0));
  set(ctx, arg0, 476, Val(0));
  set(ctx, arg0, 477, Val(0));
  set(ctx, arg0, 478, Val(0));
  set(ctx, arg0, 479, Val(0));
  set(ctx, arg0, 480, Val(0));
  set(ctx, arg0, 481, Val(0));
  set(ctx, arg0, 482, Val(0));
  set(ctx, arg0, 483, Val(0));
  set(ctx, arg0, 484, Val(0));
  set(ctx, arg0, 485, Val(0));
  set(ctx, arg0, 486, Val(0));
  set(ctx, arg0, 487, Val(0));
  set(ctx, arg0, 488, Val(0));
  set(ctx, arg0, 489, Val(0));
  set(ctx, arg0, 490, Val(0));
  set(ctx, arg0, 491, Val(0));
  set(ctx, arg0, 492, Val(0));
  set(ctx, arg0, 493, Val(0));
  set(ctx, arg0, 494, Val(0));
  set(ctx, arg0, 495, Val(0));
  set(ctx, arg0, 496, Val(0));
  set(ctx, arg0, 497, Val(0));
  set(ctx, arg0, 498, Val(0));
  set(ctx, arg0, 499, Val(0));
  set(ctx, arg0, 500, Val(0));
  set(ctx, arg0, 501, Val(0));
  set(ctx, arg0, 502, Val(0));
  set(ctx, arg0, 503, Val(0));
  set(ctx, arg0, 504, Val(0));
  set(ctx, arg0, 505, Val(0));
  set(ctx, arg0, 506, Val(0));
  set(ctx, arg0, 507, Val(0));
  set(ctx, arg0, 508, Val(0));
  set(ctx, arg0, 509, Val(0));
  set(ctx, arg0, 510, Val(0));
  set(ctx, arg0, 511, Val(0));
  set(ctx, arg0, 512, Val(0));
  set(ctx, arg0, 513, Val(0));
  set(ctx, arg0, 514, Val(0));
  set(ctx, arg0, 515, Val(0));
  set(ctx, arg0, 516, Val(0));
  set(ctx, arg0, 517, Val(0));
  set(ctx, arg0, 518, Val(0));
  set(ctx, arg0, 519, Val(0));
  set(ctx, arg0, 520, Val(0));
  set(ctx, arg0, 521, Val(0));
  set(ctx, arg0, 522, Val(0));
  set(ctx, arg0, 523, Val(0));
  set(ctx, arg0, 524, Val(0));
  set(ctx, arg0, 525, Val(0));
  set(ctx, arg0, 526, Val(0));
  set(ctx, arg0, 527, Val(0));
  set(ctx, arg0, 528, Val(0));
  set(ctx, arg0, 529, Val(0));
  set(ctx, arg0, 530, Val(0));
  set(ctx, arg0, 531, Val(0));
  set(ctx, arg0, 532, Val(0));
  set(ctx, arg0, 533, Val(0));
  set(ctx, arg0, 534, Val(0));
  set(ctx, arg0, 535, Val(0));
  set(ctx, arg0, 536, Val(0));
  set(ctx, arg0, 537, Val(0));
  set(ctx, arg0, 538, Val(0));
  set(ctx, arg0, 539, Val(0));
  set(ctx, arg0, 540, Val(0));
  set(ctx, arg0, 541, Val(0));
  set(ctx, arg0, 542, Val(0));
  set(ctx, arg0, 543, Val(0));
  set(ctx, arg0, 544, Val(0));
  set(ctx, arg0, 545, Val(0));
  set(ctx, arg0, 546, Val(0));
  set(ctx, arg0, 547, Val(0));
  set(ctx, arg0, 548, Val(0));
  set(ctx, arg0, 549, Val(0));
  set(ctx, arg0, 550, Val(0));
  set(ctx, arg0, 551, Val(0));
  set(ctx, arg0, 552, Val(0));
  set(ctx, arg0, 553, Val(0));
  set(ctx, arg0, 554, Val(0));
  set(ctx, arg0, 555, Val(0));
  set(ctx, arg0, 556, Val(0));
  set(ctx, arg0, 557, Val(0));
  set(ctx, arg0, 558, Val(0));
  set(ctx, arg0, 559, Val(0));
  set(ctx, arg0, 560, Val(0));
  set(ctx, arg0, 561, Val(0));
  set(ctx, arg0, 562, Val(0));
  set(ctx, arg0, 563, Val(0));
  set(ctx, arg0, 564, Val(0));
  set(ctx, arg0, 565, Val(0));
  set(ctx, arg0, 566, Val(0));
  set(ctx, arg0, 567, Val(0));
  set(ctx, arg0, 568, Val(0));
  set(ctx, arg0, 569, Val(0));
  set(ctx, arg0, 570, Val(0));
  set(ctx, arg0, 571, Val(0));
  set(ctx, arg0, 572, Val(0));
  set(ctx, arg0, 573, Val(0));
  set(ctx, arg0, 574, Val(0));
  set(ctx, arg0, 575, Val(0));
  set(ctx, arg0, 576, Val(0));
  set(ctx, arg0, 577, Val(0));
  set(ctx, arg0, 578, Val(0));
  set(ctx, arg0, 579, Val(0));
  set(ctx, arg0, 580, Val(0));
  set(ctx, arg0, 581, Val(0));
  set(ctx, arg0, 582, Val(0));
  set(ctx, arg0, 583, Val(0));
  set(ctx, arg0, 584, Val(0));
  set(ctx, arg0, 585, Val(0));
  set(ctx, arg0, 586, Val(0));
  set(ctx, arg0, 587, Val(0));
  set(ctx, arg0, 588, Val(0));
  set(ctx, arg0, 589, Val(0));
  set(ctx, arg0, 590, Val(0));
  set(ctx, arg0, 591, Val(0));
  set(ctx, arg0, 592, Val(0));
  set(ctx, arg0, 593, Val(0));
  set(ctx, arg0, 594, Val(0));
  set(ctx, arg0, 595, Val(0));
  set(ctx, arg0, 596, Val(0));
  set(ctx, arg0, 597, Val(0));
  set(ctx, arg0, 598, Val(0));
  set(ctx, arg0, 599, Val(0));
  set(ctx, arg0, 600, Val(0));
  set(ctx, arg0, 601, Val(0));
  set(ctx, arg0, 602, Val(0));
  set(ctx, arg0, 603, Val(0));
  set(ctx, arg0, 604, Val(0));
  set(ctx, arg0, 605, Val(0));
  set(ctx, arg0, 606, Val(0));
  set(ctx, arg0, 607, Val(0));
  set(ctx, arg0, 608, Val(0));
  set(ctx, arg0, 609, Val(0));
  set(ctx, arg0, 610, Val(0));
  set(ctx, arg0, 611, Val(0));
  set(ctx, arg0, 612, Val(0));
  set(ctx, arg0, 613, Val(0));
  set(ctx, arg0, 614, Val(0));
  set(ctx, arg0, 615, Val(0));
  set(ctx, arg0, 616, Val(0));
  set(ctx, arg0, 617, Val(0));
  set(ctx, arg0, 618, Val(0));
  set(ctx, arg0, 619, Val(0));
  set(ctx, arg0, 620, Val(0));
  set(ctx, arg0, 621, Val(0));
  set(ctx, arg0, 622, Val(0));
  set(ctx, arg0, 623, Val(0));
  set(ctx, arg0, 624, Val(0));
  set(ctx, arg0, 625, Val(0));
  set(ctx, arg0, 626, Val(0));
  set(ctx, arg0, 627, Val(0));
  set(ctx, arg0, 628, Val(0));
  set(ctx, arg0, 629, Val(0));
  set(ctx, arg0, 630, Val(0));
  set(ctx, arg0, 631, Val(0));
  set(ctx, arg0, 632, Val(0));
  set(ctx, arg0, 633, Val(0));
  set(ctx, arg0, 634, Val(0));
  set(ctx, arg0, 635, Val(0));
  set(ctx, arg0, 636, Val(0));
  set(ctx, arg0, 637, Val(0));
  set(ctx, arg0, 638, Val(0));
  set(ctx, arg0, 639, Val(0));
  set(ctx, arg0, 640, Val(0));
  set(ctx, arg0, 641, Val(0));
  set(ctx, arg0, 642, Val(0));
  set(ctx, arg0, 643, Val(0));
  set(ctx, arg0, 644, Val(0));
  set(ctx, arg0, 645, Val(0));
  set(ctx, arg0, 646, Val(0));
  set(ctx, arg0, 647, Val(0));
  set(ctx, arg0, 648, Val(0));
  set(ctx, arg0, 649, Val(0));
  set(ctx, arg0, 650, Val(0));
  set(ctx, arg0, 651, Val(0));
  set(ctx, arg0, 652, Val(0));
  set(ctx, arg0, 653, Val(0));
  set(ctx, arg0, 654, Val(0));
  set(ctx, arg0, 655, Val(0));
  set(ctx, arg0, 656, Val(0));
  set(ctx, arg0, 657, Val(0));
  set(ctx, arg0, 658, Val(0));
  set(ctx, arg0, 659, Val(0));
  set(ctx, arg0, 660, Val(0));
  set(ctx, arg0, 661, Val(0));
  set(ctx, arg0, 662, Val(0));
  set(ctx, arg0, 663, Val(0));
  set(ctx, arg0, 664, Val(0));
  set(ctx, arg0, 665, Val(0));
  set(ctx, arg0, 666, Val(0));
  set(ctx, arg0, 667, Val(0));
  set(ctx, arg0, 668, Val(0));
  set(ctx, arg0, 669, Val(0));
  set(ctx, arg0, 670, Val(0));
  set(ctx, arg0, 671, Val(0));
  set(ctx, arg0, 672, Val(0));
  set(ctx, arg0, 673, Val(0));
  set(ctx, arg0, 674, Val(0));
  set(ctx, arg0, 675, Val(0));
  set(ctx, arg0, 676, Val(0));
  set(ctx, arg0, 677, Val(0));
  set(ctx, arg0, 678, Val(0));
  set(ctx, arg0, 679, Val(0));
  set(ctx, arg0, 680, Val(0));
  set(ctx, arg0, 681, Val(0));
  set(ctx, arg0, 682, Val(0));
  set(ctx, arg0, 683, Val(0));
  set(ctx, arg0, 684, Val(0));
  set(ctx, arg0, 685, Val(0));
  set(ctx, arg0, 686, Val(0));
  set(ctx, arg0, 687, Val(0));
  set(ctx, arg0, 688, Val(0));
  set(ctx, arg0, 689, Val(0));
  set(ctx, arg0, 690, Val(0));
  set(ctx, arg0, 691, Val(0));
  set(ctx, arg0, 692, Val(0));
  set(ctx, arg0, 693, Val(0));
  set(ctx, arg0, 694, Val(0));
  set(ctx, arg0, 695, Val(0));
  set(ctx, arg0, 696, Val(0));
  set(ctx, arg0, 697, Val(0));
  set(ctx, arg0, 698, Val(0));
  set(ctx, arg0, 699, Val(0));
  set(ctx, arg0, 700, Val(0));
  set(ctx, arg0, 701, Val(0));
  set(ctx, arg0, 702, Val(0));
  set(ctx, arg0, 703, Val(0));
  set(ctx, arg0, 704, Val(0));
  set(ctx, arg0, 705, Val(0));
  set(ctx, arg0, 706, Val(0));
  set(ctx, arg0, 707, Val(0));
  set(ctx, arg0, 708, Val(0));
  set(ctx, arg0, 709, Val(0));
  set(ctx, arg0, 710, Val(0));
  set(ctx, arg0, 711, Val(0));
  set(ctx, arg0, 712, Val(0));
  set(ctx, arg0, 713, Val(0));
  set(ctx, arg0, 714, Val(0));
  set(ctx, arg0, 715, Val(0));
  set(ctx, arg0, 716, Val(0));
  set(ctx, arg0, 717, Val(0));
  set(ctx, arg0, 718, Val(0));
  set(ctx, arg0, 719, Val(0));
  set(ctx, arg0, 720, Val(0));
  set(ctx, arg0, 721, Val(0));
  set(ctx, arg0, 722, Val(0));
  set(ctx, arg0, 723, Val(0));
  set(ctx, arg0, 724, Val(0));
  set(ctx, arg0, 725, Val(0));
  set(ctx, arg0, 726, Val(0));
  set(ctx, arg0, 727, Val(0));
  set(ctx, arg0, 728, Val(0));
  set(ctx, arg0, 729, Val(0));
  set(ctx, arg0, 730, Val(0));
  set(ctx, arg0, 731, Val(0));
  set(ctx, arg0, 732, Val(0));
  set(ctx, arg0, 733, Val(0));
  set(ctx, arg0, 734, Val(0));
  set(ctx, arg0, 735, Val(0));
  set(ctx, arg0, 736, Val(0));
  set(ctx, arg0, 737, Val(0));
  set(ctx, arg0, 738, Val(0));
  set(ctx, arg0, 739, Val(0));
  set(ctx, arg0, 740, Val(0));
  set(ctx, arg0, 741, Val(0));
  set(ctx, arg0, 742, Val(0));
  set(ctx, arg0, 743, Val(0));
  set(ctx, arg0, 744, Val(0));
  set(ctx, arg0, 745, Val(0));
  set(ctx, arg0, 746, Val(0));
  set(ctx, arg0, 747, Val(0));
  set(ctx, arg0, 748, Val(0));
  set(ctx, arg0, 749, Val(0));
  set(ctx, arg0, 750, Val(0));
  set(ctx, arg0, 751, Val(0));
  set(ctx, arg0, 752, Val(0));
  set(ctx, arg0, 753, Val(0));
  set(ctx, arg0, 754, Val(0));
  set(ctx, arg0, 755, Val(0));
  set(ctx, arg0, 756, Val(0));
  set(ctx, arg0, 757, Val(0));
  set(ctx, arg0, 758, Val(0));
  set(ctx, arg0, 759, Val(0));
  set(ctx, arg0, 760, Val(0));
  set(ctx, arg0, 761, Val(0));
  set(ctx, arg0, 762, Val(0));
  set(ctx, arg0, 763, Val(0));
  set(ctx, arg0, 764, Val(0));
  set(ctx, arg0, 765, Val(0));
  set(ctx, arg0, 766, Val(0));
  set(ctx, arg0, 767, Val(0));
  set(ctx, arg0, 768, Val(0));
  set(ctx, arg0, 769, Val(0));
  set(ctx, arg0, 770, Val(0));
  set(ctx, arg0, 771, Val(0));
  set(ctx, arg0, 772, Val(0));
  set(ctx, arg0, 773, Val(0));
  set(ctx, arg0, 774, Val(0));
  set(ctx, arg0, 775, Val(0));
  set(ctx, arg0, 776, Val(0));
  set(ctx, arg0, 777, Val(0));
  set(ctx, arg0, 778, Val(0));
  set(ctx, arg0, 779, Val(0));
  set(ctx, arg0, 780, Val(0));
  set(ctx, arg0, 781, Val(0));
  set(ctx, arg0, 782, Val(0));
  set(ctx, arg0, 783, Val(0));
  set(ctx, arg0, 784, Val(0));
  set(ctx, arg0, 785, Val(0));
  set(ctx, arg0, 786, Val(0));
  set(ctx, arg0, 787, Val(0));
  set(ctx, arg0, 788, Val(0));
  set(ctx, arg0, 789, Val(0));
  set(ctx, arg0, 790, Val(0));
  set(ctx, arg0, 791, Val(0));
  set(ctx, arg0, 792, Val(0));
  set(ctx, arg0, 793, Val(0));
  set(ctx, arg0, 794, Val(0));
  set(ctx, arg0, 795, Val(0));
  set(ctx, arg0, 796, Val(0));
  set(ctx, arg0, 797, Val(0));
  set(ctx, arg0, 798, Val(0));
  set(ctx, arg0, 799, Val(0));
  set(ctx, arg0, 800, Val(0));
  set(ctx, arg0, 801, Val(0));
  set(ctx, arg0, 802, Val(0));
  set(ctx, arg0, 803, Val(0));
  set(ctx, arg0, 804, Val(0));
  set(ctx, arg0, 805, Val(0));
  set(ctx, arg0, 806, Val(0));
  set(ctx, arg0, 807, Val(0));
  set(ctx, arg0, 808, Val(0));
  set(ctx, arg0, 809, Val(0));
  set(ctx, arg0, 810, Val(0));
  set(ctx, arg0, 811, Val(0));
  set(ctx, arg0, 812, Val(0));
  set(ctx, arg0, 813, Val(0));
  set(ctx, arg0, 814, Val(0));
  set(ctx, arg0, 815, Val(0));
  // TopState(zirgen/circuit/keccak/top.zir:40)
  set(ctx, arg0, 816, get(ctx, arg0, 816, 1));
  set(ctx, arg0, 817, get(ctx, arg0, 817, 1));
  set(ctx, arg0, 818, get(ctx, arg0, 818, 1));
  set(ctx, arg0, 819, get(ctx, arg0, 819, 1));
  set(ctx, arg0, 820, get(ctx, arg0, 820, 1));
  set(ctx, arg0, 821, get(ctx, arg0, 821, 1));
  set(ctx, arg0, 822, get(ctx, arg0, 822, 1));
  set(ctx, arg0, 823, get(ctx, arg0, 823, 1));
  set(ctx, arg0, 824, get(ctx, arg0, 824, 1));
  set(ctx, arg0, 825, get(ctx, arg0, 825, 1));
  set(ctx, arg0, 826, get(ctx, arg0, 826, 1));
  set(ctx, arg0, 827, get(ctx, arg0, 827, 1));
  set(ctx, arg0, 828, get(ctx, arg0, 828, 1));
  set(ctx, arg0, 829, get(ctx, arg0, 829, 1));
  set(ctx, arg0, 830, get(ctx, arg0, 830, 1));
  set(ctx, arg0, 831, get(ctx, arg0, 831, 1));
  set(ctx, arg0, 832, get(ctx, arg0, 832, 1));
  set(ctx, arg0, 833, get(ctx, arg0, 833, 1));
  set(ctx, arg0, 834, get(ctx, arg0, 834, 1));
  set(ctx, arg0, 835, get(ctx, arg0, 835, 1));
  set(ctx, arg0, 836, get(ctx, arg0, 836, 1));
  set(ctx, arg0, 837, get(ctx, arg0, 837, 1));
  set(ctx, arg0, 838, get(ctx, arg0, 838, 1));
  set(ctx, arg0, 839, get(ctx, arg0, 839, 1));
  set(ctx, arg0, 840, get(ctx, arg0, 840, 1));
  set(ctx, arg0, 841, get(ctx, arg0, 841, 1));
  set(ctx, arg0, 842, get(ctx, arg0, 842, 1));
  set(ctx, arg0, 843, get(ctx, arg0, 843, 1));
  set(ctx, arg0, 844, get(ctx, arg0, 844, 1));
  set(ctx, arg0, 845, get(ctx, arg0, 845, 1));
  set(ctx, arg0, 846, get(ctx, arg0, 846, 1));
  set(ctx, arg0, 847, get(ctx, arg0, 847, 1));
  set(ctx, arg0, 848, get(ctx, arg0, 848, 1));
  set(ctx, arg0, 849, get(ctx, arg0, 849, 1));
  set(ctx, arg0, 850, get(ctx, arg0, 850, 1));
  set(ctx, arg0, 851, get(ctx, arg0, 851, 1));
  set(ctx, arg0, 852, get(ctx, arg0, 852, 1));
  set(ctx, arg0, 853, get(ctx, arg0, 853, 1));
  set(ctx, arg0, 854, get(ctx, arg0, 854, 1));
  set(ctx, arg0, 855, get(ctx, arg0, 855, 1));
  set(ctx, arg0, 856, get(ctx, arg0, 856, 1));
  set(ctx, arg0, 857, get(ctx, arg0, 857, 1));
  set(ctx, arg0, 858, get(ctx, arg0, 858, 1));
  set(ctx, arg0, 859, get(ctx, arg0, 859, 1));
  set(ctx, arg0, 860, get(ctx, arg0, 860, 1));
  set(ctx, arg0, 861, get(ctx, arg0, 861, 1));
  set(ctx, arg0, 862, get(ctx, arg0, 862, 1));
  set(ctx, arg0, 863, get(ctx, arg0, 863, 1));
  set(ctx, arg0, 864, get(ctx, arg0, 864, 1));
  set(ctx, arg0, 865, get(ctx, arg0, 865, 1));
  set(ctx, arg0, 866, get(ctx, arg0, 866, 1));
  set(ctx, arg0, 867, get(ctx, arg0, 867, 1));
  set(ctx, arg0, 868, get(ctx, arg0, 868, 1));
  set(ctx, arg0, 869, get(ctx, arg0, 869, 1));
  set(ctx, arg0, 870, get(ctx, arg0, 870, 1));
  set(ctx, arg0, 871, get(ctx, arg0, 871, 1));
  set(ctx, arg0, 872, get(ctx, arg0, 872, 1));
  set(ctx, arg0, 873, get(ctx, arg0, 873, 1));
  set(ctx, arg0, 874, get(ctx, arg0, 874, 1));
  set(ctx, arg0, 875, get(ctx, arg0, 875, 1));
  set(ctx, arg0, 876, get(ctx, arg0, 876, 1));
  set(ctx, arg0, 877, get(ctx, arg0, 877, 1));
  set(ctx, arg0, 878, get(ctx, arg0, 878, 1));
  set(ctx, arg0, 879, get(ctx, arg0, 879, 1));
  set(ctx, arg0, 880, get(ctx, arg0, 880, 1));
  set(ctx, arg0, 881, get(ctx, arg0, 881, 1));
  set(ctx, arg0, 882, get(ctx, arg0, 882, 1));
  set(ctx, arg0, 883, get(ctx, arg0, 883, 1));
  set(ctx, arg0, 884, get(ctx, arg0, 884, 1));
  set(ctx, arg0, 885, get(ctx, arg0, 885, 1));
  set(ctx, arg0, 886, get(ctx, arg0, 886, 1));
  set(ctx, arg0, 887, get(ctx, arg0, 887, 1));
  set(ctx, arg0, 888, get(ctx, arg0, 888, 1));
  set(ctx, arg0, 889, get(ctx, arg0, 889, 1));
  set(ctx, arg0, 890, get(ctx, arg0, 890, 1));
  set(ctx, arg0, 891, get(ctx, arg0, 891, 1));
  set(ctx, arg0, 892, get(ctx, arg0, 892, 1));
  set(ctx, arg0, 893, get(ctx, arg0, 893, 1));
  set(ctx, arg0, 894, get(ctx, arg0, 894, 1));
  set(ctx, arg0, 895, get(ctx, arg0, 895, 1));
  set(ctx, arg0, 896, get(ctx, arg0, 896, 1));
  set(ctx, arg0, 897, get(ctx, arg0, 897, 1));
  set(ctx, arg0, 898, get(ctx, arg0, 898, 1));
  set(ctx, arg0, 899, get(ctx, arg0, 899, 1));
  set(ctx, arg0, 900, get(ctx, arg0, 900, 1));
  set(ctx, arg0, 901, get(ctx, arg0, 901, 1));
  set(ctx, arg0, 902, get(ctx, arg0, 902, 1));
  set(ctx, arg0, 903, get(ctx, arg0, 903, 1));
  set(ctx, arg0, 904, get(ctx, arg0, 904, 1));
  set(ctx, arg0, 905, get(ctx, arg0, 905, 1));
  set(ctx, arg0, 906, get(ctx, arg0, 906, 1));
  set(ctx, arg0, 907, get(ctx, arg0, 907, 1));
  set(ctx, arg0, 908, get(ctx, arg0, 908, 1));
  set(ctx, arg0, 909, get(ctx, arg0, 909, 1));
  set(ctx, arg0, 910, get(ctx, arg0, 910, 1));
  set(ctx, arg0, 911, get(ctx, arg0, 911, 1));
  set(ctx, arg0, 912, get(ctx, arg0, 912, 1));
  set(ctx, arg0, 913, get(ctx, arg0, 913, 1));
  set(ctx, arg0, 914, get(ctx, arg0, 914, 1));
  set(ctx, arg0, 915, get(ctx, arg0, 915, 1));
  // TopState(zirgen/circuit/keccak/top.zir:43)
  set(ctx, arg0, 916, get(ctx, arg0, 916, 1));
  set(ctx, arg0, 917, get(ctx, arg0, 917, 1));
  set(ctx, arg0, 918, get(ctx, arg0, 918, 1));
  set(ctx, arg0, 919, get(ctx, arg0, 919, 1));
  set(ctx, arg0, 920, get(ctx, arg0, 920, 1));
  set(ctx, arg0, 921, get(ctx, arg0, 921, 1));
  set(ctx, arg0, 922, get(ctx, arg0, 922, 1));
  set(ctx, arg0, 923, get(ctx, arg0, 923, 1));
  set(ctx, arg0, 924, get(ctx, arg0, 924, 1));
  set(ctx, arg0, 925, get(ctx, arg0, 925, 1));
  set(ctx, arg0, 926, get(ctx, arg0, 926, 1));
  set(ctx, arg0, 927, get(ctx, arg0, 927, 1));
  set(ctx, arg0, 928, get(ctx, arg0, 928, 1));
  set(ctx, arg0, 929, get(ctx, arg0, 929, 1));
  set(ctx, arg0, 930, get(ctx, arg0, 930, 1));
  set(ctx, arg0, 931, get(ctx, arg0, 931, 1));
  return;
}
__device__ void step_Top_45(ExecContext& ctx, MutableBuf arg0) {
  // TopState(zirgen/circuit/keccak/top.zir:36)
  // Top(zirgen/circuit/keccak/top.zir:501)
  Val x1 = get(ctx, arg0, 727, 2);
  Val x2 = get(ctx, arg0, 738, 2);
  Val x3 = get(ctx, arg0, 723, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x4 = (x2 * Val(2));
  // builtin Sub
  Val x5 = ((x2 + x3) - (x4 * x3));
  // builtin Mul
  Val x6 = (x1 * Val(2));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x7 = get(ctx, arg0, 728, 2);
  Val x8 = get(ctx, arg0, 739, 2);
  Val x9 = get(ctx, arg0, 724, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x10 = (x8 * Val(2));
  // builtin Sub
  Val x11 = ((x8 + x9) - (x10 * x9));
  // builtin Mul
  Val x12 = (x7 * Val(2));
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x13 = (((x7 + x11) - (x12 * x11)) * Val(2));
  // builtin Add
  Val x14 = (((x1 + x5) - (x6 * x5)) + x13);
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x15 = get(ctx, arg0, 729, 2);
  Val x16 = get(ctx, arg0, 740, 2);
  Val x17 = get(ctx, arg0, 725, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x18 = (x16 * Val(2));
  // builtin Sub
  Val x19 = ((x16 + x17) - (x18 * x17));
  // builtin Mul
  Val x20 = (x15 * Val(2));
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x21 = (((x15 + x19) - (x20 * x19)) * Val(4));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x22 = get(ctx, arg0, 730, 2);
  Val x23 = get(ctx, arg0, 741, 2);
  Val x24 = get(ctx, arg0, 726, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x25 = (x23 * Val(2));
  // builtin Sub
  Val x26 = ((x23 + x24) - (x25 * x24));
  // builtin Mul
  Val x27 = (x22 * Val(2));
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x28 = (((x22 + x26) - (x27 * x26)) * Val(8));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x29 = get(ctx, arg0, 731, 2);
  Val x30 = get(ctx, arg0, 742, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x31 = (x30 * Val(2));
  // builtin Sub
  Val x32 = ((x30 + x1) - (x31 * x1));
  // builtin Mul
  Val x33 = (x29 * Val(2));
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x34 = (((x29 + x32) - (x33 * x32)) * Val(16));
  // builtin Add
  Val x35 = (((x14 + x21) + x28) + x34);
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x36 = get(ctx, arg0, 732, 2);
  Val x37 = get(ctx, arg0, 743, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x38 = (x37 * Val(2));
  // builtin Sub
  Val x39 = ((x37 + x7) - (x38 * x7));
  // builtin Mul
  Val x40 = (x36 * Val(2));
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x41 = (((x36 + x39) - (x40 * x39)) * Val(32));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x42 = get(ctx, arg0, 733, 2);
  Val x43 = get(ctx, arg0, 744, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x44 = (x43 * Val(2));
  // builtin Sub
  Val x45 = ((x43 + x15) - (x44 * x15));
  // builtin Mul
  Val x46 = (x42 * Val(2));
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x47 = (((x42 + x45) - (x46 * x45)) * Val(64));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x48 = get(ctx, arg0, 734, 2);
  Val x49 = get(ctx, arg0, 745, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x50 = (x49 * Val(2));
  // builtin Sub
  Val x51 = ((x49 + x22) - (x50 * x22));
  // builtin Mul
  Val x52 = (x48 * Val(2));
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x53 = (((x48 + x51) - (x52 * x51)) * Val(128));
  // builtin Add
  Val x54 = (((x35 + x41) + x47) + x53);
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x55 = get(ctx, arg0, 735, 2);
  Val x56 = get(ctx, arg0, 746, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x57 = (x56 * Val(2));
  // builtin Sub
  Val x58 = ((x56 + x29) - (x57 * x29));
  Val x59 = ((x55 + x58) - ((x55 * Val(2)) * x58));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x60 = get(ctx, arg0, 736, 2);
  Val x61 = get(ctx, arg0, 747, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x62 = (x61 * Val(2));
  // builtin Sub
  Val x63 = ((x61 + x36) - (x62 * x36));
  Val x64 = ((x60 + x63) - ((x60 * Val(2)) * x63));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x65 = get(ctx, arg0, 737, 2);
  Val x66 = get(ctx, arg0, 748, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x67 = (x66 * Val(2));
  // builtin Sub
  Val x68 = ((x66 + x42) - (x67 * x42));
  Val x69 = ((x65 + x68) - ((x65 * Val(2)) * x68));
  // builtin Add
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x70 = (((x54 + (x59 * Val(256))) + (x64 * Val(512))) + (x69 * Val(1024)));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x71 = get(ctx, arg0, 749, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x72 = (x71 * Val(2));
  // builtin Sub
  Val x73 = ((x71 + x48) - (x72 * x48));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x74 = (((x2 + x73) - (x4 * x73)) * Val(2048));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x75 = get(ctx, arg0, 750, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x76 = (x75 * Val(2));
  // builtin Sub
  Val x77 = ((x75 + x55) - (x76 * x55));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x78 = (((x8 + x77) - (x10 * x77)) * Val(4096));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x79 = get(ctx, arg0, 751, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x80 = (x79 * Val(2));
  // builtin Sub
  Val x81 = ((x79 + x60) - (x80 * x60));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x82 = (((x16 + x81) - (x18 * x81)) * Val(8192));
  // builtin Add
  Val x83 = (((x70 + x74) + x78) + x82);
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x84 = get(ctx, arg0, 720, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x85 = (x84 * Val(2));
  // builtin Sub
  Val x86 = ((x84 + x65) - (x85 * x65));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x87 = (((x23 + x86) - (x25 * x86)) * Val(16384));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x88 = get(ctx, arg0, 721, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x89 = (x88 * Val(2));
  // builtin Sub
  Val x90 = ((x88 + x2) - (x89 * x2));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x91 = (((x30 + x90) - (x31 * x90)) * Val(32768));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x92 = get(ctx, arg0, 641, 0);
  Val x93 = get(ctx, arg0, 643, 0);
  Val x94 = get(ctx, arg0, 634, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x95 = (x93 * Val(2));
  // builtin Sub
  Val x96 = ((x93 + x94) - (x95 * x94));
  Val x97 = ((x92 + x96) - ((x92 * Val(2)) * x96));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x98 = get(ctx, arg0, 642, 0);
  Val x99 = get(ctx, arg0, 644, 0);
  Val x100 = get(ctx, arg0, 635, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x101 = (x99 * Val(2));
  // builtin Sub
  Val x102 = ((x99 + x100) - (x101 * x100));
  Val x103 = ((x98 + x102) - ((x98 * Val(2)) * x102));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x104 = get(ctx, arg0, 645, 0);
  Val x105 = get(ctx, arg0, 636, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x106 = (x104 * Val(2));
  // builtin Sub
  Val x107 = ((x104 + x105) - (x106 * x105));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x108 = (((x93 + x107) - (x95 * x107)) * Val(4));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x109 = get(ctx, arg0, 646, 0);
  Val x110 = get(ctx, arg0, 637, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x111 = (x109 * Val(2));
  // builtin Sub
  Val x112 = ((x109 + x110) - (x111 * x110));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x113 = (((x99 + x112) - (x101 * x112)) * Val(8));
  // builtin Add
  Val x114 = (((x97 + (x103 * Val(2))) + x108) + x113);
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x115 = get(ctx, arg0, 647, 0);
  Val x116 = get(ctx, arg0, 638, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x117 = (x115 * Val(2));
  // builtin Sub
  Val x118 = ((x115 + x116) - (x117 * x116));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x119 = (((x104 + x118) - (x106 * x118)) * Val(16));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x120 = get(ctx, arg0, 648, 0);
  Val x121 = get(ctx, arg0, 639, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x122 = (x120 * Val(2));
  // builtin Sub
  Val x123 = ((x120 + x121) - (x122 * x121));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x124 = (((x109 + x123) - (x111 * x123)) * Val(32));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x125 = get(ctx, arg0, 649, 0);
  Val x126 = get(ctx, arg0, 640, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x127 = (x125 * Val(2));
  // builtin Sub
  Val x128 = ((x125 + x126) - (x127 * x126));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x129 = (((x115 + x128) - (x117 * x128)) * Val(64));
  // builtin Add
  Val x130 = (((x114 + x119) + x124) + x129);
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x131 = get(ctx, arg0, 650, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x132 = (x131 * Val(2));
  // builtin Sub
  Val x133 = ((x131 + x92) - (x132 * x92));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x134 = (((x120 + x133) - (x122 * x133)) * Val(128));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x135 = get(ctx, arg0, 651, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x136 = (x135 * Val(2));
  // builtin Sub
  Val x137 = ((x135 + x98) - (x136 * x98));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x138 = (((x125 + x137) - (x127 * x137)) * Val(256));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x139 = get(ctx, arg0, 652, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x140 = (x139 * Val(2));
  // builtin Sub
  Val x141 = ((x139 + x93) - (x140 * x93));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x142 = (((x131 + x141) - (x132 * x141)) * Val(512));
  // builtin Add
  Val x143 = (((x130 + x134) + x138) + x142);
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x144 = get(ctx, arg0, 653, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x145 = (x144 * Val(2));
  // builtin Sub
  Val x146 = ((x144 + x99) - (x145 * x99));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x147 = (((x135 + x146) - (x136 * x146)) * Val(1024));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x148 = get(ctx, arg0, 654, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x149 = (x148 * Val(2));
  // builtin Sub
  Val x150 = ((x148 + x104) - (x149 * x104));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x151 = (((x139 + x150) - (x140 * x150)) * Val(2048));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x152 = get(ctx, arg0, 655, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x153 = (x152 * Val(2));
  // builtin Sub
  Val x154 = ((x152 + x109) - (x153 * x109));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x155 = (((x144 + x154) - (x145 * x154)) * Val(4096));
  // builtin Add
  Val x156 = (((x143 + x147) + x151) + x155);
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x157 = get(ctx, arg0, 624, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x158 = (x157 * Val(2));
  // builtin Sub
  Val x159 = ((x157 + x115) - (x158 * x115));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x160 = (((x148 + x159) - (x149 * x159)) * Val(8192));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x161 = get(ctx, arg0, 625, 0);
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeAE(zirgen/circuit/keccak/sha2.zir:93)
  // DoShaStep(zirgen/circuit/keccak/top.zir:139)
  Val x162 = (x161 * Val(2));
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x163 = ((x161 + x120) - (x162 * x120));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x164 = (((x152 + x163) - (x153 * x163)) * Val(16384));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x165 = get(ctx, arg0, 626, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x166 = (x165 * Val(2));
  // builtin Sub
  Val x167 = ((x165 + x125) - (x166 * x125));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x168 = (((x157 + x167) - (x158 * x167)) * Val(32768));
  // builtin Add
  Val x169 = (((x156 + x160) + x164) + x168);
  Val x170 = (get(ctx, arg0, 688, 2) + (get(ctx, arg0, 689, 2) * Val(2)));
  Val x171 = ((x170 + (get(ctx, arg0, 690, 2) * Val(4))) + (get(ctx, arg0, 691, 2) * Val(8)));
  Val x172 = ((x171 + (get(ctx, arg0, 692, 2) * Val(16))) + (get(ctx, arg0, 693, 2) * Val(32)));
  Val x173 = ((x172 + (get(ctx, arg0, 694, 2) * Val(64))) + (get(ctx, arg0, 695, 2) * Val(128)));
  Val x174 = ((x173 + (get(ctx, arg0, 696, 2) * Val(256))) + (get(ctx, arg0, 697, 2) * Val(512)));
  Val x175 = ((x174 + (get(ctx, arg0, 698, 2) * Val(1024))) + (get(ctx, arg0, 699, 2) * Val(2048)));
  Val x176 = ((x175 + (get(ctx, arg0, 700, 2) * Val(4096))) + (get(ctx, arg0, 701, 2) * Val(8192)));
  Val x177 =
      ((x176 + (get(ctx, arg0, 702, 2) * Val(16384))) + (get(ctx, arg0, 703, 2) * Val(32768)));
  Val x178 = (get(ctx, arg0, 720, 1) + (get(ctx, arg0, 721, 1) * Val(2)));
  Val x179 = ((x178 + (get(ctx, arg0, 722, 1) * Val(4))) + (get(ctx, arg0, 723, 1) * Val(8)));
  Val x180 = ((x179 + (get(ctx, arg0, 724, 1) * Val(16))) + (get(ctx, arg0, 725, 1) * Val(32)));
  Val x181 = ((x180 + (get(ctx, arg0, 726, 1) * Val(64))) + (get(ctx, arg0, 727, 1) * Val(128)));
  Val x182 = ((x181 + (get(ctx, arg0, 728, 1) * Val(256))) + (get(ctx, arg0, 729, 1) * Val(512)));
  Val x183 = ((x182 + (get(ctx, arg0, 730, 1) * Val(1024))) + (get(ctx, arg0, 731, 1) * Val(2048)));
  Val x184 = ((x183 + (get(ctx, arg0, 732, 1) * Val(4096))) + (get(ctx, arg0, 733, 1) * Val(8192)));
  Val x185 =
      ((x184 + (get(ctx, arg0, 734, 1) * Val(16384))) + (get(ctx, arg0, 735, 1) * Val(32768)));
  // Add2(zirgen/circuit/keccak/sha2.zir:30)
  Val x186 = (((x83 + x87) + x91) + (x169 + (x177 + x185)));
  // builtin Mul
  // Div(<preamble>:22)
  // CarryExtract(zirgen/circuit/keccak/sha2.zir:35)
  // CarryAndExpand(zirgen/circuit/keccak/sha2.zir:44)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  Val x187 = (bitAnd(x186, Val(983040)) * Val(2013235201));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 1029, bitAnd(x187, Val(1)));
  // CarryExtract(zirgen/circuit/keccak/sha2.zir:36)
  set(ctx, arg0, 1030, (bitAnd(x187, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak/sha2.zir:37)
  set(ctx, arg0, 1031, (bitAnd(x187, Val(4)) * Val(1509949441)));
  // builtin Add
  // CarryExtract(zirgen/circuit/keccak/sha2.zir:38)
  Val x188 = ((get(ctx, arg0, 1031, 0) * Val(4)) + (get(ctx, arg0, 1030, 0) * Val(2)));
  Val x189 = (x188 + get(ctx, arg0, 1029, 0));
  // builtin Sub
  // CarryExtract(zirgen/circuit/keccak/sha2.zir:39)
  Val x190 = (x186 - (x189 * Val(65536)));
  // builtin NondetReg
  // Reg(<preamble>:5)
  // TopState(zirgen/circuit/keccak/top.zir:36)
  Val x191 = get(ctx, arg0, 722, 2);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x192 = (x191 * Val(2));
  // builtin Sub
  Val x193 = ((x191 + x8) - (x192 * x8));
  // builtin Mul
  Val x194 = (x3 * Val(2));
  // builtin Sub
  Val x195 = ((x3 + x16) - (x194 * x16));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x196 = (((x43 + x195) - (x44 * x195)) * Val(2));
  // builtin Add
  Val x197 = (((x37 + x193) - (x38 * x193)) + x196);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x198 = (x9 * Val(2));
  // builtin Sub
  Val x199 = ((x9 + x23) - (x198 * x23));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x200 = (((x49 + x199) - (x50 * x199)) * Val(4));
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x201 = (x17 * Val(2));
  // builtin Sub
  Val x202 = ((x17 + x30) - (x201 * x30));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x203 = (((x56 + x202) - (x57 * x202)) * Val(8));
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x204 = (x24 * Val(2));
  // builtin Sub
  Val x205 = ((x24 + x37) - (x204 * x37));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x206 = (((x61 + x205) - (x62 * x205)) * Val(16));
  // builtin Add
  Val x207 = (((x197 + x200) + x203) + x206);
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x208 = ((x1 + x43) - (x6 * x43));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x209 = (((x66 + x208) - (x67 * x208)) * Val(32));
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x210 = ((x7 + x49) - (x12 * x49));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x211 = (((x71 + x210) - (x72 * x210)) * Val(64));
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x212 = ((x15 + x56) - (x20 * x56));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x213 = (((x75 + x212) - (x76 * x212)) * Val(128));
  // builtin Add
  Val x214 = (((x207 + x209) + x211) + x213);
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x215 = ((x22 + x61) - (x27 * x61));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x216 = (((x79 + x215) - (x80 * x215)) * Val(256));
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x217 = ((x29 + x66) - (x33 * x66));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x218 = (((x84 + x217) - (x85 * x217)) * Val(512));
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x219 = ((x36 + x71) - (x40 * x71));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x220 = (((x88 + x219) - (x89 * x219)) * Val(1024));
  // builtin Add
  Val x221 = (((x214 + x216) + x218) + x220);
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x222 = ((x42 + x75) - (x46 * x75));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x223 = (((x191 + x222) - (x192 * x222)) * Val(2048));
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:78)
  Val x224 = ((x48 + x79) - (x52 * x79));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x225 = (((x3 + x224) - (x194 * x224)) * Val(4096));
  Val x226 = (((x9 + x55) - (x198 * x55)) * Val(8192));
  // builtin Add
  Val x227 = (((x221 + x223) + x225) + x226);
  // builtin Mul
  Val x228 = (((x17 + x60) - (x201 * x60)) * Val(16384));
  Val x229 = (((x24 + x65) - (x204 * x65)) * Val(32768));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x230 = get(ctx, arg0, 627, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x231 = (x230 * Val(2));
  // builtin Sub
  Val x232 = ((x230 + x131) - (x231 * x131));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x233 = get(ctx, arg0, 628, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x234 = (x233 * Val(2));
  // builtin Sub
  Val x235 = ((x233 + x135) - (x234 * x135));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x236 = (((x165 + x235) - (x166 * x235)) * Val(2));
  // builtin Add
  Val x237 = (((x161 + x232) - (x162 * x232)) + x236);
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x238 = get(ctx, arg0, 629, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x239 = (x238 * Val(2));
  // builtin Sub
  Val x240 = ((x238 + x139) - (x239 * x139));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x241 = (((x230 + x240) - (x231 * x240)) * Val(4));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x242 = get(ctx, arg0, 630, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x243 = (x242 * Val(2));
  // builtin Sub
  Val x244 = ((x242 + x144) - (x243 * x144));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x245 = (((x233 + x244) - (x234 * x244)) * Val(8));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x246 = get(ctx, arg0, 631, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x247 = (x246 * Val(2));
  // builtin Sub
  Val x248 = ((x246 + x148) - (x247 * x148));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x249 = (((x238 + x248) - (x239 * x248)) * Val(16));
  // builtin Add
  Val x250 = (((x237 + x241) + x245) + x249);
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x251 = get(ctx, arg0, 632, 0);
  // builtin Mul
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x252 = (x251 * Val(2));
  // builtin Sub
  Val x253 = ((x251 + x152) - (x252 * x152));
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x254 = (((x242 + x253) - (x243 * x253)) * Val(32));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  // ShaCycle(zirgen/circuit/keccak/top.zir:196)
  Val x255 = get(ctx, arg0, 633, 0);
  // builtin Mul
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  // DoShaStep(zirgen/circuit/keccak/top.zir:137)
  // ShaCycle(zirgen/circuit/keccak/top.zir:198)
  Val x256 = (((x246 + x255) - (x247 * x255)) * Val(64));
  Val x257 = (((x251 + x94) - (x252 * x94)) * Val(128));
  // builtin Add
  Val x258 = (((x250 + x254) + x256) + x257);
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  Val x259 = ((x255 + x100) - ((x255 * Val(2)) * x100));
  Val x260 = ((x94 + x105) - ((x94 * Val(2)) * x105));
  Val x261 = ((x100 + x110) - ((x100 * Val(2)) * x110));
  // builtin Add
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x262 = (((x258 + (x259 * Val(256))) + (x260 * Val(512))) + (x261 * Val(1024)));
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  Val x263 = ((x105 + x116) - ((x105 * Val(2)) * x116));
  Val x264 = ((x110 + x121) - ((x110 * Val(2)) * x121));
  Val x265 = ((x116 + x126) - ((x116 * Val(2)) * x126));
  // builtin Add
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x266 = (((x262 + (x263 * Val(2048))) + (x264 * Val(4096))) + (x265 * Val(8192)));
  // builtin Sub
  // BitXor(zirgen/circuit/keccak/bits.zir:34)
  // XorU32(zirgen/circuit/keccak/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:79)
  Val x267 = ((x121 + x92) - ((x121 * Val(2)) * x92));
  Val x268 = ((x126 + x98) - ((x126 * Val(2)) * x98));
  // builtin Add
  // Pack(zirgen/circuit/keccak/pack.zir:35)
  // Pack32(zirgen/circuit/keccak/sha2.zir:74)
  // ComputeW(zirgen/circuit/keccak/sha2.zir:80)
  Val x269 = (get(ctx, arg0, 704, 2) + (get(ctx, arg0, 705, 2) * Val(2)));
  Val x270 = ((x269 + (get(ctx, arg0, 706, 2) * Val(4))) + (get(ctx, arg0, 707, 2) * Val(8)));
  Val x271 = ((x270 + (get(ctx, arg0, 708, 2) * Val(16))) + (get(ctx, arg0, 709, 2) * Val(32)));
  Val x272 = ((x271 + (get(ctx, arg0, 710, 2) * Val(64))) + (get(ctx, arg0, 711, 2) * Val(128)));
  Val x273 = ((x272 + (get(ctx, arg0, 712, 2) * Val(256))) + (get(ctx, arg0, 713, 2) * Val(512)));
  Val x274 = ((x273 + (get(ctx, arg0, 714, 2) * Val(1024))) + (get(ctx, arg0, 715, 2) * Val(2048)));
  Val x275 = ((x274 + (get(ctx, arg0, 716, 2) * Val(4096))) + (get(ctx, arg0, 717, 2) * Val(8192)));
  Val x276 =
      ((x275 + (get(ctx, arg0, 718, 2) * Val(16384))) + (get(ctx, arg0, 719, 2) * Val(32768)));
  Val x277 = (get(ctx, arg0, 736, 1) + (get(ctx, arg0, 737, 1) * Val(2)));
  Val x278 = ((x277 + (get(ctx, arg0, 738, 1) * Val(4))) + (get(ctx, arg0, 739, 1) * Val(8)));
  Val x279 = ((x278 + (get(ctx, arg0, 740, 1) * Val(16))) + (get(ctx, arg0, 741, 1) * Val(32)));
  Val x280 = ((x279 + (get(ctx, arg0, 742, 1) * Val(64))) + (get(ctx, arg0, 743, 1) * Val(128)));
  Val x281 = ((x280 + (get(ctx, arg0, 744, 1) * Val(256))) + (get(ctx, arg0, 745, 1) * Val(512)));
  Val x282 = ((x281 + (get(ctx, arg0, 746, 1) * Val(1024))) + (get(ctx, arg0, 747, 1) * Val(2048)));
  Val x283 = ((x282 + (get(ctx, arg0, 748, 1) * Val(4096))) + (get(ctx, arg0, 749, 1) * Val(8192)));
  Val x284 =
      ((x283 + (get(ctx, arg0, 750, 1) * Val(16384))) + (get(ctx, arg0, 751, 1) * Val(32768)));
  // Add2(zirgen/circuit/keccak/sha2.zir:30)
  Val x285 = (((x266 + (x267 * Val(16384))) + (x268 * Val(32768))) + (x276 + x284));
  Val x286 = (((x227 + x228) + x229) + x285);
  // CarryAndExpand(zirgen/circuit/keccak/sha2.zir:45)
  // DoShaStep(zirgen/circuit/keccak/top.zir:138)
  Val x287 = (x286 + x189);
  // builtin Mul
  // Div(<preamble>:22)
  // CarryExtract(zirgen/circuit/keccak/sha2.zir:35)
  Val x288 = (bitAnd(x287, Val(983040)) * Val(2013235201));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  set(ctx, arg0, 1032, bitAnd(x288, Val(1)));
  // CarryExtract(zirgen/circuit/keccak/sha2.zir:36)
  set(ctx, arg0, 1033, (bitAnd(x288, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak/sha2.zir:37)
  set(ctx, arg0, 1034, (bitAnd(x288, Val(4)) * Val(1509949441)));
  // builtin Add
  // CarryExtract(zirgen/circuit/keccak/sha2.zir:38)
  Val x289 = ((get(ctx, arg0, 1034, 0) * Val(4)) + (get(ctx, arg0, 1033, 0) * Val(2)));
  // builtin Sub
  // CarryExtract(zirgen/circuit/keccak/sha2.zir:39)
  Val x290 = (x287 - ((x289 + get(ctx, arg0, 1032, 0)) * Val(65536)));
  // builtin NondetReg
  // NondetBitReg(zirgen/circuit/keccak/bits.zir:13)
  // UnpackReg(zirgen/circuit/keccak/pack.zir:52)
  // CarryAndExpand(zirgen/circuit/keccak/sha2.zir:46)
  set(ctx, arg0, 688, bitAnd(x190, Val(1)));
  set(ctx, arg0, 689, (bitAnd(x190, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 690, (bitAnd(x190, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 691, (bitAnd(x190, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 692, (bitAnd(x190, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 693, (bitAnd(x190, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 694, (bitAnd(x190, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 695, (bitAnd(x190, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 696, (bitAnd(x190, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 697, (bitAnd(x190, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 698, (bitAnd(x190, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 699, (bitAnd(x190, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 700, (bitAnd(x190, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 701, (bitAnd(x190, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 702, (bitAnd(x190, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 703, (bitAnd(x190, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 704, bitAnd(x290, Val(1)));
  set(ctx, arg0, 705, (bitAnd(x290, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 706, (bitAnd(x290, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 707, (bitAnd(x290, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 708, (bitAnd(x290, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 709, (bitAnd(x290, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 710, (bitAnd(x290, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 711, (bitAnd(x290, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 712, (bitAnd(x290, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 713, (bitAnd(x290, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 714, (bitAnd(x290, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 715, (bitAnd(x290, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 716, (bitAnd(x290, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 717, (bitAnd(x290, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 718, (bitAnd(x290, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 719, (bitAnd(x290, Val(32768)) * Val(2013204481)));
  return;
}

} // namespace risc0::circuit::keccak::cuda
