#include "hip/hip_runtime.h"
// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0, <LICENSE-APACHE or
// http://apache.org/licenses/LICENSE-2.0> or the MIT license <LICENSE-MIT or
// http://opensource.org/licenses/MIT>, at your option. This file may not be
// copied, modified, or distributed except according to those terms.
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
// SPDX-License-Identifier: Apache-2.0 OR MIT

#include "buffers.h"
#include "hip/hip_runtime.h"
#include "fp.h"
#include "fpext.h"
#include "steps.cuh"
#include "witgen.h"

#include <cstdint>
#include <cstdio>
#include <cuda/std/array>
#include <string.h>

using namespace risc0;

namespace risc0::circuit::keccak::cuda {

struct ExecBuffers {
  Buffer global;
  Buffer data;
};

struct ScatterInfo {
  uint32_t offset;
  uint32_t row;
  uint16_t col;
  uint16_t count;
  uint32_t bits;
};

struct ScatterContext {
  ScatterInfo* d_infos;

  ScatterContext(const ScatterInfo* infos, size_t count) {
    CUDA_OK(hipMalloc(&d_infos, count * sizeof(ScatterInfo)));
    CUDA_OK(hipMemcpy(d_infos, infos, count * sizeof(ScatterInfo), hipMemcpyHostToDevice));
  }

  ~ScatterContext() { hipFree(d_infos); }
};

struct DeviceContext {
  Buffer* data;
  Buffer* global;
  PreflightTrace* preflight;
};

struct HostContext {
  DeviceContext* ctx;

  HostContext(ExecBuffers* buffers, PreflightTrace* preflight, size_t cycles) {
    CUDA_OK(hipMallocManaged(&ctx, sizeof(DeviceContext)));

    CUDA_OK(hipMallocManaged(&ctx->data, sizeof(Buffer)));
    ctx->data->buf = buffers->data.buf;
    ctx->data->rows = buffers->data.rows;
    ctx->data->cols = buffers->data.cols;
    ctx->data->checkedReads = buffers->data.checkedReads;

    CUDA_OK(hipMallocManaged(&ctx->global, sizeof(Buffer)));
    ctx->global->buf = buffers->global.buf;
    ctx->global->rows = buffers->global.rows;
    ctx->global->cols = buffers->global.cols;
    ctx->global->checkedReads = buffers->global.checkedReads;

    CUDA_OK(hipMallocManaged(&ctx->preflight, sizeof(PreflightTrace)));

    CUDA_OK(hipMalloc(&ctx->preflight->preimages, preflight->preimagesSize * sizeof(KeccakState)));
    CUDA_OK(hipMemcpy(ctx->preflight->preimages,
                       preflight->preimages,
                       preflight->preimagesSize * sizeof(KeccakState),
                       hipMemcpyHostToDevice));

    ctx->preflight->preimagesSize = preflight->preimagesSize;

    CUDA_OK(hipMalloc(&ctx->preflight->curPreimage, cycles * sizeof(uint32_t)));
    CUDA_OK(hipMemcpy(ctx->preflight->curPreimage,
                       preflight->curPreimage,
                       cycles * sizeof(uint32_t),
                       hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&ctx->preflight->runOrder, cycles * sizeof(uint32_t)));
    CUDA_OK(hipMemcpy(ctx->preflight->runOrder,
                       preflight->runOrder,
                       cycles * sizeof(uint32_t),
                       hipMemcpyHostToDevice));
  }

  ~HostContext() {
    hipFree(ctx->preflight->curPreimage);
    hipFree(ctx->preflight->preimages);
    hipFree(ctx->preflight->runOrder);
    hipFree(ctx->preflight);
    hipFree(ctx->global);
    hipFree(ctx->data);
    hipFree(ctx);
  }
};

__device__ void nextStep(DeviceContext* ctx, uint32_t cycle) {
  // printf("nextStep: %u\n", cycle);
  ExecContext execCtx(
      *ctx->preflight, ctx->preflight->runOrder[cycle], ctx->preflight->curPreimage[cycle]);
  MutableBufObj data(*ctx->data);
  GlobalBufObj global(*ctx->global);
  step_Top(execCtx, &data, &global);
}

__global__ void par_stepExec(DeviceContext* ctx, uint32_t count) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }
  nextStep(ctx, cycle);
}

__global__ void rev_stepExec(DeviceContext* ctx, uint32_t count) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle == count - 1) {
    for (uint32_t i = 0; i < count; i++) {
      nextStep(ctx, count - i - 1);
    }
  }
}

__global__ void fwd_stepExec(DeviceContext* ctx, uint32_t count) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle == 0) {
    for (uint32_t i = 0; i < count; i++) {
      nextStep(ctx, i);
    }
  }
}

__global__ void scatter_preflight(Fp* into,
                                  const ScatterInfo* infos,
                                  const uint32_t* from,
                                  const uint32_t rows,
                                  const uint32_t count) {
  uint gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count) {
    return;
  }

  const ScatterInfo& info = infos[gid];
  uint32_t innerCount = 32 / info.bits;
  uint32_t mask = (1 << (info.bits)) - 1;
  for (size_t i = 0; i < info.count; i++) {
    uint32_t word = from[info.offset + (i / innerCount)];
    size_t j = i % innerCount;
    uint32_t val = (word >> (j * info.bits)) & mask;
    size_t col = info.col + i;
    into[col * rows + info.row] = val;
  }
}

} // namespace risc0::circuit::keccak::cuda

constexpr size_t kStepModeSeqParallel = 0;
constexpr size_t kStepModeSeqForward = 1;
constexpr size_t kStepModeSeqReverse = 2;

extern "C" {

using namespace risc0::circuit::keccak::cuda;

const char* risc0_circuit_keccak_cuda_witgen(uint32_t mode,
                                             ExecBuffers* buffers,
                                             PreflightTrace* preflight,
                                             uint32_t lastCycle) {
  try {
    HostContext ctx(buffers, preflight, lastCycle);
    CudaStream stream;

    auto cfg = getSimpleConfig(lastCycle);
    switch (mode) {
    case kStepModeSeqParallel:
      par_stepExec<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, lastCycle);
      break;
    case kStepModeSeqForward:
      fwd_stepExec<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, lastCycle);
      break;
    case kStepModeSeqReverse:
      rev_stepExec<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, lastCycle);
      break;
    }
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

const char* risc0_circuit_keccak_cuda_scatter(Fp* into,
                                              const ScatterInfo* infos,
                                              const uint32_t* from,
                                              const uint32_t rows,
                                              const uint32_t count) {
  try {
    ScatterContext ctx(infos, count);
    CudaStream stream;
    auto cfg = getSimpleConfig(count);
    scatter_preflight<<<cfg.grid, cfg.block, 0, stream>>>(into, ctx.d_infos, from, rows, count);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

const char* risc0_circuit_keccak_cuda_reset() {
  try {
    CUDA_OK(hipDeviceSetLimit(hipLimit_t::hipLimitStackSize, 0));
  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

} // extern "C"
