#include "hip/hip_runtime.h"
// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "context.h"
#include "hip/hip_runtime.h"
#include "fp.h"
#include "fpext.h"
#include "kernels.h"

#include "vendor/nvtx3/nvtx3.hpp"

#include <cstring>
#include <cuda/std/array>
#include <hip/hip_runtime.h>
#include <exception>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <vector>

constexpr size_t kStepModeParallel = 0;
constexpr size_t kStepModeSeqForward = 1;
constexpr size_t kStepModeSeqReverse = 2;

namespace sppark {
void calcPrefixProducts(void* d_inout, uint32_t count);
} // namespace sppark

__device__ void nextStepExec(ExecContext* ctx, uint32_t cycle, uint32_t count) {
  if (cycle == 0 || ctx->isParSafeExec(cycle)) {
    Fp* args[3]{ctx->buffers.ctrl, ctx->buffers.global, ctx->buffers.data};
    // printf("step_exec(%u)\n", cycle);
    step_exec(ctx, ctx->totalCycles, cycle++, args[0], args[1], args[2], nullptr, nullptr);
    while (cycle < count && !ctx->isParSafeExec(cycle)) {
      // printf("  step_exec(%u)\n", cycle);
      step_exec(ctx, ctx->totalCycles, cycle++, args[0], args[1], args[2], nullptr, nullptr);
    }
  }
}

__global__ void parStepExec(ExecContext* ctx) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t count = ctx->trace->numCycles;
  if (cycle < count) {
    nextStepExec(ctx, cycle, count);
  }
}

__global__ void fwdStepExec(ExecContext* ctx) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t count = ctx->trace->numCycles;
  if (cycle == 0) {
    Fp* args[3]{ctx->buffers.ctrl, ctx->buffers.global, ctx->buffers.data};
    for (uint32_t cycle = 0; cycle < count; cycle++) {
      // printf("step_exec(%u)\n", cycle);
      step_exec(ctx, ctx->totalCycles, cycle, args[0], args[1], args[2], nullptr, nullptr);
    }
  }
}

__global__ void revStepExec(ExecContext* ctx) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t count = ctx->trace->numCycles;
  if (cycle == count - 1) {
    Fp* args[3]{ctx->buffers.ctrl, ctx->buffers.global, ctx->buffers.data};
    for (uint32_t i = 0; i < count; i++) {
      uint32_t cycle = count - i - 1;
      nextStepExec(ctx, cycle, count);
    }
  }
}

__global__ void injectWomBacks(ExecContext* ctx) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t count = ctx->trace->numCycles;
  if (cycle < 1 || cycle >= count) {
    return;
  }

  Fp* data = ctx->buffers.data;
  uint32_t totalCycles = ctx->totalCycles;
  uint32_t idx = ctx->womIndex[cycle];
  if (idx) {
    const WomArgumentRow& prev = ctx->womRows[idx - 1];
    data[0 * totalCycles + cycle - 1] = prev.addr;
    data[1 * totalCycles + cycle - 1] = prev.value.elems[0];
    data[2 * totalCycles + cycle - 1] = prev.value.elems[1];
    data[3 * totalCycles + cycle - 1] = prev.value.elems[2];
    data[4 * totalCycles + cycle - 1] = prev.value.elems[3];
  } else {
    data[0 * totalCycles + cycle - 1] = 0;
    data[1 * totalCycles + cycle - 1] = 0;
    data[2 * totalCycles + cycle - 1] = 0;
    data[3 * totalCycles + cycle - 1] = 0;
    data[4 * totalCycles + cycle - 1] = 0;
  }
}

__global__ void parStepVerifyWom(ExecContext* ctx) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t count = ctx->trace->numCycles;
  if (cycle < count) {
    Fp* args[3]{ctx->buffers.ctrl, ctx->buffers.global, ctx->buffers.data};
    step_verify_mem(ctx, ctx->totalCycles, cycle, args[0], args[1], args[2], nullptr, nullptr);
  }
}

__global__ void fwdStepVerifyWom(ExecContext* ctx) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t count = ctx->trace->numCycles;
  if (cycle == 0) {
    Fp* args[3]{ctx->buffers.ctrl, ctx->buffers.global, ctx->buffers.data};
    for (uint32_t cycle = 0; cycle < count; cycle++) {
      // printf("step_verify_mem: %u\n", cycle);
      step_verify_mem(ctx, ctx->totalCycles, cycle, args[0], args[1], args[2], nullptr, nullptr);
    }
  }
}

__global__ void parStepComputeAccum(AccumContext* ctx) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle < ctx->workCycles) {
    step_compute_accum(ctx,
                       ctx->totalCycles,
                       cycle,
                       ctx->buffers.ctrl,
                       ctx->buffers.global,
                       ctx->buffers.data,
                       ctx->buffers.mix,
                       ctx->buffers.accum);
  }
}

__global__ void parStepVerifyAccum(AccumContext* ctx) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle < ctx->workCycles) {
    step_verify_accum(ctx,
                      ctx->totalCycles,
                      cycle,
                      ctx->buffers.ctrl,
                      ctx->buffers.global,
                      ctx->buffers.data,
                      ctx->buffers.mix,
                      ctx->buffers.accum);
  }
}

struct HostExecContext {
  ExecContext* ctx;
  CudaStream stream;
  LaunchConfig cfg;

  HostExecContext(ExecBuffers* buffers, PreflightTrace* trace, size_t totalCycles)
      : cfg(getSimpleConfig(trace->numCycles)) {
    CUDA_OK(hipMallocManaged(&ctx, sizeof(ExecContext)));
    ctx->buffers.ctrl = buffers->ctrl;
    ctx->buffers.data = buffers->data;
    ctx->buffers.global = buffers->global;
    ctx->totalCycles = totalCycles;

    CUDA_OK(hipMallocManaged(&ctx->trace, sizeof(PreflightTrace)));
    ctx->trace->numWoms = trace->numWoms;
    ctx->trace->numCycles = trace->numCycles;
    ctx->trace->numIops = trace->numIops;

    CUDA_OK(hipMalloc(&ctx->trace->wom, trace->numWoms * sizeof(FpExt)));
    CUDA_OK(hipMemcpy(
        ctx->trace->wom, trace->wom, trace->numWoms * sizeof(FpExt), hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&ctx->trace->cycles, trace->numCycles * sizeof(PreflightCycle)));
    CUDA_OK(hipMemcpy(ctx->trace->cycles,
                       trace->cycles,
                       trace->numCycles * sizeof(PreflightCycle),
                       hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&ctx->trace->iops, trace->numIops * sizeof(FpExt)));
    CUDA_OK(hipMemcpy(
        ctx->trace->iops, trace->iops, trace->numIops * sizeof(FpExt), hipMemcpyHostToDevice));

    CUDA_OK(
        hipMalloc(&ctx->womRows, trace->numCycles * kMaxWomRowsPerCycle * sizeof(WomArgumentRow)));
    CUDA_OK(hipMemset(ctx->womRows,
                       kInvalidPattern,
                       trace->numCycles * kMaxWomRowsPerCycle * sizeof(WomArgumentRow)));

    CUDA_OK(hipMalloc(&ctx->womIndex, trace->numCycles * sizeof(uint32_t)));
    CUDA_OK(hipMemset(ctx->womIndex, 0, trace->numCycles * sizeof(uint32_t)));
  }

  ~HostExecContext() {
    hipFree(ctx->womIndex);
    hipFree(ctx->womRows);
    hipFree(ctx->trace->iops);
    hipFree(ctx->trace->cycles);
    hipFree(ctx->trace->wom);
    hipFree(ctx->trace);
    hipFree(ctx);
  }

  void doStepExec(uint32_t mode) {
    nvtx3::scoped_range range("stepExec");
    switch (mode) {
    case kStepModeParallel: {
      parStepExec<<<cfg.grid, cfg.block, 0, stream>>>(ctx);
    } break;
    case kStepModeSeqForward: {
      fwdStepExec<<<cfg.grid, cfg.block, 0, stream>>>(ctx);
    } break;
    case kStepModeSeqReverse: {
      revStepExec<<<cfg.grid, cfg.block, 0, stream>>>(ctx);
    } break;
    }
    CUDA_OK(hipStreamSynchronize(stream));
  }

  void verifyWom(uint32_t mode) {
    nvtx3::scoped_range range("verifyWom");
    uint32_t numCycles = ctx->trace->numCycles;

    {
      nvtx3::scoped_range range("sortWom");
      thrust::sort(thrust::device, ctx->womRows, ctx->womRows + numCycles * kMaxWomRowsPerCycle);
    }

    {
      nvtx3::scoped_range range("scan");
      thrust::exclusive_scan(
          thrust::device, ctx->womIndex, ctx->womIndex + numCycles, ctx->womIndex);
    }

    {
      nvtx3::scoped_range range("injectWomBacks");
      injectWomBacks<<<cfg.grid, cfg.block, 0, stream>>>(ctx);
      CUDA_OK(hipStreamSynchronize(stream));
    }

    {
      nvtx3::scoped_range range("stepVerifyWom");
      parStepVerifyWom<<<cfg.grid, cfg.block, 0, stream>>>(ctx);
      CUDA_OK(hipStreamSynchronize(stream));
    }
  }
};

struct HostAccumContext {
  AccumContext* ctx;
  CudaStream stream;
  LaunchConfig cfg;

  HostAccumContext(AccumBuffers* buffers, size_t workCycles, size_t totalCycles)
      : cfg(getSimpleConfig(workCycles)) {
    CUDA_OK(hipMallocManaged(&ctx, sizeof(AccumContext)));
    ctx->buffers.ctrl = buffers->ctrl;
    ctx->buffers.global = buffers->global;
    ctx->buffers.data = buffers->data;
    ctx->buffers.mix = buffers->mix;
    ctx->buffers.accum = buffers->accum;
    ctx->totalCycles = totalCycles;
    ctx->workCycles = workCycles;

    std::vector<FpExt> accumInit(workCycles, FpExt(1));
    CUDA_OK(hipMalloc(&ctx->accum, workCycles * sizeof(FpExt)));
    CUDA_OK(hipMemcpy(
        ctx->accum, accumInit.data(), workCycles * sizeof(FpExt), hipMemcpyHostToDevice));
  }

  ~HostAccumContext() {
    hipFree(ctx->accum);
    hipFree(ctx);
  }

  void computeAccum() {
    nvtx3::scoped_range range("computeAccum");
    parStepComputeAccum<<<cfg.grid, cfg.block, 0, stream>>>(ctx);
    CUDA_OK(hipStreamSynchronize(stream));
  }

  void calcPrefixProducts() {
    nvtx3::scoped_range range("calcPrefixProducts");
    sppark::calcPrefixProducts(ctx->accum, ctx->workCycles);
    CUDA_OK(hipStreamSynchronize(stream));
  }

  void verifyAccum() {
    nvtx3::scoped_range range("verifyAccum");
    CUDA_OK(hipDeviceSynchronize());
    parStepVerifyAccum<<<cfg.grid, cfg.block, 0, stream>>>(ctx);
    CUDA_OK(hipStreamSynchronize(stream));
  }
};

extern "C" {

const char* risc0_circuit_recursion_cuda_witgen(uint32_t mode,
                                                ExecBuffers* buffers,
                                                PreflightTrace* trace,
                                                uint32_t totalCycles) {
  try {
    CUDA_OK(hipDeviceSynchronize());
    HostExecContext ctx(buffers, trace, totalCycles);
    ctx.doStepExec(mode);
    ctx.verifyWom(mode);
  } catch (const std::exception& err) {
    return strdup(err.what());
  }
  return nullptr;
}

const char* risc0_circuit_recursion_cuda_accum(AccumBuffers* buffers,
                                               uint32_t workCycles,
                                               uint32_t totalCycles) {
  try {
    CUDA_OK(hipDeviceSynchronize());
    HostAccumContext ctx(buffers, workCycles, totalCycles);
    ctx.computeAccum();
    ctx.calcPrefixProducts();
    ctx.verifyAccum();
  } catch (const std::exception& err) {
    return strdup(err.what());
  }
  return nullptr;
}

} // extern "C"
