#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hip/hip_runtime.h"
#include "supra/fp.h"

extern __constant__ FpExt poly_mix[158];

namespace risc0::circuit::recursion::cuda {

__device__ FpExt poly_fp(uint32_t idx,
                         uint32_t size,
                         const Fp* ctrl,
                         const Fp* out,
                         const Fp* data,
                         const Fp* mix,
                         const Fp* accum);

__global__ void eval_check(Fp* check,
                           const Fp* ctrl,
                           const Fp* data,
                           const Fp* accum,
                           const Fp* mix,
                           const Fp* out,
                           const Fp rou,
                           uint32_t po2,
                           uint32_t domain) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle < domain) {
    FpExt tot = poly_fp(cycle, domain, ctrl, out, data, mix, accum);
    Fp x = pow(rou, cycle);
    Fp y = pow(Fp(3) * x, 1 << po2);
    FpExt ret = tot * inv(y - Fp(1));
    check[domain * 0 + cycle] = ret[0];
    check[domain * 1 + cycle] = ret[1];
    check[domain * 2 + cycle] = ret[2];
    check[domain * 3 + cycle] = ret[3];
  }
}

} // namespace risc0::circuit::recursion::cuda

extern "C" const char* risc0_circuit_recursion_cuda_eval_check(Fp* check,
                                                               const Fp* ctrl,
                                                               const Fp* data,
                                                               const Fp* accum,
                                                               const Fp* mix,
                                                               const Fp* out,
                                                               const Fp& rou,
                                                               uint32_t po2,
                                                               uint32_t domain,
                                                               const FpExt* poly_mix_pows) {

  try {
    CUDA_OK(hipDeviceSynchronize());
    CudaStream stream;
    LaunchConfig cfg = getSimpleConfig(domain);
    hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
    risc0::circuit::recursion::cuda::eval_check<<<cfg.grid, cfg.block, 0, stream>>>(
        check, ctrl, data, accum, mix, out, rou, po2, domain);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::exception& err) {
    return strdup(err.what());
  }
  return nullptr;
}
