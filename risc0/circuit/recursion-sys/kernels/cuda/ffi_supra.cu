#include "hip/hip_runtime.h"
// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0, <LICENSE-APACHE or
// http://apache.org/licenses/LICENSE-2.0> or the MIT license <LICENSE-MIT or
// http://opensource.org/licenses/MIT>, at your option. This file may not be
// copied, modified, or distributed except according to those terms.
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
// SPDX-License-Identifier: Apache-2.0 OR MIT

#include "hip/hip_runtime.h"
#include "supra/fp.h"

extern __constant__ FpExt poly_mix[158];

namespace risc0::circuit::recursion::cuda {

__device__ FpExt poly_fp(uint32_t idx,
                         uint32_t size,
                         const Fp* ctrl,
                         const Fp* out,
                         const Fp* data,
                         const Fp* mix,
                         const Fp* accum);

__global__ void eval_check(Fp* check,
                           const Fp* ctrl,
                           const Fp* data,
                           const Fp* accum,
                           const Fp* mix,
                           const Fp* out,
                           const Fp rou,
                           uint32_t po2,
                           uint32_t domain) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle < domain) {
    FpExt tot = poly_fp(cycle, domain, ctrl, out, data, mix, accum);
    Fp x = pow(rou, cycle);
    Fp y = pow(Fp(3) * x, 1 << po2);
    FpExt ret = tot * inv(y - Fp(1));
    check[domain * 0 + cycle] = ret[0];
    check[domain * 1 + cycle] = ret[1];
    check[domain * 2 + cycle] = ret[2];
    check[domain * 3 + cycle] = ret[3];
  }
}

} // namespace risc0::circuit::recursion::cuda

extern "C" const char* risc0_circuit_recursion_cuda_eval_check(Fp* check,
                                                               const Fp* ctrl,
                                                               const Fp* data,
                                                               const Fp* accum,
                                                               const Fp* mix,
                                                               const Fp* out,
                                                               const Fp& rou,
                                                               uint32_t po2,
                                                               uint32_t domain,
                                                               const FpExt* poly_mix_pows) {

  try {
    CUDA_OK(hipDeviceSynchronize());
    CudaStream stream;
    LaunchConfig cfg = getSimpleConfig(domain);
    hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
    risc0::circuit::recursion::cuda::eval_check<<<cfg.grid, cfg.block, 0, stream>>>(
        check, ctrl, data, accum, mix, out, rou, po2, domain);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::exception& err) {
    return strdup(err.what());
  }
  return nullptr;
}
