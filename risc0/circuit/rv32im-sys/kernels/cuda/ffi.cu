#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hip/hip_runtime.h"
#include "steps.cuh"
#include "witgen.h"

#if defined(__clang__)
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wmissing-braces"
#elif defined(__GNUC__)
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wmissing-braces"
#endif

#include "vendor/nvtx3/nvtx3.hpp"

#if defined(__clang__)
#pragma clang diagnostic pop
#elif defined(__GNUC__)
#pragma GCC diagnostic pop
#endif

#include <cstdint>
#include <cstdio>
#include <cuda/std/array>
#include <string.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

namespace risc0::circuit::rv32im_v2::cuda {

constexpr size_t kUserAccumSplit = kLayout_TopAccum.columns[0].col;

struct ExecBuffers {
  Buffer global;
  Buffer data;
};

struct DeviceExecContext {
  Buffer* data;
  Buffer* global;
  PreflightTrace* preflight;
  LookupTables* tables;
};

struct HostExecContext {
  DeviceExecContext* ctx;
  PreflightTrace d_preflight;
  LookupTables d_tables;

  HostExecContext(ExecBuffers* buffers, PreflightTrace* preflight, size_t cycles) {
    CUDA_OK(hipMallocManaged(&ctx, sizeof(DeviceExecContext)));

    CUDA_OK(hipMalloc(&ctx->data, sizeof(Buffer)));
    CUDA_OK(hipMemcpy(ctx->data, &buffers->data, sizeof(Buffer), hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&ctx->global, sizeof(Buffer)));
    CUDA_OK(hipMemcpy(ctx->global, &buffers->global, sizeof(Buffer), hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&d_preflight.cycles, cycles * sizeof(PreflightCycle)));
    CUDA_OK(hipMemcpy(d_preflight.cycles,
                       preflight->cycles,
                       cycles * sizeof(PreflightCycle),
                       hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&d_preflight.txns, preflight->txnsLen * sizeof(MemoryTransaction)));
    CUDA_OK(hipMemcpy(d_preflight.txns,
                       preflight->txns,
                       preflight->txnsLen * sizeof(MemoryTransaction),
                       hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&d_preflight.bigintBytes, preflight->bigintBytesLen));
    CUDA_OK(hipMemcpy(d_preflight.bigintBytes,
                       preflight->bigintBytes,
                       preflight->bigintBytesLen,
                       hipMemcpyHostToDevice));

    d_preflight.txnsLen = preflight->txnsLen;
    d_preflight.bigintBytesLen = preflight->bigintBytesLen;
    d_preflight.tableSplitCycle = preflight->tableSplitCycle;

    CUDA_OK(hipMalloc(&ctx->preflight, sizeof(PreflightTrace)));
    CUDA_OK(
        hipMemcpy(ctx->preflight, &d_preflight, sizeof(PreflightTrace), hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&d_tables.tableU8, (1 << 8) * sizeof(uint32_t)));
    CUDA_OK(hipMemset(d_tables.tableU8, 0, (1 << 8) * sizeof(uint32_t)));

    CUDA_OK(hipMalloc(&d_tables.tableU16, (1 << 16) * sizeof(uint32_t)));
    CUDA_OK(hipMemset(d_tables.tableU16, 0, (1 << 16) * sizeof(uint32_t)));

    CUDA_OK(hipMalloc(&ctx->tables, sizeof(LookupTables)));
    CUDA_OK(hipMemcpy(ctx->tables, &d_tables, sizeof(LookupTables), hipMemcpyHostToDevice));
  }

  ~HostExecContext() {
    hipFree(d_tables.tableU16);
    hipFree(d_tables.tableU8);
    hipFree(ctx->tables);
    hipFree(d_preflight.bigintBytes);
    hipFree(d_preflight.txns);
    hipFree(d_preflight.cycles);
    hipFree(ctx->preflight);
    hipFree(ctx->global);
    hipFree(ctx->data);
    hipFree(ctx);
  }
};

struct AccumBuffers {
  Buffer data;
  Buffer accum;
  Buffer global;
  Buffer mix;
};

struct DeviceAccumContext {
  Buffer* data;
  Buffer* accum;
  Buffer* global;
  Buffer* mix;
  PreflightTrace* preflight;
  LookupTables* tables;
};

struct HostAccumContext {
  DeviceAccumContext* ctx;
  PreflightTrace d_preflight;
  LookupTables d_tables;

  HostAccumContext(AccumBuffers* buffers, PreflightTrace* preflight, size_t cycles) {
    CUDA_OK(hipMallocManaged(&ctx, sizeof(DeviceAccumContext)));

    CUDA_OK(hipMalloc(&ctx->data, sizeof(Buffer)));
    CUDA_OK(hipMemcpy(ctx->data, &buffers->data, sizeof(Buffer), hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&ctx->accum, sizeof(Buffer)));
    CUDA_OK(hipMemcpy(ctx->accum, &buffers->accum, sizeof(Buffer), hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&ctx->global, sizeof(Buffer)));
    CUDA_OK(hipMemcpy(ctx->global, &buffers->global, sizeof(Buffer), hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&ctx->mix, sizeof(Buffer)));
    CUDA_OK(hipMemcpy(ctx->mix, &buffers->mix, sizeof(Buffer), hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&d_preflight.cycles, cycles * sizeof(PreflightCycle)));
    CUDA_OK(hipMemcpy(d_preflight.cycles,
                       preflight->cycles,
                       cycles * sizeof(PreflightCycle),
                       hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&d_preflight.txns, preflight->txnsLen * sizeof(MemoryTransaction)));
    CUDA_OK(hipMemcpy(d_preflight.txns,
                       preflight->txns,
                       preflight->txnsLen * sizeof(MemoryTransaction),
                       hipMemcpyHostToDevice));

    d_preflight.txnsLen = preflight->txnsLen;
    d_preflight.tableSplitCycle = preflight->tableSplitCycle;

    CUDA_OK(hipMalloc(&ctx->preflight, sizeof(PreflightTrace)));
    CUDA_OK(
        hipMemcpy(ctx->preflight, &d_preflight, sizeof(PreflightTrace), hipMemcpyHostToDevice));

    CUDA_OK(hipMalloc(&d_tables.tableU8, (1 << 8) * sizeof(uint32_t)));
    CUDA_OK(hipMemset(d_tables.tableU8, 0, (1 << 8) * sizeof(uint32_t)));

    CUDA_OK(hipMalloc(&d_tables.tableU16, (1 << 16) * sizeof(uint32_t)));
    CUDA_OK(hipMemset(d_tables.tableU16, 0, (1 << 16) * sizeof(uint32_t)));

    CUDA_OK(hipMalloc(&ctx->tables, sizeof(LookupTables)));
    CUDA_OK(hipMemcpy(ctx->tables, &d_tables, sizeof(LookupTables), hipMemcpyHostToDevice));
  }

  ~HostAccumContext() {
    hipFree(d_tables.tableU16);
    hipFree(d_tables.tableU8);
    hipFree(ctx->tables);
    hipFree(d_preflight.txns);
    hipFree(d_preflight.cycles);
    hipFree(ctx->preflight);
    hipFree(ctx->mix);
    hipFree(ctx->global);
    hipFree(ctx->accum);
    hipFree(ctx->data);
    hipFree(ctx);
  }
};

__device__ ::cuda::std::array<uint32_t, 2>
divide_rv32im(uint32_t numer, uint32_t denom, uint32_t signType) {
  uint32_t onesComp = (signType == 2);
  bool negNumer = signType && int32_t(numer) < 0;
  bool negDenom = signType == 1 && int32_t(denom) < 0;
  if (negNumer) {
    numer = -numer - onesComp;
  }
  if (negDenom) {
    denom = -denom - onesComp;
  }
  uint32_t quot;
  uint32_t rem;
  if (denom == 0) {
    quot = 0xffffffff;
    rem = numer;
  } else {
    quot = numer / denom;
    rem = numer % denom;
  }
  uint32_t quotNegOut = (negNumer ^ negDenom) - ((denom == 0) * negNumer);
  uint32_t remNegOut = negNumer;
  if (quotNegOut) {
    quot = -quot - onesComp;
  }
  if (remNegOut) {
    rem = -rem - onesComp;
  }
  return {quot, rem};
}

__device__ ::cuda::std::array<Val, 5> extern_getMemoryTxn(ExecContext& ctx, Val addrElem) {
  uint32_t addr = addrElem.asUInt32();
  size_t txnIdx = ctx.preflight.cycles[ctx.cycle].txnIdx++;
  const MemoryTransaction& txn = ctx.preflight.txns[txnIdx];
  // printf("getMemoryTxn(%lu, 0x%08x): txn(%u, 0x%08x, 0x%08x)\n",
  //        ctx.cycle,
  //        addr,
  //        txn.cycle,
  //        txn.addr,
  //        txn.word);

  if (txn.cycle / 2 != ctx.cycle) {
    printf("txn.cycle: %u, ctx.cycle: %zu\n", txn.cycle, ctx.cycle);
    assert(false && "txn cycle mismatch");
  }

  if (txn.addr != addr) {
    printf("txn.addr: 0x%08x, addr: 0x%08x\n", txn.addr, addr);
    assert(false && "memory peek not in preflight");
  }
  return {
      txn.prevCycle,
      txn.prevWord & 0xffff,
      txn.prevWord >> 16,
      txn.word & 0xffff,
      txn.word >> 16,
  };
}

__device__ void extern_lookupDelta(ExecContext& ctx, Val table, Val index, Val count) {
  // printf("lookupDelta(table: %u, index: %u, count: %u, P: %u)\n",
  //        table.asUInt32(),
  //        index.asUInt32(),
  //        count.asUInt32(),
  //        Fp::P);
  ctx.tables.lookupDelta(table, index, count);
}

__device__ Val extern_lookupCurrent(ExecContext& ctx, Val table, Val index) {
  Val ret = ctx.tables.lookupCurrent(table, index);
  // printf("lookupCurrent(table: %u, index: %u): %u\n",
  //        table.asUInt32(),
  //        index.asUInt32(),
  //        ret.asUInt32());
  return ret;
}

__device__ void
extern_memoryDelta(ExecContext& ctx, Val addr, Val cycle, Val dataLow, Val dataHigh, Val count) {
  // printf("memoryDelta\n");
  // ctx.tables.memoryDelta(
  //     addr.asUInt32(), cycle.asUInt32(), dataLow.asUInt32() | (dataHigh.asUInt32() << 16),
  //     count);
}

__device__ uint32_t extern_getDiffCount(ExecContext& ctx, Val cycle) {
  // printf("getDiffCount\n");
  uint32_t cycleU32 = cycle.asUInt32();
  return ctx.preflight.cycles[cycleU32 / 2].diffCount[cycleU32 % 2];
}

__device__ Val extern_isFirstCycle_0(ExecContext& ctx) {
  // printf("isFirstCycle\n");
  return ctx.cycle == 0;
}

__device__ ::cuda::std::array<Val, 4> extern_divide(
    ExecContext& ctx, Val numerLow, Val numerHigh, Val denomLow, Val denomHigh, Val signType) {
  // printf("divide\n");
  uint32_t numer = numerLow.asUInt32() | (numerHigh.asUInt32() << 16);
  uint32_t denom = denomLow.asUInt32() | (denomHigh.asUInt32() << 16);
  auto [quot, rem] = divide_rv32im(numer, denom, signType.asUInt32());
  ::cuda::std::array<Val, 4> ret;
  ret[0] = quot & 0xffff;
  ret[1] = quot >> 16;
  ret[2] = rem & 0xffff;
  ret[3] = rem >> 16;
  return ret;
}

__device__ void extern_print(ExecContext& ctx, Val v) {
  // printf("LOG: %u\n", v.asUInt32());
}

__device__ ::cuda::std::array<Val, 2> extern_getMajorMinor(ExecContext& ctx) {
  uint8_t major = ctx.preflight.cycles[ctx.cycle].major;
  uint8_t minor = ctx.preflight.cycles[ctx.cycle].minor;
  // printf("getMajorMinor: %u, %u\n", major, minor);
  return {major, minor};
}

__device__ Val extern_hostReadPrepare(ExecContext& ctx, Val fp, Val len) {
  size_t txnIdx = ctx.preflight.cycles[ctx.cycle].txnIdx;
  uint32_t word = ctx.preflight.txns[txnIdx].word;
  // printf("[%lu]: hostReadPrepare(txnIdx: %zu, word: 0x%08x)\n", ctx.cycle, txnIdx, word);
  return word;
}

__device__ Val
extern_hostWrite(ExecContext& ctx, Val fdVal, Val addrLow, Val addrHigh, Val lenVal) {
  // printf("hostWrite\n");
  size_t txnIdx = ctx.preflight.cycles[ctx.cycle].txnIdx;
  return ctx.preflight.txns[txnIdx].word;
}

__device__ ::cuda::std::array<Val, 2> extern_nextPagingIdx(ExecContext& ctx) {
  uint32_t pagingIdx = ctx.preflight.cycles[ctx.cycle].pagingIdx;
  uint32_t machineMode = ctx.preflight.cycles[ctx.cycle].machineMode;
  // printf("nextPagingIdx: (0x%05x, %u)\n", pagingIdx, machineMode);
  return {pagingIdx, machineMode};
}

__device__ ::cuda::std::array<Val, 16> extern_bigIntExtern(ExecContext& ctx) {
  ::cuda::std::array<Val, 16> ret;
  size_t bigintIdx = ctx.preflight.cycles[ctx.cycle].bigintIdx;
  for (size_t i = 0; i < 16; i++) {
    ret[i] = ctx.preflight.bigintBytes[bigintIdx + i];
  }
  return ret;
}

__device__ void nextStep(DeviceExecContext* ctx, uint32_t cycle) {
  // printf("nextStep: %u\n", cycle);
  ExecContext execCtx(*ctx->preflight, *ctx->tables, cycle);
  MutableBufObj data(*ctx->data);
  GlobalBufObj global(*ctx->global);
  step_Top(execCtx, &data, &global);
}

__global__ void par_stepExec(DeviceExecContext* ctx, uint32_t start, uint32_t count) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }
  nextStep(ctx, start + cycle);
}

__global__ void rev_stepExec(DeviceExecContext* ctx, uint32_t split, uint32_t lastCycle) {
  for (uint32_t cycle = split; cycle-- > 0;) {
    nextStep(ctx, cycle);
  }
  for (uint32_t cycle = lastCycle; cycle-- > split;) {
    nextStep(ctx, cycle);
  }
}

__global__ void fwd_stepExec(DeviceExecContext* ctx, uint32_t count) {
  for (uint32_t cycle = 0; cycle < count; cycle++) {
    nextStep(ctx, cycle);
  }
}

__global__ void stepAccum(DeviceAccumContext* ctx, uint32_t count) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  ExecContext execCtx(*ctx->preflight, *ctx->tables, cycle);
  MutableBufObj data(*ctx->data);
  MutableBufObj accum(*ctx->accum, /*zeroBack=*/kUserAccumSplit);
  GlobalBufObj mix(*ctx->mix);
  GlobalBufObj global(*ctx->global);
  step_TopAccum(execCtx, &accum, &data, &global, &mix);
}

__global__ void finalizeAccum(DeviceAccumContext* ctx, uint32_t lastCycle) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= lastCycle) {
    return;
  }

  Buffer& accum = *ctx->accum;

  size_t machineColumns = (accum.cols - kUserAccumSplit) / 4;
  size_t back1 = (cycle + lastCycle - 1) % lastCycle;
  Fp prev[4];
  for (size_t k = 0; k < 4; k++) {
    prev[k] = accum.get(back1, accum.cols - 4 + k);
  }
  for (size_t j = 0; j < machineColumns - 1; j++) {
    for (size_t k = 0; k < 4; k++) {
      size_t col = kUserAccumSplit + j * 4 + k;
      accum.set(cycle, col, accum.get(cycle, col) + prev[k]);
    }
  }
}

} // namespace risc0::circuit::rv32im_v2::cuda

constexpr size_t kStepModeParallel = 0;
constexpr size_t kStepModeSeqForward = 1;
constexpr size_t kStepModeSeqReverse = 2;

extern "C" {

using namespace risc0::circuit::rv32im_v2::cuda;

const char* risc0_circuit_rv32im_v2_cuda_witgen(uint32_t mode,
                                                ExecBuffers* buffers,
                                                PreflightTrace* preflight,
                                                uint32_t lastCycle) {
  try {
    HostExecContext ctx(buffers, preflight, lastCycle);
    CudaStream stream;
    size_t split = preflight->tableSplitCycle;

    switch (mode) {
    case kStepModeParallel: {
      auto cfg1 = getSimpleConfig(split);
      size_t phase2Count = lastCycle - split;
      // printf("phase1: %zu, phase2: %zu\n", split, phase2Count);
      auto cfg2 = getSimpleConfig(phase2Count);
      {
        nvtx3::scoped_range range("phase1");
        par_stepExec<<<cfg1.grid, cfg1.block, 0, stream>>>(ctx.ctx, 0, split);
        CUDA_OK(hipStreamSynchronize(stream));
      }
      {
        nvtx3::scoped_range range("phase2");
        par_stepExec<<<cfg2.grid, cfg2.block, 0, stream>>>(ctx.ctx, split, phase2Count);
        CUDA_OK(hipStreamSynchronize(stream));
      }
    } break;
    case kStepModeSeqForward:
      fwd_stepExec<<<1, 1, 0, stream>>>(ctx.ctx, lastCycle);
      CUDA_OK(hipStreamSynchronize(stream));
      break;
    case kStepModeSeqReverse:
      rev_stepExec<<<1, 1, 0, stream>>>(ctx.ctx, split, lastCycle);
      CUDA_OK(hipStreamSynchronize(stream));
      break;
    }
  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

const char* risc0_circuit_rv32im_v2_cuda_accum(AccumBuffers* buffers,
                                               PreflightTrace* preflight,
                                               uint32_t lastCycle) {
  try {
    HostAccumContext ctx(buffers, preflight, lastCycle);
    CudaStream stream;
    auto cfg = getSimpleConfig(lastCycle);

    {
      nvtx3::scoped_range range("phase1");
      stepAccum<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, lastCycle);
      CUDA_OK(hipStreamSynchronize(stream));
    }

    {
      nvtx3::scoped_range range("phase2");
      size_t rows = buffers->accum.rows;
      for (size_t j = 0; j < 4; j++) {
        size_t col = buffers->accum.cols - 4 + j;
        Fp* itBegin = buffers->accum.buf + col * rows;
        Fp* itEnd = buffers->accum.buf + col * rows + lastCycle;
        thrust::inclusive_scan(thrust::device, itBegin, itEnd, itBegin);
      }
      CUDA_OK(hipStreamSynchronize(stream));
    }

    {
      nvtx3::scoped_range range("phase3");
      finalizeAccum<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, lastCycle);
      CUDA_OK(hipStreamSynchronize(stream));
    }

  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

} // extern "C"
