#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "eval_check.cuh"

#include "hip/hip_runtime.h"
#include "supra/fp.h"

#include <exception>

namespace risc0::circuit::rv32im_v2::cuda {

__constant__ FpExt poly_mix[kNumPolyMixPows];

__global__ void eval_check(Fp* check,
                           const Fp* ctrl,
                           const Fp* data,
                           const Fp* accum,
                           const Fp* mix,
                           const Fp* out,
                           const Fp rou,
                           uint32_t po2,
                           uint32_t domain) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle < domain) {
    FpExt tot = poly_fp(cycle, domain, ctrl, out, data, mix, accum);
    Fp x = pow(rou, cycle);
    Fp y = pow(Fp(3) * x, 1 << po2);
    FpExt ret = tot * inv(y - Fp(1));
    check[domain * 0 + cycle] = ret[0];
    check[domain * 1 + cycle] = ret[1];
    check[domain * 2 + cycle] = ret[2];
    check[domain * 3 + cycle] = ret[3];
  }
}

} // namespace risc0::circuit::rv32im_v2::cuda

using namespace risc0::circuit::rv32im_v2::cuda;

extern "C" {

const char* risc0_circuit_rv32im_v2_cuda_eval_check(Fp* check,
                                                    const Fp* ctrl,
                                                    const Fp* data,
                                                    const Fp* accum,
                                                    const Fp* mix,
                                                    const Fp* out,
                                                    const Fp& rou,
                                                    uint32_t po2,
                                                    uint32_t domain,
                                                    const FpExt* poly_mix_pows) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(domain);
    hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
    eval_check<<<cfg.grid, cfg.block, 0, stream>>>(
        check, ctrl, data, accum, mix, out, rou, po2, domain);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

} // extern "C"
