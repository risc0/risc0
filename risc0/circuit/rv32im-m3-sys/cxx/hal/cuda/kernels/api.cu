#include "hip/hip_runtime.h"
// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <ff/baby_bear.hpp>

#include <util/gpu_t.cuh>
#include <util/rusterror.h>

#include <polynomial/div_by_x_minus_z.cuh>

#include <thrust/execution_policy.h>
#include <thrust/scan.h>

extern "C" void prefix_sum(fr_t* buf, uint32_t count) {
  thrust::inclusive_scan(thrust::device, buf, buf + count, buf);
}

extern "C" RustError::by_value
rv32im_m3_poly_divide(fr4_t d_inout[/*len*/], size_t len, fr4_t* remainder, fr4_t pow) {
  const gpu_t& gpu = select_gpu();
  try {
    div_by_x_minus_z<true>(d_inout, len, pow, gpu);
    gpu.DtoH(remainder, &d_inout[len - 1], 1);
    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  }

  return RustError{hipSuccess};
}
