// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#define FEATURE_BABY_BEAR 1
#include <ff/baby_bear.hpp>
#include <ntt/ntt.cuh>

extern "C" RustError::by_value sppark_init() {
  uint32_t lg_domain_size = 1;
  uint32_t domain_size = 1U << lg_domain_size;

  std::vector<fr_t> inout{domain_size};
  inout[0] = fr_t(1);
  inout[1] = fr_t(1);

  const gpu_t& gpu = select_gpu();

  try {
    CUDA_OK(hipDeviceSynchronize());

    NTT::Base(gpu,
              &inout[0],
              lg_domain_size,
              NTT::InputOutputOrder::NR,
              NTT::Direction::forward,
              NTT::Type::standard);
    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  } catch (...) {
    return RustError(hipErrorUnknown, "Generic exception");
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value sppark_batch_expand(
    fr_t* d_out, fr_t* d_in, uint32_t lg_domain_size, uint32_t lg_blowup, uint32_t poly_count) {
  if (lg_domain_size == 0)
    return RustError{hipSuccess};

  uint32_t domain_size = 1U << lg_domain_size;
  uint32_t ext_domain_size = domain_size << lg_blowup;

  const gpu_t& gpu = select_gpu();

  try {
    CUDA_OK(hipDeviceSynchronize());

    for (size_t c = 0; c < poly_count; c++) {
      NTT::LDE_expand(
          gpu, &d_out[c * ext_domain_size], &d_in[c * domain_size], lg_domain_size, lg_blowup);
    }

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  } catch (...) {
    return RustError(hipErrorUnknown, "Generic exception");
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_batch_NTT(fr_t* d_inout, uint32_t lg_domain_size, uint32_t poly_count) {
  if (lg_domain_size == 0)
    return RustError{hipSuccess};

  uint32_t domain_size = 1U << lg_domain_size;

  const gpu_t& gpu = select_gpu();

  try {
    CUDA_OK(hipDeviceSynchronize());

    for (size_t c = 0; c < poly_count; c++) {
      NTT::Base_dev_ptr(gpu,
                        &d_inout[c * domain_size],
                        lg_domain_size,
                        NTT::InputOutputOrder::RN,
                        NTT::Direction::forward,
                        NTT::Type::standard);
    }

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  } catch (...) {
    return RustError(hipErrorUnknown, "Generic exception");
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_batch_iNTT(fr_t* d_inout, uint32_t lg_domain_size, uint32_t poly_count) {
  if (lg_domain_size == 0)
    return RustError{hipSuccess};

  uint32_t domain_size = 1U << lg_domain_size;

  const gpu_t& gpu = select_gpu();

  try {
    CUDA_OK(hipDeviceSynchronize());

    for (size_t c = 0; c < poly_count; c++) {
      NTT::Base_dev_ptr(gpu,
                        &d_inout[c * domain_size],
                        lg_domain_size,
                        NTT::InputOutputOrder::NR,
                        NTT::Direction::inverse,
                        NTT::Type::standard);
    }

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  } catch (...) {
    return RustError(hipErrorUnknown, "Generic exception");
  }

  return RustError{hipSuccess};
}

extern "C" RustError::by_value
sppark_batch_zk_shift(fr_t* d_inout, uint32_t lg_domain_size, uint32_t poly_count) {
  if (lg_domain_size == 0)
    return RustError{hipSuccess};

  uint32_t domain_size = 1U << lg_domain_size;

  const gpu_t& gpu = select_gpu();

  try {
    CUDA_OK(hipDeviceSynchronize());

    for (size_t c = 0; c < poly_count; c++) {
      NTT::LDE_powers(gpu, &d_inout[c * domain_size], lg_domain_size);
    }

    gpu.sync();
  } catch (const cuda_error& e) {
    gpu.sync();
    return RustError{e.code(), e.what()};
  } catch (...) {
    return RustError(hipErrorUnknown, "Generic exception");
  }

  return RustError{hipSuccess};
}
