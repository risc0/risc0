#include "hip/hip_runtime.h"
// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"

__global__ void query_kernel(Fp* out,
                  const Fp* data,
                  const Fp* tree,
                  uint32_t querySize,
                  uint32_t rows,
                  uint32_t cols,
                  uint32_t idx) {
  uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= querySize) { return; }
  if (i < cols) {
    out[i] = data[rows * i + idx];
  } else {
    uint i2 = i - cols;
    uint up = i2 / 8;
    uint elem = i2 % 8;
    uint cidx = (idx + rows)  >> up;
    uint other = (cidx % 2) ? cidx - 1 : cidx + 1;
    out[i] = tree[8*other + elem];
  }
}

extern "C" bool cuda_query(Fp* out, Fp* data, Fp* tree, size_t querySize, size_t rows, size_t cols, size_t idx) {
  size_t block_size = querySize < 256 ? querySize : 256;
  size_t num_blocks = (querySize + block_size - 1) / block_size;
  query_kernel<<<num_blocks, block_size, 0>>>(out, data, tree, querySize, rows, cols, idx);
  return true;
}
