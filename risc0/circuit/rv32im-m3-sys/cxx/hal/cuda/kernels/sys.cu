// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>

#include <iostream>

extern "C" void* cuda_malloc(size_t size) {
  void* out;
  hipError_t err = hipMalloc(&out, size);
  if (err != hipSuccess) {
    return nullptr;
  }
  return out;
}

extern "C" void cuda_free(void* buf) {
  hipFree(buf);
}

extern "C" bool cuda_sync() {
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    fprintf(stderr,"Failed to sync: %s\n", hipGetErrorString(err));
  }
  return err == hipSuccess;
}

extern "C" bool cuda_copy_to_host_sync(void* host, void *dev, size_t size) {
  hipError_t err = hipMemcpy(host, dev, size,  hipMemcpyDeviceToHost);
  return err == hipSuccess;
}

extern "C" bool cuda_copy_to_dev_sync(void* dev, void *host, size_t size) {
  hipError_t err = hipMemcpy(dev, host, size,  hipMemcpyHostToDevice);
  return err == hipSuccess;
}

extern "C" bool cuda_copy_dev(void* dst, void *src, size_t size) {
  hipError_t err = hipMemcpy(dst, src, size,  hipMemcpyDeviceToDevice);
  return err == hipSuccess;
}

extern "C" bool cuda_zero_dev(void* buf, size_t size) {
  hipError_t err = hipMemset(buf, 0, size);
  return err == hipSuccess;
}


