// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "hal/po2s.h"
#include "hal/cuda/kernels/base.h"

#define PO2(x) \
extern "C" void data_witgen_cuda_ ## x(Fp* data, Fp* globals, const RowInfo* info, const uint32_t* aux, uint32_t* tables, Fp rou);
PO2S
#undef PO2

extern "C" void data_witgen_cuda(Fp* data, Fp* globals, const RowInfo* info, const uint32_t* aux, uint32_t* tables, Fp rou, uint32_t numRows) {
  uint32_t po2 = 31 - __builtin_clz(numRows);
  switch(po2) {
#define PO2(x) \
    case x: \
      data_witgen_cuda_ ## x(data, globals, info, aux, tables, rou); \
      break;
PO2S
#undef PO2
  }
}
