#include "hip/hip_runtime.h"
// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"

constexpr size_t kFriFold = 16;

/// Compute `ceil(log_2(in))`, i.e. find the smallest value `out` such that `2^out >= in`.
__device__ inline constexpr size_t log2Ceil(size_t in) {
  size_t r = 0;
  while ((1 << r) < in) {
    r++;
  }
  return r;
}

__global__ void batch_bit_reverse(Fp* io, const uint32_t nBits, const uint32_t count) {
  uint totIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (totIdx < count) {
    uint32_t rowSize = 1 << nBits;
    uint32_t idx = totIdx & (rowSize - 1);
    uint32_t s = totIdx >> nBits;
    uint32_t ridx = __brev(idx) >> (32 - nBits);
    if (idx < ridx) {
      size_t idx1 = s * rowSize + idx;
      size_t idx2 = s * rowSize + ridx;
      Fp tmp = io[idx1];
      io[idx1] = io[idx2];
      io[idx2] = tmp;
    }
  }
}

extern "C" bool cuda_batch_bit_reverse(Fp* io, uint32_t po2, uint32_t cols) {
  uint32_t count = (uint32_t(1) << po2) * cols;
  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;
  batch_bit_reverse<<<num_blocks, block_size, 0>>>(io, po2, count);
  return true;
}

__global__ void batch_evaluate_any(
    FpExt* out, const Fp* coeffs, const uint32_t* which, const FpExt* xs, uint32_t deg) {
  const Fp* cur_poly = coeffs + which[blockIdx.x] * deg;
  FpExt x = xs[blockIdx.x];
  FpExt stepx = pow(x, blockDim.x);
  FpExt powx = pow(x, threadIdx.x);
  FpExt tot(0);
  for (size_t i = threadIdx.x; i < deg; i += blockDim.x) {
    tot += powx * cur_poly[i];
    powx *= stepx;
  }
  extern __shared__ uint32_t totsBuf[];
  FpExt* tots = reinterpret_cast<FpExt*>(totsBuf);
  tots[threadIdx.x] = tot;
  __syncthreads();
  unsigned cur = blockDim.x;
  while (cur) {
    cur /= 2;
    if (threadIdx.x < cur) {
      tots[threadIdx.x] = FpExt(tots[threadIdx.x]) + FpExt(tots[threadIdx.x + cur]);
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    out[blockIdx.x] = tots[0];
  }
}

extern "C" bool cuda_batch_evaluate_any(FpExt* out, const Fp* coeffs, const uint32_t* which, const FpExt* xs, uint32_t outSize, uint32_t deg) {
  size_t block_size = (deg < 256 ? deg : 256);
  size_t num_blocks = outSize;
  batch_evaluate_any<<<num_blocks, block_size, block_size * sizeof(FpExt)>>>(out, coeffs, which, xs, deg);
  return true;
}

__global__ void fri_fold(Fp* out, const Fp* in, FpExt mix, uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    FpExt tot;
    FpExt curMix(1);
    for (uint32_t i = 0; i < kFriFold; i++) {
      size_t rev_i = __brev(i) >> (32 - log2Ceil(kFriFold));
      size_t rev_idx = rev_i * count + idx;
      FpExt factor(in[0 * count * kFriFold + rev_idx],
                   in[1 * count * kFriFold + rev_idx],
                   in[2 * count * kFriFold + rev_idx],
                   in[3 * count * kFriFold + rev_idx]);
      tot += curMix * factor;
      curMix *= mix;
    }
    for (size_t i = 0; i < 4; i++) {
      out[count * i + idx] = tot.elem(i);
    }
  }
}

extern "C" bool cuda_fri_fold(Fp* out, const Fp* in, FpExt mix, uint32_t count) {
  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;
  fri_fold<<<num_blocks, block_size, 0>>>(out, in, mix, count);
  return true;
}

__global__ void gather_sample(
    Fp* dst, const Fp* src, const uint32_t idx, const uint32_t size, const uint32_t stride) {
  uint gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    dst[gid] = src[gid * stride + idx];
  }
}

__global__ void scatter(Fp* into,
                        const uint32_t* index,
                        const uint32_t* offsets,
                        const Fp* values,
                        uint32_t count) {
  uint gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < count) {
    for (uint32_t idx = index[gid]; idx < index[gid + 1]; idx++) {
      into[offsets[idx]] = values[idx];
    }
  }
}

__global__ void mix_poly_coeffs(FpExt* out,
                                const Fp* in,
                                const uint32_t* combos,
                                FpExt mixStart,
                                FpExt mix,
                                uint32_t inputSize,
                                uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    FpExt cur = mixStart;
    for (size_t i = 0; i < inputSize; i++) {
      size_t id = combos[i];
      out[count * id + idx] += cur * in[count * i + idx];
      cur *= mix;
    }
  }
}

extern "C" bool cuda_mix_poly_coeffs(FpExt* out,
                                     const Fp* in,
                                     const uint32_t* combos,
                                     FpExt mixStart,
                                     FpExt mix,
                                     uint32_t inputSize,
                                     uint32_t count) {
  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;
  mix_poly_coeffs<<<num_blocks, block_size, 0>>>(out, in, combos, mixStart, mix, inputSize, count);
  return true;
}

struct EvalInfo {
  uint32_t group;
  uint32_t column;
  uint32_t comboId;
  uint32_t coeffIndex;
};

__global__ void combos_prepare(FpExt* combos,
                               const FpExt* eval,
                               const EvalInfo* info,
                               FpExt mix,
                               uint32_t rows,
                               uint32_t evalSize) {
  FpExt cur(1);
  for (size_t i = 0; i < evalSize; i++) {
    combos[info[i].coeffIndex + info[i].comboId*rows] -= cur * eval[i];
    if (i + 1 < evalSize && (info[i + 1].group != info[i].group || info[i + 1].column != info[i].column)) {
      cur *= mix;
    }
  }
}

extern "C" bool cuda_combos_prepare(FpExt* combos, const FpExt* eval, const EvalInfo* info, FpExt mix, uint32_t rows, uint32_t evalSize) {
  combos_prepare<<<1, 1, 0>>>(combos, eval, info, mix, rows, evalSize);
  return true;
}

__global__ void combos_finalize(Fp* out, const FpExt* combos, uint32_t numCombos, uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    FpExt tot;
    for (size_t i = 0; i < numCombos; i++) {
      tot += combos[i * count + idx];
    }
    for (size_t i = 0; i < 4; i++) {
      out[i * count + idx] = elem(tot, i);
    }
  }
}

extern "C" bool cuda_combos_finalize(Fp* out, const FpExt* combos, uint32_t numCombos, uint32_t count) {
  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;
  combos_finalize<<<num_blocks, block_size, 0>>>(out, combos, numCombos, count);
  return true;
}

