#include "hip/hip_runtime.h"
// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"

#include "zkp/cuda/kernels.h"

/// Compute `ceil(log_2(in))`, i.e. find the smallest value `out` such that
/// `2^out >= in`.
__device__ inline constexpr size_t log2Ceil(size_t in) {
  size_t r = 0;
  while ((1 << r) < in) {
    r++;
  }
  return r;
}

extern "C" bool cuda_batch_bit_reverse(Fp* io, uint32_t po2, uint32_t cols) {
  uint32_t count = (uint32_t(1) << po2) * cols;
  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;
  batch_bit_reverse<<<num_blocks, block_size, 0>>>(io, po2, count);
  return true;
}

extern "C" bool cuda_batch_evaluate_any(FpExt* out,
                                        const Fp* coeffs,
                                        const uint32_t* which,
                                        const FpExt* xs,
                                        uint32_t outSize,
                                        uint32_t deg) {
  size_t block_size = (deg < 256 ? deg : 256);
  size_t num_blocks = outSize;
  batch_evaluate_any<<<num_blocks, block_size, block_size * sizeof(FpExt)>>>(
      out, coeffs, which, xs, deg);
  return true;
}

extern "C" bool cuda_fri_fold(Fp* out, const Fp* in, FpExt mix, uint32_t count) {
  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;
  fri_fold<<<num_blocks, block_size, 0>>>(out, in, mix, count);
  return true;
}

extern "C" bool cuda_mix_poly_coeffs(FpExt* out,
                                     const Fp* in,
                                     const uint32_t* combos,
                                     FpExt mixStart,
                                     FpExt mix,
                                     uint32_t inputSize,
                                     uint32_t count) {
  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;
  mix_poly_coeffs<<<num_blocks, block_size, 0>>>(out, in, combos, mixStart, mix, inputSize, count);
  return true;
}

struct EvalInfo {
  uint32_t group;
  uint32_t column;
  uint32_t comboId;
  uint32_t coeffIndex;
};

__global__ void rv32im_m3_combos_prepare(FpExt* combos,
                                         const FpExt* eval,
                                         const EvalInfo* info,
                                         FpExt mix,
                                         uint32_t rows,
                                         uint32_t evalSize) {
  FpExt cur(1);
  for (size_t i = 0; i < evalSize; i++) {
    combos[info[i].coeffIndex + info[i].comboId * rows] -= cur * eval[i];
    if (i + 1 < evalSize &&
        (info[i + 1].group != info[i].group || info[i + 1].column != info[i].column)) {
      cur *= mix;
    }
  }
}

extern "C" bool cuda_combos_prepare(FpExt* combos,
                                    const FpExt* eval,
                                    const EvalInfo* info,
                                    FpExt mix,
                                    uint32_t rows,
                                    uint32_t evalSize) {
  rv32im_m3_combos_prepare<<<1, 1, 0>>>(combos, eval, info, mix, rows, evalSize);
  return true;
}

__global__ void combos_finalize(Fp* out, const FpExt* combos, uint32_t numCombos, uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    FpExt tot;
    for (size_t i = 0; i < numCombos; i++) {
      tot += combos[i * count + idx];
    }
    for (size_t i = 0; i < 4; i++) {
      out[i * count + idx] = elem(tot, i);
    }
  }
}

extern "C" bool
cuda_combos_finalize(Fp* out, const FpExt* combos, uint32_t numCombos, uint32_t count) {
  size_t block_size = count < 256 ? count : 256;
  size_t num_blocks = (count + block_size - 1) / block_size;
  combos_finalize<<<num_blocks, block_size, 0>>>(out, combos, numCombos, count);
  return true;
}
