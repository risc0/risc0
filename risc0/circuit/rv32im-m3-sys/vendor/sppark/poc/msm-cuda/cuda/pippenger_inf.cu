// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381-fp2.hpp>
#elif defined(FEATURE_BLS12_377)
# include <ff/bls12-377-fp2.hpp>
#elif defined(FEATURE_BN254)
# include <ff/alt_bn128-fp2.hpp>
#else
# error "no FEATURE"
#endif

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_inf_t affine_t;
typedef fr_t scalar_t;

#define SPPARK_DONT_INSTANTIATE_TEMPLATES
#include <msm/pippenger.cuh>

extern "C"
RustError::by_value mult_pippenger_inf(point_t* out, const affine_t points[],
                                       size_t npoints, const scalar_t scalars[],
                                       size_t ffi_affine_sz)
{
    return mult_pippenger<bucket_t>(out, points, npoints, scalars, false, ffi_affine_sz);
}

#if defined(FEATURE_BLS12_381) || defined(FEATURE_BLS12_377) || defined(FEATURE_BN254)
typedef jacobian_t<fp2_t> point_fp2_t;
typedef xyzz_t<fp2_t> bucket_fp2_t;
typedef bucket_fp2_t::affine_inf_t affine_fp2_t;

extern "C"
RustError::by_value mult_pippenger_fp2_inf(point_fp2_t* out, const affine_fp2_t points[],
                                           size_t npoints, const scalar_t scalars[],
                                           size_t ffi_affine_sz)
{
    return mult_pippenger<bucket_fp2_t>(out, points, npoints, scalars, false, ffi_affine_sz);
}
#endif
