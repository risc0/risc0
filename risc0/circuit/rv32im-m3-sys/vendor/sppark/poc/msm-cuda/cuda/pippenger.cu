// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ff/bls12-381.hpp>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__
extern "C"
RustError mult_pippenger(point_t* out, const affine_t points[], size_t npoints,
                                       const scalar_t scalars[])
{
    return mult_pippenger<bucket_t>(out, points, npoints, scalars, false);
}
#endif
