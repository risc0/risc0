
#include <hip/hip_runtime.h>
// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

// #include <util/cuda_available.cu> or compile separately...

#if defined(__NVCC__) && !defined(__CUDA_ARCH__)
#include <atomic>

extern "C"
bool cuda_available()
{
    static std::atomic<int> available(-1);
    int ret = available;
    if (ret < 0) {
        cudaDeviceProp prop;
        ret = cudaGetDeviceProperties(&prop, 0) == cudaSuccess &&
              prop.major >= 7;
        available = ret;
    }
    return (bool)ret;
}
#endif
