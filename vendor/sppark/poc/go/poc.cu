
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#ifdef _MSC_VER
# define strdup _strdup
#endif

__global__ void kernel()
{
    printf("hello from GPU\n");
}

struct Error {
    int code;
    char *message;
};

extern "C"
#if defined(_WIN32) && !defined(__HIP_DEVICE_COMPILE__)
__declspec(dllexport)
#else
__attribute__((visibility("default")))
#endif
Error cuda_func(void *ptr)
{
    kernel<<<1,1>>>();
    auto err = hipGetLastError();
    if (err != hipSuccess)
        return {err, strdup(hipGetErrorString(err))};
    err = hipDeviceSynchronize();
    return {err, strdup(hipGetErrorString(err))};
    (void)ptr;
}
